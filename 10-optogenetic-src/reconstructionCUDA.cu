#include "hip/hip_runtime.h"
#pragma once
//#include "header.cuh"
//#include "initANDcheck.h"
#include "reconstructionCUDA.cuh"



__global__ void Zhuan_Complex_kernel(float *PSF_1_gpu, hipfftComplex *PSF_1_gpu_Complex, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		PSF_1_gpu_Complex[i].x = PSF_1_gpu[i];
		PSF_1_gpu_Complex[i].y = 0;
	}
}
__global__ void PSF_unshort(float *PSF_1_gpu, unsigned short *PSF, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		if (PSF_1_gpu[i] < 0)
		{
			PSF[i] = 0;
		}
		else if (PSF_1_gpu[i] > 65535)
		{
			PSF[i] = 65535;
		}
		else
		{
			PSF[i] = (int)(PSF_1_gpu[i] + 0.5);
		}
	}
}
__global__ void initial_kernel_1(float *ImgEst, float *Ratio, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		ImgEst[i] = 0;
		Ratio[i] = 1;
	}
}
__global__ void gpuObjRecon_fuzhi(float *gpuObjRecon, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		gpuObjRecon[i] = 1;
	}
}
__global__ void initial_kernel_3(float *gpuObjRecROI, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		gpuObjRecROI[i] = 1;
	}
}
__global__ void ImgExp_ge(unsigned short *Img_gpu, int BkgMean, float *ImgExp, int total)
{
	//Turn the result of the difference less than 0 into 0, greater than 0 rounded, greater than 65535 into 65535
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		if ((Img_gpu[i] - BkgMean) < 0)
		{
			ImgExp[i] = 0;
		}
		else if ((Img_gpu[i] - BkgMean) > 65535)
		{
			ImgExp[i] = 65535;
		}
		else
		{
			ImgExp[i] = (int)((Img_gpu[i] - BkgMean) + 0.5);
		}
	}
}
__global__ void Ratio_fuzhi(float *Ratio, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		Ratio[i] = 1;
	}
}
__global__ void OTF_mul_gpuObjRecon_Complex(hipfftComplex *OTF, hipfftComplex *gpuObjRecon_Complex, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		float aaa = OTF[i].x*gpuObjRecon_Complex[i].x - OTF[i].y*gpuObjRecon_Complex[i].y;//Real Department Results
		float bbb = OTF[i].x*gpuObjRecon_Complex[i].y + OTF[i].y*gpuObjRecon_Complex[i].x;//Virtual Part Results
		gpuObjRecon_Complex[i].x = aaa;
		gpuObjRecon_Complex[i].y = bbb;
	}
}
__global__ void ifftshift_real_max(hipfftComplex *OTF, float *float_temp, int PSF_size_1, int PSF_size_2, int PSF_size_3)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	const int k = blockDim.z * blockIdx.z + threadIdx.z;
	int lie_half = PSF_size_2 / 2;
	if (i < PSF_size_1 / 2 && j < PSF_size_2 && k < PSF_size_3)
	{
		//Implement the image ifftshift+real+max, i.e.: divide the image into 4 quadrants, swap the first and third translations, swap the second and fourth translations
		float_temp[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * 512] = OTF[k*PSF_size_1*PSF_size_2 + i * PSF_size_2 + j].x >= 0 ? OTF[k*PSF_size_1*PSF_size_2 + i * PSF_size_2 + j].x : 0;
		float_temp[k*PSF_size_1*PSF_size_2 + i * PSF_size_2 + j] = OTF[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * PSF_size_2].x >= 0 ? OTF[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * PSF_size_2].x : 0;
	}
}
__global__ void ifftshift(hipfftComplex *OTF, float *float_temp, int PSF_size_1, int PSF_size_2, int PSF_size_3, hipfftComplex *OTF_ifftshift)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	const int k = blockDim.z * blockIdx.z + threadIdx.z;
	int lie_half = PSF_size_2 / 2;
	if (i < PSF_size_1 / 2 && j < PSF_size_2 && k < PSF_size_3)
	{
		//Implement the image ifftshift, i.e.: divide the image into 4 quadrants, first and third translation swap, second and fourth translation swap
		OTF_ifftshift[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * 512] = OTF[k*PSF_size_1*PSF_size_2 + i * PSF_size_2 + j];
		OTF_ifftshift[k*PSF_size_1*PSF_size_2 + i * PSF_size_2 + j] = OTF[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * PSF_size_2];
	}
}
__global__ void float_temp_sum(float *float_temp, float *ImgEst, int PSF_size_1, int PSF_size_2, int PSF_size_3)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < PSF_size_1 && j < PSF_size_2)
	{
		ImgEst[i*PSF_size_2 + j] = 0;
		for (int k = 0; k < PSF_size_3; k++)
		{
			ImgEst[i*PSF_size_2 + j] += float_temp[k*PSF_size_1*PSF_size_2 + (i*PSF_size_2 + j)];
		}
	}
}
__global__ void Ratio_fuzhi_2(float *ImgExp, float *ImgEst, float Tmp, int SNR, float *Ratio, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		Ratio[i] = ImgExp[i] / (ImgEst[i] + Tmp / SNR);
	}
}
__global__ void Ratio_Complex_ge(float *ImgExp, float *ImgEst, float Tmp, int SNR, hipfftComplex *Ratio_Complex, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		Ratio_Complex[i].x = ImgExp[i] / (ImgEst[i] + Tmp / SNR);
		Ratio_Complex[i].y = 0;
	}
}
__global__ void fftRatio_ge(hipfftComplex *Ratio_Complex, hipfftComplex *fftRatio, int PSF_size_1, int PSF_size_2, int PSF_size_3)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	const int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i < PSF_size_1 && j < PSF_size_2 && k < PSF_size_3)
	{
		fftRatio[k*PSF_size_1*PSF_size_2 + i * PSF_size_2 + j] = Ratio_Complex[i*PSF_size_2 + j];
	}
}
__global__ void fftceshi_gpu_fuzhi(hipfftComplex *PSF_1_gpu_Complex, hipfftComplex *fftceshi_gpu, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		fftceshi_gpu[i] = PSF_1_gpu_Complex[i];
	}
}
__global__ void ifft2_divide(hipfftComplex *OTF, int total, int scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		OTF[i].x = OTF[i].x / scale;
		OTF[i].y = OTF[i].y / scale;
	}
}
__global__ void real_multiply(float *gpuObjRecon, float *float_temp, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		gpuObjRecon[i] = gpuObjRecon[i] * float_temp[i];
	}
}
__global__ void fftRatio_mul_conjOTF(hipfftComplex *fftRatio, hipfftComplex *OTF, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		float aaa = fftRatio[i].x*OTF[i].x + fftRatio[i].y*OTF[i].y;//Real Department Results
		float bbb = -fftRatio[i].x*OTF[i].y + fftRatio[i].y*OTF[i].x;//Virtual Part Results
		fftRatio[i].x = aaa;
		fftRatio[i].y = bbb;
	}
}


__global__ void cropReconImage_kernel(float *gpuObjRecon, float *gpuObjRecon_crop)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;//XObj
	const int y = blockDim.y * blockIdx.y + threadIdx.y;//YObj
	const int z = blockDim.z * blockIdx.z + threadIdx.z;//ZObj

	int line_start = 156;
	int line_end = 355;
	int line_total = 200;
	int col_start = 156;
	int col_end = 355;
	int col_total = 200;
	int band = 50;

	if (z < 50 && x < 200 && y < 200)
	{
		gpuObjRecon_crop[z * 200 * 200 + y * 200 + x] = gpuObjRecon[z*512*512 + (y + line_start)*255 + x + col_start];
		//gpuObjRecon_crop[z*200*200 + y * 200 + x] = gpuObjRecon[z * 512 * 512 + (256 - 100 + y) * 512 + 256 - 100 + x];
	}
}