#define _CRT_SECURE_NO_WARNINGS

#include"initANDcheck.h"

#include "gdal_alg.h";
#include "gdal_priv.h"
#include <gdal.h>

#include <chrono>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include<fstream>
#include <iomanip>

using namespace std;

void printDeviceProp(const hipDeviceProp_t &prop)
{
	printf("GPU Parament��\n");
	printf(" Device Name : %s.\n", prop.name);
	printf(" totalGlobalMem : %I64d.\n", prop.totalGlobalMem);
	printf(" sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf(" regsPerBlock : %d.\n", prop.regsPerBlock);
	printf(" warpSize : %d.\n", prop.warpSize);
	printf(" memPitch : %d.\n", prop.memPitch);
	printf(" maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf(" maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf(" maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf(" totalConstMem : %d.\n", prop.totalConstMem);
	printf(" major.minor : %d.%d.\n", prop.major, prop.minor);
	printf(" clockRate : %d.\n", prop.clockRate);
	printf(" textureAlignment : %d.\n", prop.textureAlignment);
	printf(" deviceOverlap : %d.\n", prop.deviceOverlap);
	printf(" multiProcessorCount : %d.\n", prop.multiProcessorCount);
	std::printf(" CUDA core: %d\r\n", 2 * prop.multiProcessorCount* prop.maxThreadsPerMultiProcessor / prop.warpSize);
	printf("\n\n");
}

bool InitCUDA()
{
	int count;
	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printDeviceProp(prop);
		int clockRate = prop.clockRate;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}
	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}
void check(hipError_t res, string warningstring)
{
	if (res != hipSuccess)
	{
		printf((warningstring + " !\n").c_str());
		system("pause");
		exit(0);
	}
}


void check1(hipError_t res, string warningstring, const char *file, int linenum)
{
	if (res != hipSuccess)
	{
		printf((warningstring + " !\n").c_str());
		printf("   Error text: %s   Error code: %d\n", hipGetErrorString(res), res);
		printf("   Line:    %d    File:    %s\n", linenum, file);
		system("pause");
		exit(0);
	}
}


void checkGPUStatus(hipError_t hipGetLastError, string warningstring)
{
	if (hipGetLastError != hipSuccess)
	{
		printf("\n\n");
		printf((warningstring + " !\n").c_str());
		fprintf(stderr, "%s\n", hipGetErrorString(hipGetLastError));
		system("pause");
		exit(0);
	}
}

