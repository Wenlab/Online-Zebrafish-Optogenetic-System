#include "hip/hip_runtime.h"
#include "header.cuh"
#include <chrono>//��׼ģ�������ʱ���йص�ͷ�ļ�
using namespace chrono;
//��ӡ�豸��Ϣ
void printDeviceProp(const hipDeviceProp_t &prop)
{
	printf("GPU Parament��\n");
	printf(" Device Name : %s.\n", prop.name);
	printf(" totalGlobalMem : %I64d.\n", prop.totalGlobalMem);
	printf(" sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf(" regsPerBlock : %d.\n", prop.regsPerBlock);
	printf(" warpSize : %d.\n", prop.warpSize);
	printf(" memPitch : %d.\n", prop.memPitch);
	printf(" maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf(" maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf(" maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf(" totalConstMem : %d.\n", prop.totalConstMem);
	printf(" major.minor : %d.%d.\n", prop.major, prop.minor);
	printf(" clockRate : %d.\n", prop.clockRate);
	printf(" textureAlignment : %d.\n", prop.textureAlignment);
	printf(" deviceOverlap : %d.\n", prop.deviceOverlap);
	printf(" multiProcessorCount : %d.\n", prop.multiProcessorCount);
	std::printf(" CUDA core: %d\r\n", 2 * prop.multiProcessorCount* prop.maxThreadsPerMultiProcessor / prop.warpSize);
	printf("\n\n");
}
//CUDA ��ʼ��
bool InitCUDA()
{
	int count;
	//ȡ��֧��Cuda��װ�õ���Ŀ
	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		//��ӡ�豸��Ϣ
		printDeviceProp(prop);
		//����Կ���ʱ��Ƶ��
		clockRate = prop.clockRate;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}
	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}
void check(hipError_t res, string warningstring)
{
	if (res != hipSuccess)
	{
		printf((warningstring + " !\n").c_str());
		system("pause");
		exit(0);
	}
}
//�鿴GPU�����Ƿ���ȷ
void checkGPUStatus(hipError_t hipGetLastError, string warningstring)
{
	if (hipGetLastError != hipSuccess)
	{
		printf("\n\n");
		printf((warningstring + " !\n").c_str());
		fprintf(stderr, "%s\n", hipGetErrorString(hipGetLastError));
		system("pause");
		exit(0);
	}
}

__global__ void Zhuan_Complex_kernel(float *PSF_1_gpu, hipfftComplex *PSF_1_gpu_Complex, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		PSF_1_gpu_Complex[i].x = PSF_1_gpu[i];
		PSF_1_gpu_Complex[i].y = 0;
	}
}
__global__ void PSF_unshort(float *PSF_1_gpu, unsigned short *PSF, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		if (PSF_1_gpu[i] < 0)
		{
			PSF[i] = 0;
		}
		else if (PSF_1_gpu[i] > 65535)
		{
			PSF[i] = 65535;
		}
		else
		{
			PSF[i] = (int)(PSF_1_gpu[i] + 0.5);
		}
	}
}
__global__ void initial_kernel_1(float *ImgEst, float *Ratio, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		ImgEst[i] = 0;
		Ratio[i] = 1;
	}
}
__global__ void gpuObjRecon_fuzhi(float *gpuObjRecon, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		gpuObjRecon[i] = 1;
	}
}
__global__ void initial_kernel_3(float *gpuObjRecROI, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		gpuObjRecROI[i] = 1;
	}
}
__global__ void ImgExp_ge(unsigned short *Img_gpu, int BkgMean, float *ImgExp, int total)
{
	//�Ѳ�Ľ��С��0�ı��0������0���������룬����65535�ı��65535
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		if ((Img_gpu[i] - BkgMean) < 0)
		{
			ImgExp[i] = 0;
		}
		else if ((Img_gpu[i] - BkgMean) > 65535)
		{
			ImgExp[i] = 65535;
		}
		else
		{
			ImgExp[i] = (int)((Img_gpu[i] - BkgMean) + 0.5);
		}
	}
}
__global__ void Ratio_fuzhi(float *Ratio, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		Ratio[i] = 1;
	}
}
__global__ void OTF_mul_gpuObjRecon_Complex(hipfftComplex *OTF, hipfftComplex *gpuObjRecon_Complex, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		float aaa = OTF[i].x*gpuObjRecon_Complex[i].x - OTF[i].y*gpuObjRecon_Complex[i].y;//��˵�ʵ�����
		float bbb = OTF[i].x*gpuObjRecon_Complex[i].y + OTF[i].y*gpuObjRecon_Complex[i].x;//��˵��鲿���
		gpuObjRecon_Complex[i].x = aaa;
		gpuObjRecon_Complex[i].y = bbb;
	}
}
__global__ void ifftshift_real_max(hipfftComplex *OTF, float *float_temp, int PSF_size_1, int PSF_size_2, int PSF_size_3)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	const int k = blockDim.z * blockIdx.z + threadIdx.z;
	int lie_half = PSF_size_2 / 2;
	if (i < PSF_size_1 / 2 && j < PSF_size_2 && k < PSF_size_3)
	{
		//ʵ��ͼ���ifftshift+real+max��������ͼ�񻮷ֳ�4�����ޣ���һ�͵���ƽ�ƽ������ڶ��͵���ƽ�ƽ�����
		float_temp[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * 512] = OTF[k*PSF_size_1*PSF_size_2 + i*PSF_size_2 + j].x >= 0 ? OTF[k*PSF_size_1*PSF_size_2 + i*PSF_size_2 + j].x : 0;
		float_temp[k*PSF_size_1*PSF_size_2 + i*PSF_size_2 + j] = OTF[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * PSF_size_2].x >= 0 ? OTF[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * PSF_size_2].x : 0;
	}
}
__global__ void ifftshift(hipfftComplex *OTF, float *float_temp, int PSF_size_1, int PSF_size_2, int PSF_size_3, hipfftComplex *OTF_ifftshift)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	const int k = blockDim.z * blockIdx.z + threadIdx.z;
	int lie_half = PSF_size_2 / 2;
	if (i < PSF_size_1 / 2 && j < PSF_size_2 && k < PSF_size_3)
	{
		//ʵ��ͼ���ifftshift��������ͼ�񻮷ֳ�4�����ޣ���һ�͵���ƽ�ƽ������ڶ��͵���ƽ�ƽ���
		OTF_ifftshift[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * 512] = OTF[k*PSF_size_1*PSF_size_2 + i*PSF_size_2 + j];
		OTF_ifftshift[k*PSF_size_1*PSF_size_2 + i*PSF_size_2 + j] = OTF[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * PSF_size_2];
	}
}
__global__ void float_temp_sum(float *float_temp, float *ImgEst, int PSF_size_1, int PSF_size_2, int PSF_size_3)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < PSF_size_1 && j < PSF_size_2)
	{
		ImgEst[i*PSF_size_2 + j] = 0;
		for (int k = 0; k < PSF_size_3; k++)
		{
			ImgEst[i*PSF_size_2 + j] += float_temp[k*PSF_size_1*PSF_size_2 + (i*PSF_size_2 + j)];
		}
	}
}
__global__ void Ratio_fuzhi_2(float *ImgExp, float *ImgEst, float Tmp, int SNR, float *Ratio, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		Ratio[i] = ImgExp[i]/(ImgEst[i] + Tmp / SNR);
	}
}
__global__ void Ratio_Complex_ge(float *ImgExp, float *ImgEst, float Tmp, int SNR, hipfftComplex *Ratio_Complex, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		Ratio_Complex[i].x = ImgExp[i] / (ImgEst[i] + Tmp / SNR);
		Ratio_Complex[i].y = 0;
	}
}
__global__ void fftRatio_ge(hipfftComplex *Ratio_Complex, hipfftComplex *fftRatio, int PSF_size_1, int PSF_size_2, int PSF_size_3)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	const int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i < PSF_size_1 && j < PSF_size_2 && k < PSF_size_3)
	{
		fftRatio[k*PSF_size_1*PSF_size_2 + i*PSF_size_2 + j] = Ratio_Complex[i*PSF_size_2 + j];
	}
}
__global__ void fftceshi_gpu_fuzhi(hipfftComplex *PSF_1_gpu_Complex, hipfftComplex *fftceshi_gpu, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		fftceshi_gpu[i] = PSF_1_gpu_Complex[i];
	}
}
__global__ void ifft2_divide(hipfftComplex *OTF, int total, int scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		OTF[i].x = OTF[i].x / scale;
		OTF[i].y = OTF[i].y / scale;
	}
}
__global__ void real_multiply(float *gpuObjRecon, float *float_temp, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		gpuObjRecon[i] = gpuObjRecon[i] * float_temp[i];
	}
}
__global__ void fftRatio_mul_conjOTF(hipfftComplex *fftRatio, hipfftComplex *OTF, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		float aaa = fftRatio[i].x*OTF[i].x + fftRatio[i].y*OTF[i].y;//������˵�ʵ�����
		float bbb = -fftRatio[i].x*OTF[i].y + fftRatio[i].y*OTF[i].x;//������˵��鲿���
		fftRatio[i].x = aaa;
		fftRatio[i].y = bbb;
	}
}


int main()
{
	//int geshu = 32;
	//hipfftComplex *aa = new hipfftComplex[geshu];
	//for (int i = 0; i < geshu; i++)
	//{
	//	aa[i].x = i + 1; aa[i].x += aa[i].x / 10;
	//	aa[i].y = 0;
	//}
	//hipfftComplex *aa_gpu;
	//check(hipMalloc((void**)&aa_gpu, geshu * sizeof(hipfftComplex)), "aa_gpu hipMalloc Error");
	//check(hipMemcpy(aa_gpu, aa, geshu * sizeof(hipfftComplex), hipMemcpyHostToDevice), "aa_gpu hipMemcpy Error");
	//hipfftComplex *aa_gpu_fft2;
	//check(hipMalloc((void**)&aa_gpu_fft2, geshu * sizeof(hipfftComplex)), "aa_gpu_fft2 hipMalloc Error");
	//hipfftHandle plan_aa;
	//hipfftResult res1 = hipfftPlan2d(&plan_aa, 4, 8, HIPFFT_C2C);
	//res1 = hipfftExecC2C(plan_aa, aa_gpu, aa_gpu_fft2, HIPFFT_FORWARD);
	//hipDeviceSynchronize();
	//checkGPUStatus(hipGetLastError(), "aa_gpu hipfftExecC2C Error");
	//check(hipMemcpy(aa, aa_gpu_fft2, geshu * sizeof(hipfftComplex), hipMemcpyDeviceToHost), "aa_gpu hipMemcpy Error");
	//float qq = 0; float ww = 0;
	//for (int i = 0; i < geshu; i++)
	//{
	//	qq += aa[i].x; ww += aa[i].y;
	//}

	//��ʼ��ʱ
	auto time_start = system_clock::now();

	const char *PSF_1_file = "F:/matlab-cuda-20220125/PSF_1_zhuanzhi_float.dat";//matlab�б��������float����
	const char *X31_file = "F:/matlab-cuda-20220125/r20210924_2_X31_resize.tif";
	const char *OutFile = "F:/matlab-cuda-20220125/result_C.dat";
	FILE *PSF_1_fid = fopen(PSF_1_file, "rb");
	if (PSF_1_fid == NULL)
	{
		cout << "PSF_1_file open failed!" << endl;
		system("pause");
		return 0;
	}
	float *PSF_1 = new float[PSF_size_1*PSF_size_2*PSF_size_3]();
	fread(PSF_1, sizeof(float), PSF_size_1*PSF_size_2*PSF_size_3, PSF_1_fid);
	//float ccc_sum = 0;
	//for (int i = 0; i < PSF_size_3; i++)
	//{
	//	ccc_sum = 0.0;
	//	for (int j = i*PSF_size_1*PSF_size_2; j < (i + 1)*PSF_size_1*PSF_size_2; j++)
	//	{
	//		ccc_sum += PSF_1[j];
	//	}
	//	cout << "��" << i+1 << "�����εĺ��ǣ�" << fixed << ccc_sum << endl;
	//}

	//ʹ��GDAL��ȡtif��ʹ�õ���matlab�ز����õ�����
	GDALAllRegister(); OGRRegisterAll();
	//����֧������·��
	CPLSetConfigOption("GDAL_FILENAME_IS_UTF8", "NO");
	CPLSetConfigOption("SHAPE_ENCODING", "");
	GDALDataset* poSrcDS = (GDALDataset*)GDALOpen(X31_file, GA_ReadOnly);    //��դ��ͼ��
	if (poSrcDS == NULL)
	{
		cout << "X31_file open failed!" << endl;
		return;
	}
	int wheight = poSrcDS->GetRasterYSize();//��
	int wwidth = poSrcDS->GetRasterXSize();//��
	int bandNum = poSrcDS->GetRasterCount();//������
	GDALDataType dataType = poSrcDS->GetRasterBand(1)->GetRasterDataType();//����
	unsigned short *Img = new unsigned short[PSF_size_1*PSF_size_2]();
	for (int i = 0; i < bandNum; i++)
	{
		//////////////////////////////////////��ȡ��ʼ�У�ʼ�У�������������ָ�룬��������������������
		poSrcDS->GetRasterBand(i + 1)->RasterIO(GF_Read, 0, 0, wwidth, wheight, Img, PSF_size_1, PSF_size_2, dataType, 0, 0);
	}
	GDALClose(poSrcDS);

	/*-------׼�������������ڴ桢�Դ棬fft���--------------------------*/
	const int rank = 2;//ά��
	int n[rank] = { PSF_size_1, PSF_size_2 };//n*m
	int *inembed = n;//���������size
	int istride = 1;//����������������Ϊ1
	int idist = n[0] * n[1];//1��������ڴ��С
	int *onembed = n;//�����һ�������size
	int ostride = 1;//ÿ��DFT������������Ϊ1
	int odist = n[0] * n[1];//�����һ��������ڶ�������ľ��룬�������������Ԫ�صľ���
	int batch = PSF_size_3;//�������������
	hipfftHandle fftplanfwd;//�������
	hipfftPlanMany(&fftplanfwd, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);

	float *PSF_1_gpu;
	check(hipMalloc((void**)&PSF_1_gpu, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float)), "PSF_1_gpu hipMalloc Error");
	hipfftComplex *PSF_1_gpu_Complex;
	check(hipMalloc((void**)&PSF_1_gpu_Complex, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "PSF_1_gpu_Complex hipMalloc Error");
	hipfftComplex *OTF;
	check(hipMalloc((void**)&OTF, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "OTF hipMalloc Error");
	float *ImgEst;
	check(hipMalloc((void**)&ImgEst, PSF_size_1*PSF_size_2 * sizeof(float)), "ImgEst hipMalloc Error");
	float *Ratio;
	check(hipMalloc((void**)&Ratio, PSF_size_1*PSF_size_2 * sizeof(float)), "Ratio hipMalloc Error");
	float *gpuObjRecon;
	check(hipMalloc((void**)&gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float)), "Ratio hipMalloc Error");
	float *gpuObjRecROI;
	check(hipMalloc((void**)&gpuObjRecROI, ROISize * 2 * ROISize * 2 * PSF_size_3 * sizeof(float)), "gpuObjRecROI hipMalloc Error");
	unsigned short *Img_gpu;
	check(hipMalloc((void**)&Img_gpu, PSF_size_1*PSF_size_2 * sizeof(unsigned short)), "Img_gpu hipMalloc Error");
	float *ImgExp;
	check(hipMalloc((void**)&ImgExp, PSF_size_1*PSF_size_2 * sizeof(float)), "ImgExp hipMalloc Error");
	hipfftComplex *gpuObjRecon_Complex;
	check(hipMalloc((void**)&gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "gpuObjRecon_Complex hipMalloc Error");
	float *float_temp;
	check(hipMalloc((void**)&float_temp, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float)), "float_temp hipMalloc Error");
	hipfftComplex *Ratio_Complex;
	check(hipMalloc((void**)&Ratio_Complex, PSF_size_1*PSF_size_2 * sizeof(hipfftComplex)), "Ratio_Complex hipMalloc Error");
	hipfftComplex *fftRatio;
	check(hipMalloc((void**)&fftRatio, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "fftRatio hipMalloc Error");




	/*---��ʼ��������------------------------------------------------------------------------*/
	auto time_1 = system_clock::now();
	/*-----��PSF_1��fft2-------*/
	check(hipMemcpy(PSF_1_gpu, PSF_1, PSF_size_1*PSF_size_2*PSF_size_3*sizeof(float), hipMemcpyHostToDevice), "PSF_1_gpu hipMemcpy Error");
	//ת���ɸ������鲿��0
	Zhuan_Complex_kernel << <blockNum_123, threadNum_123 >> > (PSF_1_gpu, PSF_1_gpu_Complex, PSF_size_1*PSF_size_2*PSF_size_3);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "PSF_1_gpu Zhuan_Complex_kernel Error");
	///////----����һ�����ε�fft2�Ƿ���ȷ-----------
	//hipfftComplex *fftceshi_gpu;
	//check(hipMalloc((void**)&fftceshi_gpu, PSF_size_1*PSF_size_2*sizeof(hipfftComplex)), "fftceshi_gpu hipMalloc Error");
	//hipfftComplex *fftceshi_gpu_shuchu;
	//check(hipMalloc((void**)&fftceshi_gpu_shuchu, PSF_size_1*PSF_size_2 * sizeof(hipfftComplex)), "fftceshi_gpu_shuchu hipMalloc Error");
	////ת�ɸ������鲿��0
	//fftceshi_gpu_fuzhi << <(PSF_size_1*PSF_size_2 - 1) / 256 + 1, 256 >> > (PSF_1_gpu_Complex, fftceshi_gpu, PSF_size_1*PSF_size_2);
	//hipDeviceSynchronize();
	//checkGPUStatus(hipGetLastError(), "fftceshi_gpu Error");
	//hipfftHandle plan;
	//hipfftResult res = hipfftPlan2d(&plan, PSF_size_1, PSF_size_2, HIPFFT_C2C);
	//res = hipfftExecC2C(plan, fftceshi_gpu, fftceshi_gpu_shuchu, HIPFFT_FORWARD);
	//if (res != HIPFFT_SUCCESS)
	//{
	//	cout << "fftceshi_gpu hipfftExecC2C error:" << res << endl;
	//	system("pause");
	//	return;
	//}
	//
	////*----�������ڴ�鿴�Ƿ���ȷ-------*/
	//hipfftComplex *abc = new hipfftComplex[PSF_size_1*PSF_size_2];
	//check(hipMemcpy(abc, fftceshi_gpu_shuchu, PSF_size_1*PSF_size_2*sizeof(hipfftComplex), hipMemcpyDeviceToHost), "abc hipMemcpy Error");
	//float *abc_real = new float[PSF_size_1*PSF_size_2];
	//float *abc_imag = new float[PSF_size_1*PSF_size_2];
	//float abc_real_sum = 0; float abc_imag_sum = 0;
	//for (int j = 0; j < PSF_size_1*PSF_size_2; j++)
	//{
	//	abc_real[j] = abc[j].x; abc_real_sum += abs(abc_real[j]);
	//	abc_imag[j] = abc[j].y; abc_imag_sum += abs(abc_imag[j]);
	//}
	////��ÿһ�е�ʵ�����鲿��
	//float *abc_real_lineSUM = new float[PSF_size_1]();
	//float *abc_imag_lineSUM = new float[PSF_size_1]();
	//for (int i = 0; i < PSF_size_1; i++)
	//{
	//	for (int j = 0; j < PSF_size_2; j++)
	//	{
	//		abc_real_lineSUM[i] += abs(abc_real[i*PSF_size_2 + j]);
	//		abc_imag_lineSUM[i] += abs(abc_imag[i*PSF_size_2 + j]);
	//	}
	//}

	////*----ʹ��hipfftPlanMany�ķ�������������άfft---------------------*/
	hipfftExecC2C(fftplanfwd, PSF_1_gpu_Complex, OTF, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "PSF_1_gpu_Complex hipfftExecC2C Error");
	//////����������ڴ棬�鿴�Ƿ��matlabһ��
	//hipfftComplex *OTF_cpu = new hipfftComplex[PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)];
	//check(hipMemcpy(OTF_cpu, OTF, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex), hipMemcpyDeviceToHost), "OTF_cpu hipMemcpy Error");
	//float aaa_sum = 0; float bbb_sum = 0;
	//for (int i = 0; i < PSF_size_3; i++)
	//{
	//	aaa_sum = 0; bbb_sum = 0;
	//	for (int j = i*PSF_size_1*PSF_size_2; j < (i + 1)*PSF_size_1*PSF_size_2; j++)
	//	{
	//		aaa_sum += abs(OTF_cpu[j].x);
	//		bbb_sum += abs(OTF_cpu[j].y);
	//	}
	//	cout << "��" << i + 1 << "�����ε�ʵ������ֵ�ͣ�" << fixed << aaa_sum << "�鲿����ֵ�ͣ�" << fixed << bbb_sum <<endl;
	//}

	/////PSF_1ת��uint16���ͣ��浽����PSF_gpu��
	//unsigned short *PSF_gpu;
	//check(hipMalloc((void**)&PSF_gpu, sizeof(unsigned short) * PSF_size_1*PSF_size_2*PSF_size_3), "PSF_gpu hipMalloc Error");
	//PSF_unshort << <blockNum_123, threadNum_123 >> > (PSF_1_gpu, PSF_gpu, PSF_size_1*PSF_size_2*PSF_size_3);
	//hipDeviceSynchronize();
	//checkGPUStatus(hipGetLastError(), "PSF_unshort Error");
	//////----�������ڴ棬�鿴�Ƿ���ȷ
	//unsigned short *aaa = new unsigned short[PSF_size_1*PSF_size_2*PSF_size_3];
	//check(hipMemcpy(aaa, PSF_gpu, sizeof(unsigned short) * PSF_size_1*PSF_size_2*PSF_size_3, hipMemcpyDeviceToHost), "aaa hipMemcpy Error");
	//float PSF_gpu_sum = 0;
	//for (int i = 0; i < PSF_size_1*PSF_size_2*PSF_size_3; i++)
	//{
	//	PSF_gpu_sum += aaa[i];
	//}

	////ImgEst��ֵΪ0��Ratio��ֵΪ1
	initial_kernel_1 << <blockNum_12, threadNum_12 >> > (ImgEst, Ratio, PSF_size_1*PSF_size_2);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "initial_kernel_1 Error");
	gpuObjRecon_fuzhi << <blockNum_123, threadNum_123 >> > (gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "gpuObjRecon_fuzhi Error");
	////gpuObjRecROI��ֵΪ1
	initial_kernel_3 << <blockNum_ROI, threadNum_ROI >> > (gpuObjRecROI, ROISize * 2 * ROISize * 2 * Nz);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "initial_kernel_3 Error");

	//tif���ݿ������Դ�
	check(hipMemcpy(Img_gpu, Img, PSF_size_1*PSF_size_2 * sizeof(unsigned short), hipMemcpyHostToDevice), "Img_gpu hipMemcpy Error");
	//��ȥ������ֵ���������float���͵�����ImgExp��
	ImgExp_ge << <blockNum_12, threadNum_12 >> > (Img_gpu, BkgMean, ImgExp, PSF_size_1*PSF_size_2);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "ImgExp_ge Error");

	//Ratio��gpuObjRecon��Ԫ�ض���ֵ1
	Ratio_fuzhi << <blockNum_12, threadNum_12 >> > (Ratio, PSF_size_1*PSF_size_2);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "Ratio_fuzhi Error");
	gpuObjRecon_fuzhi << <blockNum_123, threadNum_123 >> > (gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "gpuObjRecon_fuzhi Error");


	//��ʼѭ������
	for (int i = 0; i < ItN; i++)
	{
		////1��fft2(gpuObjRecon)
		Zhuan_Complex_kernel << <blockNum_123, threadNum_123 >> > (gpuObjRecon, gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "Zhuan_Complex_kernel Error");
		hipfftExecC2C(fftplanfwd, gpuObjRecon_Complex, gpuObjRecon_Complex, HIPFFT_FORWARD);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon_Complex hipfftExecC2C Error");
		//hipfftComplex *gpuObjRecon_Complex_cpu = new hipfftComplex[PSF_size_1*PSF_size_2*PSF_size_3];
		//check(hipMemcpy(gpuObjRecon_Complex_cpu, gpuObjRecon_Complex, sizeof(hipfftComplex) * PSF_size_1*PSF_size_2*PSF_size_3, hipMemcpyDeviceToHost), "gpuObjRecon_Complex_cpu hipMemcpy Error");
		//float aaa_sum = 0; float bbb_sum = 0;
		//float gpuObjRecon_Complex_cpu_real_sum = 0; float gpuObjRecon_Complex_cpu_imag_sum = 0;
		//for (int i = 0; i < PSF_size_3; i++)
		//{
		//	aaa_sum = 0; bbb_sum = 0;
		//	for (int j = i*PSF_size_1*PSF_size_2; j < (i + 1)*PSF_size_1*PSF_size_2; j++)
		//	{
		//		aaa_sum += abs(gpuObjRecon_Complex_cpu[j].x);
		//		bbb_sum += abs(gpuObjRecon_Complex_cpu[j].y);
		//		gpuObjRecon_Complex_cpu_real_sum += abs(gpuObjRecon_Complex_cpu[j].x);
		//		gpuObjRecon_Complex_cpu_imag_sum += abs(gpuObjRecon_Complex_cpu[j].y);
		//	}
		//	cout << "��" << i + 1 << "�����ε�ʵ���ͣ�" << fixed << aaa_sum << "�鲿�ͣ�" << fixed << bbb_sum << endl;
		//}
		//cout << "gpuObjRecon_Complex_cpuʵ���ͣ�" << fixed << gpuObjRecon_Complex_cpu_real_sum << "�鲿�ͣ�" << fixed << gpuObjRecon_Complex_cpu_imag_sum << endl;

		////2��OTF.*fft2(gpuObjRecon_Complex)���������gpuObjRecon_Complex��
		OTF_mul_gpuObjRecon_Complex << <blockNum_123, threadNum_123 >> > (OTF, gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "2��OTF.*fft2(gpuObjRecon_Complex) Error");
		//hipfftComplex *gpuObjRecon_Complex_cpu = new hipfftComplex[PSF_size_1*PSF_size_2*PSF_size_3];
		//check(hipMemcpy(gpuObjRecon_Complex_cpu, gpuObjRecon_Complex, sizeof(hipfftComplex) * PSF_size_1*PSF_size_2*PSF_size_3, hipMemcpyDeviceToHost), "gpuObjRecon_Complex_cpu hipMemcpy Error");
		//float aaa_sum = 0; float bbb_sum = 0;
		//float gpuObjRecon_Complex_cpu_real_sum = 0; float gpuObjRecon_Complex_cpu_imag_sum = 0;
		//for (int i = 0; i < PSF_size_3; i++)
		//{
		//	aaa_sum = 0; bbb_sum = 0;
		//	for (int j = i*PSF_size_1*PSF_size_2; j < (i + 1)*PSF_size_1*PSF_size_2; j++)
		//	{
		//		aaa_sum += abs(gpuObjRecon_Complex_cpu[j].x);
		//		bbb_sum += abs(gpuObjRecon_Complex_cpu[j].y);
		//		gpuObjRecon_Complex_cpu_real_sum += abs(gpuObjRecon_Complex_cpu[j].x);
		//		gpuObjRecon_Complex_cpu_imag_sum += abs(gpuObjRecon_Complex_cpu[j].y);
		//	}
		//	cout << "��" << i + 1 << "�����ε�ʵ���ͣ�" << fixed << aaa_sum << "�鲿�ͣ�" << fixed << bbb_sum << endl;
		//}
		//cout << "gpuObjRecon_Complex_cpuʵ���ͣ�" << fixed << gpuObjRecon_Complex_cpu_real_sum << "�鲿�ͣ�" << fixed << gpuObjRecon_Complex_cpu_imag_sum << endl;

		////3��ifft2(OTF.*fft2(gpuObjRecon))����任��Ҫ���������ظ���
		hipfftExecC2C(fftplanfwd, gpuObjRecon_Complex, gpuObjRecon_Complex, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon_Complex hipfftExecC2C cufft_inverse Error");
		////4������������������ȷ
		ifft2_divide << <blockNum_123, threadNum_123 >> > (gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3, PSF_size_1*PSF_size_2);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon_Complex ifft2_divide Error");
		//hipfftComplex *gpuObjRecon_Complex_cpu = new hipfftComplex[PSF_size_1*PSF_size_2*PSF_size_3];
		//check(hipMemcpy(gpuObjRecon_Complex_cpu, gpuObjRecon_Complex, sizeof(hipfftComplex) * PSF_size_1*PSF_size_2*PSF_size_3, hipMemcpyDeviceToHost), "gpuObjRecon_Complex_cpu hipMemcpy Error");
		//float aaa_sum = 0; float bbb_sum = 0;
		//float gpuObjRecon_Complex_cpu_real_sum = 0; float gpuObjRecon_Complex_cpu_imag_sum = 0;
		//for (int i = 0; i < PSF_size_3; i++)
		//{
		//	aaa_sum = 0; bbb_sum = 0;
		//	for (int j = i*PSF_size_1*PSF_size_2; j < (i + 1)*PSF_size_1*PSF_size_2; j++)
		//	{
		//		aaa_sum += abs(gpuObjRecon_Complex_cpu[j].x); bbb_sum += abs(gpuObjRecon_Complex_cpu[j].y);
		//		gpuObjRecon_Complex_cpu_real_sum += abs(gpuObjRecon_Complex_cpu[j].x);
		//		gpuObjRecon_Complex_cpu_imag_sum += abs(gpuObjRecon_Complex_cpu[j].y);
		//	}
		//	cout << "��" << i + 1 << "�����ε�ʵ���ͣ�" << fixed << aaa_sum << "�鲿�ͣ�" << fixed << bbb_sum << endl;
		//}
		//cout << "gpuObjRecon_Complex_cpuʵ���ͣ�" << fixed << gpuObjRecon_Complex_cpu_real_sum << "�鲿�ͣ�" << fixed << gpuObjRecon_Complex_cpu_imag_sum << endl;

		/*----�ڶ���gpuObjRecon_Complex��ʵ����ȷ���鲿����ȷ������Ĵ���ֻ����gpuObjRecon_Complex��ʵ����û�õ��鲿----------*/

		////5��ifftshift + real + max(,0)�����ʵ������float_temp��С��0�ĸ�ֵ0
		ifftshift_real_max << <grid, block >> > (gpuObjRecon_Complex, float_temp, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "5��gpuObjRecon_Complex ifftshift_real_max Error");
		//float *float_temp_cpu = new float[PSF_size_1*PSF_size_2*PSF_size_3];
		//check(hipMemcpy(float_temp_cpu, float_temp, sizeof(float) * PSF_size_1*PSF_size_2*PSF_size_3, hipMemcpyDeviceToHost), "float_temp_cpu hipMemcpy Error");
		//float aaa_sum = 0; float float_temp_cpu_sum = 0;
		//for (int i = 0; i < PSF_size_3; i++)
		//{
		//	aaa_sum = 0; 
		//	for (int j = i*PSF_size_1*PSF_size_2; j < (i + 1)*PSF_size_1*PSF_size_2; j++)
		//	{
		//		aaa_sum += abs(float_temp_cpu[j]);
		//		float_temp_cpu_sum += abs(float_temp_cpu[j]);
		//	}
		//	cout << "��" << i + 1 << "�����ξ���ֵ�ͣ�" << fixed << aaa_sum <<endl;
		//}
		//cout << "float_temp_cpu����ֵ�ͣ�" << fixed << float_temp_cpu_sum << endl;

		////6��sum( ,3)���ڵ���ά�ϼ���ͣ�����PSF_size_1��PSF_size_2�еľ���ImgEst
		float_temp_sum << <grid_sum, block_sum >> > (float_temp, ImgEst, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "float_temp_sum Error");
		
		////7��Tmp=mean(   ImgEst(:)   );
		thrust::device_ptr<float> dev_ptr(ImgEst);
		float Tmp = thrust::reduce(dev_ptr, dev_ptr + size_t(PSF_size_1*PSF_size_2), (float)0, thrust::plus<float>()) / (PSF_size_1*PSF_size_2);
		/**********************************************************************************************************/
		/*----������ȷ��Tmp����matlab����47424472��C������47424477.675621979�����ǳ��ǳ�С��Ӧ�ÿ��Ժ���
		�ڶ���matlab��51785136��C������51785130.147748277�����Ҳ�ǳ�С�����Ժ���----*/
		/**********************************************************************************************************/

		////8��Ratio(1:end,1:end)=ImgExp(1:end,1:end)./(ImgEst(1:end,1:end)+Tmp/SNR)����ת�ɸ��������鲿Ϊ��;
		Ratio_Complex_ge << <blockNum_12, threadNum_12 >> > (ImgExp, ImgEst, Tmp, SNR, Ratio_Complex, PSF_size_1*PSF_size_2);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "Ratio_Complex_ge Error");
		//hipfftComplex *Ratio_Complex_cpu = new hipfftComplex[PSF_size_1*PSF_size_2]();
		//check(hipMemcpy(Ratio_Complex_cpu, Ratio_Complex, sizeof(hipfftComplex) * PSF_size_1*PSF_size_2, hipMemcpyDeviceToHost), "Ratio_Complex_cpu hipMemcpy Error");
		//float Ratio_Complex_cpu_real_sum = 0; float Ratio_Complex_cpu_imag_sum = 0;
		//for (int j = 0; j < PSF_size_1*PSF_size_2; j++)
		//{
		//	Ratio_Complex_cpu_real_sum += (Ratio_Complex_cpu[j].x);
		//	Ratio_Complex_cpu_imag_sum += (Ratio_Complex_cpu[j].y);
		//}
		//cout << "ʵ���ͣ�" << fixed << Ratio_Complex_cpu_real_sum << " �鲿�ͣ�" << fixed << Ratio_Complex_cpu_imag_sum << endl;

		/*******************************************************************************************/
		/*----������ȷ������ĺͣ�matlab��0.3017935��C������0.301793���ڶ�����ȷ-------------------*/
		/*******************************************************************************************/

		////9��fft2(Ratio)
		res = hipfftExecC2C(plan, Ratio_Complex, Ratio_Complex, HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS)
		{
			cout << "Ratio_Complex hipfftExecC2C error:" << res << endl;
			system("pause");
			return;
		}
		//hipfftComplex *Ratio_Complex_cpu = new hipfftComplex[PSF_size_1*PSF_size_2];
		//check(hipMemcpy(Ratio_Complex_cpu, Ratio_Complex, sizeof(hipfftComplex) * PSF_size_1*PSF_size_2, hipMemcpyDeviceToHost), "Ratio_Complex_cpu hipMemcpy Error");
		////��ÿһ�е�ʵ�����鲿�͡����еĺ�
		//float Ratio_Complex_cpu_realSUM = 0; float Ratio_Complex_cpu_imagSUM = 0;
		//float *abc_real_lineSUM = new float[PSF_size_1]();
		//float *abc_imag_lineSUM = new float[PSF_size_1]();
		//for (int i = 0; i < PSF_size_1; i++)
		//{
		//	for (int j = 0; j < PSF_size_2; j++)
		//	{
		//		abc_real_lineSUM[i] += Ratio_Complex_cpu[i*PSF_size_2 + j].x;
		//		abc_imag_lineSUM[i] += Ratio_Complex_cpu[i*PSF_size_2 + j].y;
		//		Ratio_Complex_cpu_realSUM += abs(Ratio_Complex_cpu[i*PSF_size_2 + j].x);
		//		Ratio_Complex_cpu_imagSUM += abs(Ratio_Complex_cpu[i*PSF_size_2 + j].y);
		//	}
		//}
		//cout << "ʵ���ͣ�" << fixed << Ratio_Complex_cpu_realSUM << " �鲿�ͣ�" << fixed << Ratio_Complex_cpu_imagSUM << endl;

		/*******************************************************************************************/
		/*----������ȷ������ĺͺ�matlab��һ����������С�������λ��̫���ˣ�����ֵ�ĺ���һ����-------*/
		/*******************************************************************************************/

		////10��repmat����ֵNz�飬Ratio_Complex�����ά��fftRatio
		fftRatio_ge << <grid, block >> > (Ratio_Complex, fftRatio, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio_ge Error");
		//hipfftComplex *fftRatio_cpu = new hipfftComplex[PSF_size_1*PSF_size_2*PSF_size_3];
		//check(hipMemcpy(fftRatio_cpu, fftRatio, sizeof(hipfftComplex) * PSF_size_1*PSF_size_2*PSF_size_3, hipMemcpyDeviceToHost), "fftRatio_cpu hipMemcpy Error");
		//float aaa_sum = 0; float bbb_sum = 0;
		//for (int i = 0; i < PSF_size_3; i++)
		//{
		//	aaa_sum = 0; bbb_sum = 0;
		//	for (int j = i*PSF_size_1*PSF_size_2; j < (i + 1)*PSF_size_1*PSF_size_2; j++)
		//	{
		//		aaa_sum += abs(fftRatio_cpu[j].x);
		//		bbb_sum += abs(fftRatio_cpu[j].y);
		//	}
		//	cout << "��" << i + 1 << "�����ε�ʵ������ֵ�ͣ�" << fixed << aaa_sum << "�鲿����ֵ�ͣ�" << fixed << bbb_sum <<endl;
		//}

		////11��fftRatio.*conj(OTF)���浽fftRatio��
		fftRatio_mul_conjOTF << <blockNum_123, threadNum_123 >> > (fftRatio, OTF, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio_mul_conjOTF Error");
		//hipfftComplex *fftRatio_cpu = new hipfftComplex[PSF_size_1*PSF_size_2*PSF_size_3]();
		//check(hipMemcpy(fftRatio_cpu, fftRatio, sizeof(hipfftComplex)*PSF_size_1*PSF_size_2*PSF_size_3, hipMemcpyDeviceToHost), "fftRatio_cpu hipMemcpy Error");
		//float fftRatio_cpu_real_sum = 0; float fftRatio_cpu_imag_sum = 0;
		//float aaa_sum = 0; float bbb_sum = 0;
		//for (int i = 0; i < PSF_size_3; i++)
		//{
		//	aaa_sum = 0; bbb_sum = 0;
		//	for (int j = i*PSF_size_1*PSF_size_2; j < (i + 1)*PSF_size_1*PSF_size_2; j++)
		//	{
		//		aaa_sum += abs(fftRatio_cpu[j].x);
		//		bbb_sum += abs(fftRatio_cpu[j].y);
		//		fftRatio_cpu_real_sum += abs(fftRatio_cpu[j].x);
		//		fftRatio_cpu_imag_sum += abs(fftRatio_cpu[j].y);
		//	}
		//	cout << "��" << i + 1 << "�����ε�ʵ������ֵ�ͣ�" << fixed << aaa_sum << "�鲿����ֵ�ͣ�" << fixed << bbb_sum <<endl;
		//}
		//cout << "ʵ���ͣ�" << fixed << fftRatio_cpu_real_sum << " �鲿�ͣ�" << fixed << fftRatio_cpu_imag_sum << endl;


		////12��ifft2(       fftRatio.*conj(OTF)       )�������������ظ���
		hipfftExecC2C(fftplanfwd, fftRatio, fftRatio, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio hipfftExecC2C Error");
		ifft2_divide << <blockNum_123, threadNum_123 >> > (fftRatio, PSF_size_1*PSF_size_2*PSF_size_3, PSF_size_1*PSF_size_2);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio ifft2_divide Error");
		//hipfftComplex *fftRatio_cpu = new hipfftComplex[PSF_size_1*PSF_size_2*PSF_size_3]();
		//check(hipMemcpy(fftRatio_cpu, fftRatio, sizeof(hipfftComplex)*PSF_size_1*PSF_size_2*PSF_size_3, hipMemcpyDeviceToHost), "fftRatio_cpu hipMemcpy Error");
		//float fftRatio_cpu_real_sum = 0; float fftRatio_cpu_imag_sum = 0;
		//float aaa_sum = 0; float bbb_sum = 0;
		//for (int i = 0; i < PSF_size_3; i++)
		//{
		//	aaa_sum = 0; bbb_sum = 0;
		//	for (int j = i*PSF_size_1*PSF_size_2; j < (i + 1)*PSF_size_1*PSF_size_2; j++)
		//	{
		//		aaa_sum += abs(fftRatio_cpu[j].x);
		//		bbb_sum += abs(fftRatio_cpu[j].y);
		//		fftRatio_cpu_real_sum += abs(fftRatio_cpu[j].x);
		//		fftRatio_cpu_imag_sum += abs(fftRatio_cpu[j].y);
		//	}
		//	cout << "��" << i + 1 << "�����ε�ʵ������ֵ�ͣ�" << fixed << aaa_sum << "�鲿����ֵ�ͣ�" << fixed << bbb_sum <<endl;
		//}
		//cout << "ʵ���ͣ�" << fixed << fftRatio_cpu_real_sum << " �鲿�ͣ�" << fixed << fftRatio_cpu_imag_sum << endl;

		////13��max(   real(   ifftshift(   ifftshift(     1),   2)   ),   0);
		ifftshift_real_max << <grid, block >> > (fftRatio, float_temp, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "13��fftRatio ifftshift_real_max Error");
		////�鿴��ȷ��
		//float *float_temp_cpu = new float[PSF_size_1*PSF_size_2*PSF_size_3];
		//check(hipMemcpy(float_temp_cpu, float_temp, sizeof(float) * PSF_size_1*PSF_size_2*PSF_size_3, hipMemcpyDeviceToHost), "float_temp_cpu hipMemcpy Error");
		//float float_temp_cpu_sum = 0;
		//for (int i = 0; i < PSF_size_3; i++)
		//{
		//	float_temp_cpu_sum = 0;
		//	for (int j = i*PSF_size_1*PSF_size_2; j < (i + 1)*PSF_size_1*PSF_size_2; j++)
		//	{
		//		float_temp_cpu_sum += abs(float_temp_cpu[j]);
		//	}
		//	cout << "��" << i + 1 << "�����εľ���ֵ�ͣ�" << fixed << float_temp_cpu_sum << endl;
		//}

		////14��gpuObjRecon = gpuObjRecon.*max(  )
		real_multiply << <blockNum_123, threadNum_123 >> > (gpuObjRecon, float_temp, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon real_multiply Error");

		//cout << "��ɵ�" << i << "��ѭ��" << endl << endl << endl;
	}
	//������ϣ�ȡֵ����PSF_1
	check(hipMemcpy(PSF_1, gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3*sizeof(float), hipMemcpyDeviceToHost), "gpuObjRecon to PSF_1 hipMemcpy Error");
	//float PSF_1_sum = 0;
	//for (int i = 0; i < PSF_size_1*PSF_size_2*PSF_size_3; i++)
	//{
	//	PSF_1_sum += PSF_1[i];
	//}
	//cout << "gpuObjRecon�ͣ�" << fixed << PSF_1_sum << endl;

	////matlab���Ǵ�157��356�У��ܹ�356-127+1=200�С�157-356�У��ܹ�356-127+1=200�С�
	int line_start = Nxy / 2 - ROISize; int line_end = Nxy / 2 + ROISize - 1; int line_total = line_end - line_start + 1;
	int col_start = Nxy / 2 - ROISize; 	int col_end = Nxy / 2 + ROISize - 1; int col_total = col_end - col_start + 1;
	//float *ObjRecon = new float[line_total*col_total*PSF_size_3]();
	//float ObjRecon_sum = 0;
	//for (int band = 0; band < PSF_size_3; band++)
	//{
	//	for (int i = 0; i < line_total; i++)//��ѭ��
	//	{
	//		for (int j = 0; j < col_total; j++)//��ѭ��
	//		{
	//			ObjRecon[band*line_total*col_total + i*col_total + j] = PSF_1[band*PSF_size_1*PSF_size_2 + (i + line_start)*PSF_size_2 + j + col_start];
	//			ObjRecon_sum += abs(ObjRecon[band*line_total*col_total + i*col_total + j]);
	//		}
	//	}
	//}
	//cout << "ObjRecon�ͣ�" << fixed << ObjRecon_sum << endl;


	//���ͼ��
	GDALDriver * pDriver = GetGDALDriverManager()->GetDriverByName("ENVI");
	GDALDataset *ds = pDriver->Create(OutFile, col_total, line_total, PSF_size_3, GDT_Float32, NULL);
	if (ds == NULL)
	{
		cout << "Failed to create output file!" << endl;
		system("pause");
		return 0;
	}
	float *ObjRecon_buffer = new float[col_total];
	for (int band = 0; band < PSF_size_3; band++)
	{
		for (int i = 0; i < line_total; i++)//��ѭ��
		{
			for (int j = 0; j < col_total; j++)//��ѭ��
			{
				ObjRecon_buffer[j] = PSF_1[band*PSF_size_1*PSF_size_2 + (i + line_start)*PSF_size_2 + j + col_start];
			}
			ds->GetRasterBand(band + 1)->RasterIO(GF_Write, 0, i, col_total, 1, ObjRecon_buffer, col_total, 1, GDT_Float32, 0, 0);
		}
	}
	
	auto time_end = system_clock::now();
	auto duration1 = duration_cast<microseconds>(time_end - time_1);
	float usetime1 = float(duration1.count()) * microseconds::period::num / microseconds::period::den;
	auto duration2 = duration_cast<microseconds>(time_end - time_start);
	float usetime2 = float(duration2.count()) * microseconds::period::num / microseconds::period::den;
	cout << "��ȥ���ݶ�ȡ���ڴ��Դ����룬�������㲿����ʱ��" << usetime1 << "��" << endl;
	cout << "����ʱ��" << usetime2 << "��" << endl;
	system("pause");
    return 0;
}






//int main(void)
//{
//	using namespace std;
//#define CHANNEL_NUM  31 //ͨ������FFT����
//	const int dataH = 32; //ͼ��߶�
//	const int dataW = 8;  //ͼ����
//	hipfftHandle fftplanfwd;//�������
//	/* ���������ˡ��豸�˵��ڴ�ռ� */
//	hipfftComplex *h_Data = (hipfftComplex*)malloc(dataH*CHANNEL_NUM*dataW * sizeof(hipfftComplex));
//	hipfftComplex *d_Data;//device��ʾGPU�ڴ棬�洢��cpu������GPU������
//	hipfftComplex *fd_Data;//device��ʾGPU�ڴ�,R2C�����hipfftComplex��������
//	hipMalloc((void**)&d_Data, dataH*dataW*CHANNEL_NUM * sizeof(hipfftComplex));
//	hipMalloc((void**)&fd_Data, dataH*dataW*CHANNEL_NUM * sizeof(hipfftComplex));
//	//�����ʼ����������
//	for (int i = 0; i < dataH*CHANNEL_NUM; i++)
//	{
//		for (int j = 0; j < dataW; j++)
//		{
//			h_Data[i*dataW + j].x = float(rand() % 255);
//			h_Data[i*dataW + j].y = float(rand() % 255);
//		}
//	}
//	const int rank = 2;//ά��
//	int n[rank] = { 32, 8 };//n*m
//	int *inembed = n;//���������size
//	int istride = 1;//����������������Ϊ1
//	int idist = n[0] * n[1];//1��������ڴ��С
//	int *onembed = n;//�����һ�������size
//	int ostride = 1;//ÿ��DFT������������Ϊ1
//	int odist = n[0] * n[1];//�����һ��������ڶ�������ľ��룬�������������Ԫ�صľ���
//	int batch = CHANNEL_NUM;//�������������
//	hipfftPlanMany(&fftplanfwd, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);//��Զ��ź�ͬʱ����FFT
//	hipMemcpy(d_Data, h_Data, dataW * dataH*CHANNEL_NUM * sizeof(hipfftComplex), hipMemcpyHostToDevice);
//	hipfftExecC2C(fftplanfwd, d_Data, fd_Data, HIPFFT_FORWARD);
//	hipfftComplex *h_resultFFT = (hipfftComplex*)malloc(dataH*dataW*CHANNEL_NUM*sizeof(hipfftComplex));
//	hipMemcpy(h_resultFFT, fd_Data, dataW*dataH*CHANNEL_NUM * sizeof(hipfftComplex), hipMemcpyDeviceToHost);//��fft������ݿ���������
//}



