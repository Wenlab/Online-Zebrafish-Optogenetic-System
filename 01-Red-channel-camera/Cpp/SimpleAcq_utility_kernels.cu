#include "hip/hip_runtime.h"
﻿#include "SimpleAcq_utility_kernels.h"
#include <iostream>
#include <cmath>

//--- Very simple CUDA kernel to add constant value to input buffer
__global__ void ImgReverse_kernel(unsigned short * inputBuffer, int bufferWidth, int bufferHeight) 
{
  //--- compute idx, the x and y location of the element in the original array
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;

	unsigned short *pus_ptr1 = inputBuffer;

	if ( idx < bufferWidth && idy < bufferHeight) {
		int i_index = idx + idy*bufferWidth;	
		pus_ptr1[i_index] = (pus_ptr1[i_index] > 255) ? 0 : (255 - pus_ptr1[i_index]);
		/*pus_ptr1[i_index] = pow(pus_ptr1[i_index], 20) / pow(4096, 19);
		pus_ptr1[i_index] = pow(pus_ptr1[i_index], 3) / pow(4096, 2);
		pus_ptr1[i_index] = pow(pus_ptr1[i_index], 2) / pow(4096, 1);*/
		//pus_ptr1[i_index] = ((i_index/16) % 2) ? 100 : 255;
	}
}

//--- wrapper function for 'addConstant_kernel'
void ImgReverseOnGpuFunc(unsigned short * inputBuffer, unsigned short bufferWidth, unsigned short bufferHeight, 
													hipStream_t *stream) 
{
	//--- Set up CUDA kernel vars
	dim3 threadsPerBlock;
	dim3 blocksPerGrid;

	int devID;
	hipDeviceProp_t deviceProp;
	hipGetDevice(&devID);
	hipGetDeviceProperties(&deviceProp, devID);

	// Use a larger block size for Fermi and above
	int block_size = (deviceProp.major < 2) ? 16 : 32;
  
	threadsPerBlock.x = block_size;
	threadsPerBlock.y = block_size;

	blocksPerGrid.x = bufferWidth / threadsPerBlock.x + (bufferWidth % threadsPerBlock.x == 0 ? 0:1);
	blocksPerGrid.y = bufferHeight / threadsPerBlock.y + (bufferHeight % threadsPerBlock.y == 0 ? 0:1);

	//--- call CUDA Kernel
	ImgReverse_kernel<<<blocksPerGrid, threadsPerBlock, 0, *stream>>>(inputBuffer, bufferWidth, bufferHeight);

#ifdef _DEBUG
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();																										
	if( hipSuccess != err)																																		
	{
		printf("kernel error, err is %s\n", hipGetErrorString(err));
	}
#endif

};
