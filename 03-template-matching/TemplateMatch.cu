#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <tchar.h>
#include <io.h>
#include <string>
#include <string.h>
#include <assert.h>
#include <math.h>
#include <windows.h>
#include "hip/hip_runtime.h"
#include ""
#include <FreeImage.h>
#include <npp.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h> 
#include <thrust/sort.h> 
#include <thrust/copy.h> 
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/sequence.h>
#include <thrust/count.h>
#include <chrono>//��׼ģ�������ʱ���йص�ͷ�ļ�
#include "gdal_alg.h";
#include "gdal_priv.h"
#include <gdal.h>
#include "gdal_mdreader.h"
#include "gdalwarper.h"
#include "ogrsf_frmts.h"

using namespace std;
using namespace chrono;
//ͼ�񼸺α任C++ʵ��--����ƽ�ƣ���ת�����У�����
//https://blog.csdn.net/duiwangxiaomi/article/details/109532590


int clockRate = 1.0;
//��ӡ�豸��Ϣ
void printDeviceProp(const hipDeviceProp_t &prop)
{
	printf("GPU Parament��\n");
	printf(" Device Name : %s.\n", prop.name);
	printf(" totalGlobalMem : %I64d.\n", prop.totalGlobalMem);
	printf(" sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf(" regsPerBlock : %d.\n", prop.regsPerBlock);
	printf(" warpSize : %d.\n", prop.warpSize);
	printf(" memPitch : %d.\n", prop.memPitch);
	printf(" maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf(" maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf(" maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf(" totalConstMem : %d.\n", prop.totalConstMem);
	printf(" major.minor : %d.%d.\n", prop.major, prop.minor);
	printf(" clockRate : %d.\n", prop.clockRate);
	printf(" textureAlignment : %d.\n", prop.textureAlignment);
	printf(" deviceOverlap : %d.\n", prop.deviceOverlap);
	printf(" multiProcessorCount : %d.\n", prop.multiProcessorCount);
	std::printf(" CUDA core: %d\r\n", 2 * prop.multiProcessorCount* prop.maxThreadsPerMultiProcessor / prop.warpSize);
	printf("\n\n");
}
//CUDA ��ʼ��
bool InitCUDA()
{
	int count;
	//ȡ��֧��Cuda��װ�õ���Ŀ
	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		//��ӡ�豸��Ϣ
		printDeviceProp(prop);
		//����Կ���ʱ��Ƶ��
		clockRate = prop.clockRate;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}
	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}
void check(hipError_t res, string warningstring)
{
	if (res != hipSuccess)
	{
		printf((warningstring + " !\n").c_str());
		system("pause");
		exit(0);
	}
}
void check1(hipError_t res, string warningstring, const char *file, int linenum)
{
	//�÷�check1(hipMalloc((void**)&aa, 10 * sizeof(float)), "aa hipMalloc Error", __FILE__, __LINE__);
	if (res != hipSuccess)
	{
		printf((warningstring + " !\n").c_str());
		printf("   Error text: %s   Error code: %d\n", hipGetErrorString(res), res);
		printf("   Line:    %d    File:    %s\n", linenum, file);
		system("pause");
		exit(0);
	}
}
//�鿴GPU�����Ƿ���ȷ
void checkGPUStatus(hipError_t hipGetLastError, string warningstring)
{
	if (hipGetLastError != hipSuccess)
	{
		printf("\n\n");
		printf((warningstring + " !\n").c_str());
		fprintf(stderr, "%s\n", hipGetErrorString(hipGetLastError));
		system("pause");
		exit(0);
	}
}

//npp��ͼ����ת����nppiRotate_32f_C1Rʹ�÷�����
//����ת�Ƕ�Ϊ����ͼ������½������ȿ�ʼ���ģ�˳ʱ����ת�����Ͻǿ�ʼ���ģ���ʱ����ת
//����ת�Ƕ�Ϊ����ͼ������½������ȿ�ʼ���ģ���ʱ����ת�����Ͻǿ�ʼ���ģ�˳ʱ����ת
float *ObjRecon_imrotate3(float *ObjRecon, double nAngle)
{
	//float *input_image = new float[200*200];
	float *imageRotated3D = new float[200 * 200 * 50];
	//for (int i = 0; i < 200 * 200; i++)
	//{
	//	input_image[i] = ObjRecon[i];
	//}

	NppiSize Input_Size;//����ͼ���������
	Input_Size.width = 200;
	Input_Size.height = 200;
	/* �����Դ棬��ԭͼ�����Դ� */
	int nSrcPitchCUDA = Input_Size.width * sizeof(float);//ÿ����ռ���ֽ���
	float *input_image_gpu;
	check1(hipMalloc((void**)&input_image_gpu, sizeof(float)*Input_Size.width*Input_Size.height), "input_image_gpu hipMalloc Error", __FILE__, __LINE__);
	//check(hipMemcpy(input_image_gpu, input_image, sizeof(float)*Input_Size.width*Input_Size.height, hipMemcpyHostToDevice), "input_image_gpu hipMemcpy Error");


	/* ������ת�󳤿� */
	NppiRect Input_ROI;//�ض��������ת���൱�ڲü�ͼ���һ�飬���β���ȫ��ͼ��
	Input_ROI.x = Input_ROI.y = 0;
	Input_ROI.width = Input_Size.width;
	Input_ROI.height = Input_Size.height;
	double aBoundingBox[2][2];
	nppiGetRotateBound(Input_ROI, aBoundingBox, nAngle, 0, 0);
	int bb = ((int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0])) - Input_ROI.width) / 2 + aBoundingBox[0][0];//��ʼ��
	int cc = ((int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1])) - Input_ROI.height) / 2 + aBoundingBox[0][1];//��ʼ��
	aBoundingBox[0][0] = bb;//��ʼ��
	aBoundingBox[0][1] = cc;//��ʼ��
	NppiSize Output_Size;
	Output_Size.width = (int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0]));
	Output_Size.height = (int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1]));
	Output_Size.width = Input_Size.width;
	Output_Size.height = Input_Size.height;


	/* ת�����ͼ���Դ���� */
	int nDstPitchCUDA = Output_Size.width * sizeof(float);
	float *output_image_gpu;
	check1(hipMalloc((void**)&output_image_gpu, sizeof(float)*Output_Size.width*Output_Size.height), "output_image_gpu hipMalloc Error", __FILE__, __LINE__);


	//�������Ȥ���Ĵ�С���൱�ڰ����ͼ���ٲü�һ�飬Ӧ������������û���ԣ��������
	NppiRect Output_ROI;
	Output_ROI.x = 0; Output_ROI.y = 0;
	Output_ROI.width = Input_Size.width;
	Output_ROI.height = Input_Size.height;

	for (int i = 0; i < 50; i++)
	{
		check(hipMemcpy(input_image_gpu, ObjRecon + Input_Size.width*Input_Size.height * i, sizeof(float)*Input_Size.width*Input_Size.height, hipMemcpyHostToDevice), "input_image_gpu hipMemcpy Error");
		/* ������ת */
		NppStatus nppRet = nppiRotate_32f_C1R(input_image_gpu, Input_Size, nSrcPitchCUDA, Input_ROI,
			output_image_gpu, nDstPitchCUDA, Output_ROI, nAngle, -aBoundingBox[0][0], -aBoundingBox[0][1], NPPI_INTER_NN);
		assert(nppRet == NPP_NO_ERROR);
		check(hipMemcpy(imageRotated3D + Input_Size.width*Input_Size.height * i, output_image_gpu, sizeof(float) * Output_Size.width*Output_Size.height, hipMemcpyDeviceToHost), "output_image hipMemcpy Error");
	}

	////��תǰ��ĵ�һ�����ηֱ�д��������
	//float *ObjRecon_1 = new float[200 * 200];
	//float *imageRotated3D_1 = new float[200 * 200];
	//for (int i = 0; i < 200*200; i++)
	//{
	//	ObjRecon_1[i] = ObjRecon[i];
	//	imageRotated3D_1[i] = imageRotated3D[i];
	//}
	//GDALDriver * pDriver = GetGDALDriverManager()->GetDriverByName("GTiff");
	//GDALDataset *ds1 = pDriver->Create("ObjRecon_1_c", 200, 200, 1, GDT_Float32, NULL);
	//GDALDataset *ds2 = pDriver->Create("imageRotated3D_1_c", 200, 200, 1, GDT_Float32, NULL);
	//if ((ds1 == NULL) || (ds2 == NULL))
	//{
	//	cout << "create ObjRecon_1 imageRotated3D_1 output_file error!" << endl;
	//	system("pause");
	//	return 0;
	//}
	////��ͼ������Ͻ�һ��һ�е�д
	//ds1->GetRasterBand(1)->RasterIO(GF_Write, 0, 0, 200, 200, ObjRecon_1, 200, 200, GDT_Float32, 0, 0);
	//ds2->GetRasterBand(1)->RasterIO(GF_Write, 0, 0, 200, 200, imageRotated3D_1, 200, 200, GDT_Float32, 0, 0);
	//GDALClose(ds1);
	//GDALClose(ds2);



	return imageRotated3D;
}

//����X����ת
float *ObjRecon_imrotate3_X(float *imageRotated3D, double nAngle)
{
	//imageRotated3D(200*200*50)ת���ɣ��б�ɲ��Σ����α���У��б�ɷ��ţ����200��*50��*200����
	float *ObjRecon = new float[200 * 50 * 200];
	for (int i = 0; i < 200; i++)//�������ѭ�����������ѭ��
	{
		for (int j = 0; j < 200; j++)//�����ѭ�����������ѭ����������
		{
			for (int k = 0; k < 50; k++)//�����ѭ��������Ĳ���ѭ��
			{
				//ObjRecon[i * 200 * 50 + j * 50 + k] = imageRotated3D[199-j][i][49-k];
				ObjRecon[i * 200 * 50 + j * 50 + k] = imageRotated3D[(49 - k) * 200 * 200 + (199 - j) * 200 + i];
			}
		}
	}

	float *imageRotated3D_rotate = new float[200 * 50 * 200];

	NppiSize Input_Size;//����ͼ���������
	Input_Size.width = 200;
	Input_Size.height = 50;
	/* �����Դ棬��ԭͼ�����Դ� */
	int nSrcPitchCUDA = Input_Size.width * sizeof(float);//ÿ����ռ���ֽ���
	float *input_image_gpu;
	check1(hipMalloc((void**)&input_image_gpu, sizeof(float)*Input_Size.width*Input_Size.height), "input_image_gpu hipMalloc Error", __FILE__, __LINE__);


	/* ������ת�󳤿� */
	NppiRect Input_ROI;//�ض��������ת���൱�ڲü�ͼ���һ�飬���β���ȫ��ͼ��
	Input_ROI.x = Input_ROI.y = 0;
	Input_ROI.width = Input_Size.width;
	Input_ROI.height = Input_Size.height;
	double aBoundingBox[2][2];
	nppiGetRotateBound(Input_ROI, aBoundingBox, nAngle, 0, 0);
	int bb = ((int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0])) - Input_ROI.width) / 2 + aBoundingBox[0][0];//��ʼ��
	int cc = ((int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1])) - Input_ROI.height) / 2 + aBoundingBox[0][1];//��ʼ��
	aBoundingBox[0][0] = bb;//��ʼ��
	aBoundingBox[0][1] = cc;//��ʼ��
	NppiSize Output_Size;
	Output_Size.width = (int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0]));
	Output_Size.height = (int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1]));
	Output_Size.width = Input_Size.width;
	Output_Size.height = Input_Size.height;


	/* ת�����ͼ���Դ���� */
	int nDstPitchCUDA = Output_Size.width * sizeof(float);
	float *output_image_gpu;
	check1(hipMalloc((void**)&output_image_gpu, sizeof(float)*Output_Size.width*Output_Size.height), "output_image_gpu hipMalloc Error", __FILE__, __LINE__);


	//�������Ȥ���Ĵ�С���൱�ڰ����ͼ���ٲü�һ�飬Ӧ������������û���ԣ��������
	NppiRect Output_ROI;
	Output_ROI.x = 0; Output_ROI.y = 0;
	Output_ROI.width = Input_Size.width;
	Output_ROI.height = Input_Size.height;

	for (int i = 0; i < 200; i++)
	{
		check(hipMemcpy(input_image_gpu, ObjRecon + Input_Size.width*Input_Size.height * i, sizeof(float)*Input_Size.width*Input_Size.height, hipMemcpyHostToDevice), "input_image_gpu hipMemcpy Error");
		/* ������ת */
		NppStatus nppRet = nppiRotate_32f_C1R(input_image_gpu, Input_Size, nSrcPitchCUDA, Input_ROI,
			output_image_gpu, nDstPitchCUDA, Output_ROI, nAngle, -aBoundingBox[0][0], -aBoundingBox[0][1], NPPI_INTER_NN);
		assert(nppRet == NPP_NO_ERROR);
		check(hipMemcpy(imageRotated3D_rotate + Input_Size.width*Input_Size.height * i, output_image_gpu, sizeof(float) * Output_Size.width*Output_Size.height, hipMemcpyDeviceToHost), "output_image hipMemcpy Error");
	}

	////��תǰ��ĵ�һ�����ηֱ�д��������
	//float *ObjRecon_1 = new float[200 * 200];
	//float *imageRotated3D_1 = new float[200 * 200];
	//for (int i = 0; i < 200*200; i++)
	//{
	//	ObjRecon_1[i] = ObjRecon[i];
	//	imageRotated3D_1[i] = imageRotated3D[i];
	//}
	//GDALDriver * pDriver = GetGDALDriverManager()->GetDriverByName("GTiff");
	//GDALDataset *ds1 = pDriver->Create("ObjRecon_1_c", 200, 200, 1, GDT_Float32, NULL);
	//GDALDataset *ds2 = pDriver->Create("imageRotated3D_1_c", 200, 200, 1, GDT_Float32, NULL);
	//if ((ds1 == NULL) || (ds2 == NULL))
	//{
	//	cout << "create ObjRecon_1 imageRotated3D_1 output_file error!" << endl;
	//	system("pause");
	//	return 0;
	//}
	////��ͼ������Ͻ�һ��һ�е�д
	//ds1->GetRasterBand(1)->RasterIO(GF_Write, 0, 0, 200, 200, ObjRecon_1, 200, 200, GDT_Float32, 0, 0);
	//ds2->GetRasterBand(1)->RasterIO(GF_Write, 0, 0, 200, 200, imageRotated3D_1, 200, 200, GDT_Float32, 0, 0);
	//GDALClose(ds1);
	//GDALClose(ds2);

	//�ٱ任��ԭ����ά�ȷֲ�
	//200��*50��*200���Ρ�>>200��*200��*50��
	float *imageRotated3D_rotate_return = new float[200 * 50 * 200];
	for (int i = 0; i < 200; i++)//�������ѭ�����������ѭ��
	{
		for (int j = 0; j < 200; j++)//�����ѭ�����������ѭ����������
		{
			for (int k = 0; k < 50; k++)//�����ѭ��������Ĳ���ѭ��
			{
				imageRotated3D_rotate_return[(49 - k) * 200 * 200 + (199 - j) * 200 + i] = imageRotated3D_rotate[i * 200 * 50 + j * 50 + k];
			}
		}
	}

	return imageRotated3D_rotate_return;
}

//cpu�汾
int main0()
{
	//��ʼ��ʱ
	auto time_start = system_clock::now();
	GDALAllRegister();
	//����֧������·��
	CPLSetConfigOption("GDAL_FILENAME_IS_UTF8", "NO");
	CPLSetConfigOption("SHAPE_ENCODING", "");
	OGRRegisterAll();
	//CUDA ��ʼ��
	if (!InitCUDA())
	{
		cout << "CUDA֧���µ��Կ��豸��ʼ��ʧ��!" << endl;
		system("pause");
		return 0;
	}

	const char *rotationAngleXY_file = "F:/Archive/rotationAngleXY.dat";//360��double
	const char *rotationAngleYZ_file = "F:/Archive/rotationAngleYZ.dat";//31��double
	const char *template_roXY_file = "F:/Archive/template_roXY.dat";//200*200*360��float������matlab�������ȴ洢������һ�������ٴ�ڶ�������
	const char *template_roYZ_file = "F:/Archive/template_roYZ.dat";//200*50*31��float������matlab�������ȴ洢������һ�������ٴ�ڶ�������
	const char *ObjRecon_file = "F:/Archive/ObjRecon.dat";//200*200*50��float������matlab�������ȴ洢������һ�������ٴ�ڶ�������

	FILE * rotationAngleXY_fid = fopen(rotationAngleXY_file, "rb");
	if (rotationAngleXY_fid == NULL)
	{
		cout << rotationAngleXY_file << " open failed!" << endl;
		system("pause");
		return 0;
	}
	int rotationAngleXY_size = 360;
	double *rotationAngleXY = new double[rotationAngleXY_size];
	fread(rotationAngleXY, sizeof(double), rotationAngleXY_size, rotationAngleXY_fid);
	fclose(rotationAngleXY_fid);
	FILE * rotationAngleYZ_fid = fopen(rotationAngleYZ_file, "rb");
	if (rotationAngleYZ_fid == NULL)
	{
		cout << rotationAngleYZ_file << " open failed!" << endl;
		system("pause");
		return 0;
	}
	int rotationAngleYZ_size = 31;
	double *rotationAngleYZ = new double[rotationAngleYZ_size];
	fread(rotationAngleYZ, sizeof(double), rotationAngleYZ_size, rotationAngleYZ_fid);
	fclose(rotationAngleYZ_fid);
	FILE * template_roXY_fid = fopen(template_roXY_file, "rb");
	if (template_roXY_fid == NULL)
	{
		cout << template_roXY_file << " open failed!" << endl;
		system("pause");
		return 0;
	}
	int template_roXY_size = 200*200*360;
	float *template_roXY = new float[template_roXY_size];
	fread(template_roXY, sizeof(float), template_roXY_size, template_roXY_fid);
	fclose(template_roXY_fid);
	FILE * template_roYZ_fid = fopen(template_roYZ_file, "rb");
	if (template_roYZ_fid == NULL)
	{
		cout << template_roYZ_file << " open failed!" << endl;
		system("pause");
		return 0;
	}
	int template_roYZ_size = 200 * 50 * 31;
	float *template_roYZ = new float[template_roYZ_size];
	fread(template_roYZ, sizeof(float), template_roYZ_size, template_roYZ_fid);
	fclose(template_roYZ_fid);
	FILE * ObjRecon_fid = fopen(ObjRecon_file, "rb");
	if (ObjRecon_fid == NULL)
	{
		cout << ObjRecon_file << " open failed!" << endl;
		system("pause");
		return 0;
	}
	int ObjRecon_size = 200 * 200 * 50;
	float *ObjRecon = new float[ObjRecon_size];
	fread(ObjRecon, sizeof(float), ObjRecon_size, ObjRecon_fid);
	fclose(ObjRecon_fid);

	//����ObjReconһ�����������в����е����ֵ������matlab�еľ��������ȴ洢
	float *image2D_XY = new float[200 * 200];//��������������
	double image2D_XY_sum = 0;
	for (int i = 0; i < 200; i++)//��ѭ��
	{
		for (int j = 0; j < 200; j++)//��ѭ��
		{
			image2D_XY[i * 200 + j] = ObjRecon[i * 200 + j];
			for (int b = 0; b < 50; b++)//����ѭ��
			{
				if (image2D_XY[i * 200 + j] < ObjRecon[b * 200 * 200 + i * 200 + j])
				{
					image2D_XY[i * 200 + j] = ObjRecon[b * 200 * 200 + i * 200 + j];
				}
			}//����ѭ��
			image2D_XY_sum += image2D_XY[i * 200 + j];
		}
	}
	// ��ͶӰ��ֵ��: ����mean��ȡ1�� С�ڵ���mean��ȡ0
	double image2D_XY_mean = image2D_XY_sum / (200 * 200);
	float *img2DBW_XY = new float[200 * 200]();
	for (int i = 0; i < 200*200; i++)
	{
		if (image2D_XY[i] > image2D_XY_mean)
			img2DBW_XY[i] = 1.0;
		else
			img2DBW_XY[i] = 0.0;
	}

	// ��ÿ���Ƕȵ���� ��ʼ��
	double *err_XY = new double[rotationAngleXY_size];
	double err_XY_min = DBL_MAX;
	//���ֵ���������ÿһ���Ƕȵ����,GPU�п���ֱ�Ӳ���һ��ִ�����ѭ��
	for (int i = 0; i < rotationAngleXY_size; i++)
	{
		//������������ľ������
		double sum_temp = 0;
		for (int j = 0; j < 200; j++)//��ѭ��
		{
			for (int k = 0; k < 200; k++)//��ѭ��
			{
				sum_temp += (template_roXY[i * 200 * 200 + j * 200 + k] - img2DBW_XY[j * 200 + k])*(template_roXY[i * 200 * 200 + j * 200 + k] - img2DBW_XY[j * 200 + k]);
			}
		}
		err_XY[i] = sum_temp / (200 * 200);
		if (err_XY[i] < err_XY_min)
			err_XY_min = err_XY[i];
	}
	//�ҵ���Сֵ��Ӧ������
	int idx;
	for (int i = 0; i < rotationAngleXY_size; i++)
	{
		if (err_XY[i] == err_XY_min)
		{
			idx = i;
			break;
		}
	}
	
	//ObjRecon��200*200*50��float�����������У�����Z��˳ʱ����תrotationAngleXY[idx]��
	//��ת�Ƕ�Ϊ������ʱ�룬����˳ʱ��
	float *imageRotated3D = ObjRecon_imrotate3(ObjRecon, -rotationAngleXY[idx]);

	/* Y - Z rotation */
	//�� y-z���ͶӰ,����imageRotated3Dһ���������з�������ֵ
	float *image2D_YZ = new float[200 * 50];//200��*50�а���imageRotated3D�����������У���matlab�а�������������
	double image2D_YZ_sum = 0;
	for (int i = 0; i < 50; i++)//����ѭ��
	{
		for (int j = 0; j < 200; j++)//��ѭ��
		{
			image2D_YZ[i * 200 + j] = -FLT_MAX;
			for (int k = 0; k < 200; k++)//��ѭ������һ�е����ֵ
			{
				if (image2D_YZ[i * 200 + j] < imageRotated3D[i * 200 * 200 + j * 200 + k])
				{
					image2D_YZ[i * 200 + j] = imageRotated3D[i * 200 * 200 + j * 200 + k];
				}
			}
			image2D_YZ_sum += image2D_YZ[i * 200 + j];
		}
	}
	double image2D_YZ_mean = image2D_YZ_sum / (200 * 50) + 14;

	//��ֵ�� y-z�棬����mean��ȡ1�� С�ڵ���mean��ȡ0
	float *img2DBW_YZ = new float[200 * 50];
	for (int i = 0; i < 200 * 50; i++)
	{
		if (image2D_YZ[i] > image2D_YZ_mean)
			img2DBW_YZ[i] = 1.0;
		else
			img2DBW_YZ[i] = 0.0;
	}

	//��ÿ���Ƕȵ���� ��ʼ��
	double *err_YZ = new double[rotationAngleYZ_size];
	double err_YZ_min = DBL_MAX;
	//���ֵ���������ÿһ���Ƕȵ���GPU�п���ֱ�Ӳ���һ��ִ�����ѭ��
	for (int i = 0; i < rotationAngleYZ_size; i++)
	{
		//������������ľ������
		double sum_temp = 0;
		for (int j = 0; j < 200; j++)//��ѭ��
		{
			for (int k = 0; k < 50; k++)//��ѭ��
			{
				//template_roYZ��200��*50��*31���Σ����������У�img2DBW_YZ�����������е�
				sum_temp += (template_roYZ[i * 200 * 50 + j * 50 + k] - img2DBW_YZ[k * 200 + j])*(template_roYZ[i * 200 * 50 + j * 50 + k] - img2DBW_YZ[k * 200 + j]);
			}
		}
		err_YZ[i] = sum_temp / (200 * 50);
		if (err_YZ[i] < err_YZ_min)
			err_YZ_min = err_YZ[i];
	}
	//�ҵ���Сֵ��Ӧ������
	int idx2;
	for (int i = 0; i < rotationAngleYZ_size; i++)
	{
		if (err_YZ[i] == err_YZ_min)
		{
			idx2 = i;
			break;
		}
	}

	//imageRotated3D��ת����X������תrotationAngleYZ(idx2)��
	float *imageRotated3D_x = ObjRecon_imrotate3_X(imageRotated3D, rotationAngleYZ[idx2]);

	// Crop Out������ת���3Dͼ���е����г���
	// ��ֵ����ת���ͼ��
	//����imageRotated3D_x�ľ�ֵ
	double imageRotated3D_x_sum = 0;
	for (int i = 0; i < 200*200*50; i++)
	{
		imageRotated3D_x_sum += imageRotated3D_x[i];
	}
	double imageRotated3D_x_mean = imageRotated3D_x_sum / (200 * 200 * 50) + 4;

	int *BWObjRecon = new int[200 * 200 * 50]();
	int *idx_2 = new int[200 * 200 * 50]();//imageRotated3D_x���ھ�ֵ������
	int idx_2_size = 0;
	for (int i = 0; i < 200 * 200 * 50; i++)
	{
		if (imageRotated3D_x[i] > imageRotated3D_x_mean)
		{
			idx_2_size++;
			idx_2[idx_2_size] = i;
			BWObjRecon[i] = 1;
		}
		else
			BWObjRecon[i] = 0;
	}
	//idx_2�����ÿһ������ת����BWObjRecon��200��*200��*50���Σ����кţ��кţ����κ�
	float *x = new float[idx_2_size]; float x_sum = 0;
	float *y = new float[idx_2_size]; float y_sum = 0;
	float *z = new float[idx_2_size]; float z_sum = 0;
	for (int i = 0; i < idx_2_size; i++)
	{
		z[i] = idx_2[i] / (200 * 200);
		int yushu = idx_2[i] % (200 * 200);
		x[i] = yushu / 200;
		y[i] = yushu % 200;

		x_sum += x[i];
		y_sum += y[i];
		z_sum += z[i];
	}
	int CentroID[3];
	CentroID[0] = int(x_sum / idx_2_size + 0.5);
	CentroID[1] = int(y_sum / idx_2_size + 0.5);
	CentroID[2] = int(z_sum / idx_2_size + 0.5);
	//CentroID������matlab����[89,91,24]���Ҽ������[86,91,24],x���3����npp��ת��matlab�Ľ���������ɵģ����Ҳ��������������

	// ��������������Χ����������������matlab������Ҫ��ȥ1
	// �з�Χ����CentroID(0)-61��CentroID(0)+33�� ���з�Χ����CentroID(2)-38��CentroID(2)+37�������еĲ���
	int XObj = CentroID[0] + 33 - (CentroID[0] - 61) + 1;//��
	int	YObj = CentroID[2] + 37 - (CentroID[2] - 38) + 1;//��
	int	ZObj = 50;//����
	float *ObjReconRed = new float[XObj*YObj*ZObj];
	for (int i = 0; i < ZObj; i++)//����ѭ��
	{
		for (int j = 0; j < XObj; j++)//��ѭ��
		{
			for (int k = 0; k < YObj; k++)//��ѭ��
			{
				ObjReconRed[i*XObj*YObj + j*YObj + k] =
					imageRotated3D_x[i*200*200 + (CentroID[0] - 61 + j)*200 + CentroID[2] - 38 + k];
			}
		}
	}
	// size of reference atlas
	//int XRef = 95; int YRef = 76; int ZRef = 50;

	//������matlab�е�interp3������������ObjReconRed�������Ľ������matlab��RescaledRed����



	auto time_end = system_clock::now();
	auto duration = duration_cast<microseconds>(time_end - time_start);
	float usetime_total = double(duration.count()) * microseconds::period::num / microseconds::period::den;
	cout << "finish����ʱ��" << usetime_total << endl;
	system("pause");
    return 0;
}


__global__ void kernel_1(float *ObjRecon_gpu, int height, int width, float *image2D_XY_gpu)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//��ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//��ѭ��

	if (i < 200 && j < 200)
	{
		image2D_XY_gpu[i * 200 + j] = ObjRecon_gpu[i * 200 + j];
		for (int b = 0; b < 50; b++)//����ѭ��
		{
			if (image2D_XY_gpu[i * 200 + j] < ObjRecon_gpu[b * 200 * 200 + i * 200 + j])
			{
				image2D_XY_gpu[i * 200 + j] = ObjRecon_gpu[b * 200 * 200 + i * 200 + j];
			}
		}//����ѭ��
	}
}
__global__ void kernel_2(float *image2D_XY_gpu, int total, double image2D_XY_mean, float *img2DBW_XY_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		if (image2D_XY_gpu[i] > image2D_XY_mean)
			img2DBW_XY_gpu[i] = 1.0;
		else
			img2DBW_XY_gpu[i] = 0.0;
	}

}
__global__ void kernel_3(float *template_roXY_gpu, float *img2DBW_XY_gpu, int rotationAngleXY_size, double *err_XY_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < rotationAngleXY_size)
	{
		//������������ľ������
		double sum_temp = 0;
		for (int j = 0; j < 200; j++)//��ѭ��
		{
			for (int k = 0; k < 200; k++)//��ѭ��
			{
				sum_temp += (template_roXY_gpu[i * 200 * 200 + j * 200 + k] - img2DBW_XY_gpu[j * 200 + k])*
					(template_roXY_gpu[i * 200 * 200 + j * 200 + k] - img2DBW_XY_gpu[j * 200 + k]);
			}
		}
		err_XY_gpu[i] = sum_temp / (200 * 200);
	}
}
void ObjRecon_imrotate3_gpu(float *ObjRecon_gpu, double nAngle, float *imageRotated3D_gpu)
{
	NppiSize Input_Size;//����ͼ���������
	Input_Size.width = 200;
	Input_Size.height = 200;
	/* �����Դ棬��ԭͼ�����Դ� */
	int nSrcPitchCUDA = Input_Size.width * sizeof(float);//ÿ����ռ���ֽ���
	float *input_image_gpu;
	check1(hipMalloc((void**)&input_image_gpu, sizeof(float)*Input_Size.width*Input_Size.height), "input_image_gpu hipMalloc Error", __FILE__, __LINE__);


	/* ������ת�󳤿� */
	NppiRect Input_ROI;//�ض��������ת���൱�ڲü�ͼ���һ�飬���β���ȫ��ͼ��
	Input_ROI.x = Input_ROI.y = 0;
	Input_ROI.width = Input_Size.width;
	Input_ROI.height = Input_Size.height;
	double aBoundingBox[2][2];
	nppiGetRotateBound(Input_ROI, aBoundingBox, nAngle, 0, 0);
	int bb = ((int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0])) - Input_ROI.width) / 2 + aBoundingBox[0][0];//��ʼ��
	int cc = ((int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1])) - Input_ROI.height) / 2 + aBoundingBox[0][1];//��ʼ��
	aBoundingBox[0][0] = bb;//��ʼ��
	aBoundingBox[0][1] = cc;//��ʼ��
	NppiSize Output_Size;
	Output_Size.width = (int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0]));
	Output_Size.height = (int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1]));
	Output_Size.width = Input_Size.width;
	Output_Size.height = Input_Size.height;


	/* ת�����ͼ���Դ���� */
	int nDstPitchCUDA = Output_Size.width * sizeof(float);
	float *output_image_gpu;
	check1(hipMalloc((void**)&output_image_gpu, sizeof(float)*Output_Size.width*Output_Size.height), "output_image_gpu hipMalloc Error", __FILE__, __LINE__);


	//�������Ȥ���Ĵ�С���൱�ڰ����ͼ���ٲü�һ�飬Ӧ������������û���ԣ��������
	NppiRect Output_ROI;
	Output_ROI.x = 0; Output_ROI.y = 0;
	Output_ROI.width = Input_Size.width;
	Output_ROI.height = Input_Size.height;

	for (int i = 0; i < 50; i++)
	{
		check(hipMemcpy(input_image_gpu, ObjRecon_gpu + Input_Size.width*Input_Size.height * i, sizeof(float)*Input_Size.width*Input_Size.height, hipMemcpyDeviceToDevice), "input_image_gpu hipMemcpy Error");
		/* ������ת */
		NppStatus nppRet = nppiRotate_32f_C1R(input_image_gpu, Input_Size, nSrcPitchCUDA, Input_ROI,
			output_image_gpu, nDstPitchCUDA, Output_ROI, nAngle, -aBoundingBox[0][0], -aBoundingBox[0][1], NPPI_INTER_NN);
		assert(nppRet == NPP_NO_ERROR);
		check(hipMemcpy(imageRotated3D_gpu + Input_Size.width*Input_Size.height * i, output_image_gpu, sizeof(float) * Output_Size.width*Output_Size.height, hipMemcpyDeviceToDevice), "output_image hipMemcpy Error");
	}
}
__global__ void kernel_4(float *imageRotated3D_gpu, float *image2D_YZ_gpu)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//����ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//��ѭ��

	if (i < 50 && j < 200)
	{
		image2D_YZ_gpu[i * 200 + j] = -FLT_MAX;
		for (int k = 0; k < 200; k++)//��ѭ������һ�е����ֵ
		{
			if (image2D_YZ_gpu[i * 200 + j] < imageRotated3D_gpu[i * 200 * 200 + j * 200 + k])
			{
				image2D_YZ_gpu[i * 200 + j] = imageRotated3D_gpu[i * 200 * 200 + j * 200 + k];
			}
		}
	}
}
__global__ void kernel_5(float *image2D_YZ_gpu, double image2D_YZ_mean, float *img2DBW_YZ_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < 200 * 50)
	{
		if (image2D_YZ_gpu[i] > image2D_YZ_mean)
			img2DBW_YZ_gpu[i] = 1.0;
		else
			img2DBW_YZ_gpu[i] = 0.0;
	}
}
__global__ void kernel_6(float *template_roYZ_gpu, float *img2DBW_YZ_gpu, int rotationAngleYZ_size, double *err_YZ_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < rotationAngleYZ_size)
	{
		//������������ľ������
		double sum_temp = 0;
		for (int j = 0; j < 200; j++)//��ѭ��
		{
			for (int k = 0; k < 50; k++)//��ѭ��
			{
				//template_roYZ��200��*50��*31���Σ����������У�img2DBW_YZ�����������е�
				sum_temp += (template_roYZ_gpu[i * 200 * 50 + j * 50 + k] - img2DBW_YZ_gpu[k * 200 + j])*
					(template_roYZ_gpu[i * 200 * 50 + j * 50 + k] - img2DBW_YZ_gpu[k * 200 + j]);
			}
		}
		err_YZ_gpu[i] = sum_temp / (200 * 50);
	}
}
//ά�ȱ任
__global__ void kernel_7(float *imageRotated3D_gpu, float *imageRotated3D_gpu_1)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//�������ѭ�����������ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//�����ѭ�����������ѭ����������
	const int k = blockDim.z * blockIdx.z + threadIdx.z;//�����ѭ��������Ĳ���ѭ��

	if (i < 200 && j < 200 && k < 50)
	{
		//ObjRecon[i * 200 * 50 + j * 50 + k] = imageRotated3D[199-j][i][49-k];
		imageRotated3D_gpu_1[i * 200 * 50 + j * 50 + k] = imageRotated3D_gpu[(49 - k) * 200 * 200 + (199 - j) * 200 + i];
	}
}
//����X����ת
void ObjRecon_imrotate3_X_gpu(float *imageRotated3D_gpu_1, double nAngle, float *imageRotated3D_gpu_2)
{
	NppiSize Input_Size;//����ͼ���������
	Input_Size.width = 200;
	Input_Size.height = 50;
	/* �����Դ棬��ԭͼ�����Դ� */
	int nSrcPitchCUDA = Input_Size.width * sizeof(float);//ÿ����ռ���ֽ���
	float *input_image_gpu;
	check1(hipMalloc((void**)&input_image_gpu, sizeof(float)*Input_Size.width*Input_Size.height), "input_image_gpu hipMalloc Error", __FILE__, __LINE__);


	/* ������ת�󳤿� */
	NppiRect Input_ROI;//�ض��������ת���൱�ڲü�ͼ���һ�飬���β���ȫ��ͼ��
	Input_ROI.x = Input_ROI.y = 0;
	Input_ROI.width = Input_Size.width;
	Input_ROI.height = Input_Size.height;
	double aBoundingBox[2][2];
	nppiGetRotateBound(Input_ROI, aBoundingBox, nAngle, 0, 0);
	int bb = ((int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0])) - Input_ROI.width) / 2 + aBoundingBox[0][0];//��ʼ��
	int cc = ((int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1])) - Input_ROI.height) / 2 + aBoundingBox[0][1];//��ʼ��
	aBoundingBox[0][0] = bb;//��ʼ��
	aBoundingBox[0][1] = cc;//��ʼ��
	NppiSize Output_Size;
	Output_Size.width = (int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0]));
	Output_Size.height = (int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1]));
	Output_Size.width = Input_Size.width;
	Output_Size.height = Input_Size.height;


	/* ת�����ͼ���Դ���� */
	int nDstPitchCUDA = Output_Size.width * sizeof(float);
	float *output_image_gpu;
	check1(hipMalloc((void**)&output_image_gpu, sizeof(float)*Output_Size.width*Output_Size.height), "output_image_gpu hipMalloc Error", __FILE__, __LINE__);


	//�������Ȥ���Ĵ�С���൱�ڰ����ͼ���ٲü�һ�飬Ӧ������������û���ԣ��������
	NppiRect Output_ROI;
	Output_ROI.x = 0; Output_ROI.y = 0;
	Output_ROI.width = Input_Size.width;
	Output_ROI.height = Input_Size.height;

	for (int i = 0; i < 200; i++)
	{
		check(hipMemcpy(input_image_gpu, imageRotated3D_gpu_1 + Input_Size.width*Input_Size.height * i, sizeof(float)*Input_Size.width*Input_Size.height, hipMemcpyDeviceToDevice), "input_image_gpu hipMemcpy Error");
		/* ������ת */
		NppStatus nppRet = nppiRotate_32f_C1R(input_image_gpu, Input_Size, nSrcPitchCUDA, Input_ROI,
			output_image_gpu, nDstPitchCUDA, Output_ROI, nAngle, -aBoundingBox[0][0], -aBoundingBox[0][1], NPPI_INTER_NN);
		assert(nppRet == NPP_NO_ERROR);
		check(hipMemcpy(imageRotated3D_gpu_2 + Input_Size.width*Input_Size.height * i, output_image_gpu, sizeof(float) * Output_Size.width*Output_Size.height, hipMemcpyDeviceToDevice), "output_image hipMemcpy Error");
	}
}
//�ٱ任��ԭ����ά��
__global__ void kernel_8(float *imageRotated3D_gpu_2, float *imageRotated3D_gpu)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//�������ѭ�����������ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//�����ѭ�����������ѭ����������
	const int k = blockDim.z * blockIdx.z + threadIdx.z;//�����ѭ��������Ĳ���ѭ��

	if (i < 200 && j < 200 && k < 50)//�������ѭ�����������ѭ��
	{
		imageRotated3D_gpu[(49 - k) * 200 * 200 + (199 - j) * 200 + i] = imageRotated3D_gpu_2[i * 200 * 50 + j * 50 + k];
	}
}
__global__ void kernel_9(float *imageRotated3D_gpu, double imageRotated3D_x_mean, int *BWObjRecon_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < 200 * 200 * 50)
	{
		if (imageRotated3D_gpu[i] > imageRotated3D_x_mean)
			BWObjRecon_gpu[i] = 1;
		else
			BWObjRecon_gpu[i] = 0;
	}
}


//gpu�汾
int main()
{
	//��ʼ��ʱ
	auto time_start = system_clock::now();
	GDALAllRegister();
	//����֧������·��
	CPLSetConfigOption("GDAL_FILENAME_IS_UTF8", "NO");
	CPLSetConfigOption("SHAPE_ENCODING", "");
	OGRRegisterAll();
	//CUDA ��ʼ��
	if (!InitCUDA())
	{
		cout << "CUDA֧���µ��Կ��豸��ʼ��ʧ��!" << endl;
		system("pause");
		return 0;
	}

	const char *rotationAngleXY_file = "F:/Archive/rotationAngleXY.dat";//360��double
	const char *rotationAngleYZ_file = "F:/Archive/rotationAngleYZ.dat";//31��double
	const char *template_roXY_file = "F:/Archive/template_roXY.dat";//200*200*360��float������matlab�������ȴ洢������һ�������ٴ�ڶ�������
	const char *template_roYZ_file = "F:/Archive/template_roYZ.dat";//200*50*31��float������matlab�������ȴ洢������һ�������ٴ�ڶ�������
	const char *ObjRecon_file = "F:/Archive/ObjRecon.dat";//200*200*50��float������matlab�������ȴ洢������һ�������ٴ�ڶ�������

	FILE * rotationAngleXY_fid = fopen(rotationAngleXY_file, "rb");
	if (rotationAngleXY_fid == NULL)
	{
		cout << rotationAngleXY_file << " open failed!" << endl;
		system("pause");
		return 0;
	}
	int rotationAngleXY_size = 360;
	double *rotationAngleXY = new double[rotationAngleXY_size];
	fread(rotationAngleXY, sizeof(double), rotationAngleXY_size, rotationAngleXY_fid);
	fclose(rotationAngleXY_fid);
	FILE * rotationAngleYZ_fid = fopen(rotationAngleYZ_file, "rb");
	if (rotationAngleYZ_fid == NULL)
	{
		cout << rotationAngleYZ_file << " open failed!" << endl;
		system("pause");
		return 0;
	}
	int rotationAngleYZ_size = 31;
	double *rotationAngleYZ = new double[rotationAngleYZ_size];
	fread(rotationAngleYZ, sizeof(double), rotationAngleYZ_size, rotationAngleYZ_fid);
	fclose(rotationAngleYZ_fid);
	FILE * template_roXY_fid = fopen(template_roXY_file, "rb");
	if (template_roXY_fid == NULL)
	{
		cout << template_roXY_file << " open failed!" << endl;
		system("pause");
		return 0;
	}
	int template_roXY_size = 200 * 200 * 360;
	float *template_roXY = new float[template_roXY_size];
	fread(template_roXY, sizeof(float), template_roXY_size, template_roXY_fid);
	fclose(template_roXY_fid);
	FILE * template_roYZ_fid = fopen(template_roYZ_file, "rb");
	if (template_roYZ_fid == NULL)
	{
		cout << template_roYZ_file << " open failed!" << endl;
		system("pause");
		return 0;
	}
	int template_roYZ_size = 200 * 50 * 31;
	float *template_roYZ = new float[template_roYZ_size];
	fread(template_roYZ, sizeof(float), template_roYZ_size, template_roYZ_fid);
	fclose(template_roYZ_fid);
	FILE * ObjRecon_fid = fopen(ObjRecon_file, "rb");
	if (ObjRecon_fid == NULL)
	{
		cout << ObjRecon_file << " open failed!" << endl;
		system("pause");
		return 0;
	}
	int ObjRecon_size = 200 * 200 * 50;
	float *ObjRecon = new float[ObjRecon_size];
	fread(ObjRecon, sizeof(float), ObjRecon_size, ObjRecon_fid);
	fclose(ObjRecon_fid);

	//cpu
	//����ObjReconһ�����������в����е����ֵ������matlab�еľ��������ȴ洢
	float *image2D_XY = new float[200 * 200];//��������������
	double image2D_XY_sum = 0;
	for (int i = 0; i < 200; i++)//��ѭ��
	{
		for (int j = 0; j < 200; j++)//��ѭ��
		{
			image2D_XY[i * 200 + j] = ObjRecon[i * 200 + j];
			for (int b = 0; b < 50; b++)//����ѭ��
			{
				if (image2D_XY[i * 200 + j] < ObjRecon[b * 200 * 200 + i * 200 + j])
				{
					image2D_XY[i * 200 + j] = ObjRecon[b * 200 * 200 + i * 200 + j];
				}
			}//����ѭ��
			image2D_XY_sum += image2D_XY[i * 200 + j];
		}
	}
	//// ��ͶӰ��ֵ��: ����mean��ȡ1�� С�ڵ���mean��ȡ0
	//double image2D_XY_mean = image2D_XY_sum / (200 * 200);
	//float *img2DBW_XY = new float[200 * 200]();
	//for (int i = 0; i < 200 * 200; i++)
	//{
	//	if (image2D_XY[i] > image2D_XY_mean)
	//		img2DBW_XY[i] = 1.0;
	//	else
	//		img2DBW_XY[i] = 0.0;
	//}



	// ��CUDA..................
	float *ObjRecon_gpu;
	check1(hipMalloc((void**)&ObjRecon_gpu, sizeof(float)*ObjRecon_size), "ObjRecon_gpu hipMalloc Error", __FILE__, __LINE__);
	check(hipMemcpy(ObjRecon_gpu, ObjRecon, sizeof(float)*ObjRecon_size, hipMemcpyHostToDevice), "ObjRecon_gpu hipMemcpy Error");
	float *image2D_XY_gpu;
	check1(hipMalloc((void**)&image2D_XY_gpu, sizeof(float)* 200 * 200), "image2D_XY_gpu hipMalloc Error", __FILE__, __LINE__);
	dim3 block_1(32, 32, 1);
	dim3 grid_1((200 + block_1.x - 1) / block_1.x, (200 + block_1.y - 1) / block_1.y, 1);
	kernel_1 << <grid_1, block_1 >> > (ObjRecon_gpu, 200, 200, image2D_XY_gpu);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_1 Error");

	thrust::device_ptr<float> dev_ptr(image2D_XY_gpu);
	double image2D_XY_mean = thrust::reduce(dev_ptr, dev_ptr + size_t(200 * 200), (float)0, thrust::plus<float>()) / (200 * 200);
	float *img2DBW_XY_gpu;
	check1(hipMalloc((void**)&img2DBW_XY_gpu, sizeof(float)* 200 * 200), "img2DBW_XY_gpu hipMalloc Error", __FILE__, __LINE__);
	int threadNum_2 = 256;
	int blockNum_2 = (200 * 200 - 1) / threadNum_2 + 1;
	kernel_2 << <blockNum_2, threadNum_2 >> > (image2D_XY_gpu, 200*200, image2D_XY_mean, img2DBW_XY_gpu);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_2 Error");

	float *template_roXY_gpu;
	check1(hipMalloc((void**)&template_roXY_gpu, sizeof(float) * template_roXY_size), "template_roXY_gpu hipMalloc Error", __FILE__, __LINE__);
	check(hipMemcpy(template_roXY_gpu, template_roXY, sizeof(float)*template_roXY_size, hipMemcpyHostToDevice), "template_roXY_gpu hipMemcpy Error");
	double *err_XY_gpu;
	check1(hipMalloc((void**)&err_XY_gpu, sizeof(double) * rotationAngleXY_size), "err_XY_gpu hipMalloc Error", __FILE__, __LINE__);
	int threadNum_3 = 256;
	int blockNum_3 = (rotationAngleXY_size - 1) / threadNum_2 + 1;
	kernel_3 << <blockNum_3, threadNum_3 >> > (template_roXY_gpu, img2DBW_XY_gpu, rotationAngleXY_size, err_XY_gpu);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_3 Error");

	//��err_XY_gpu����Сֵ
	double *err_XY = new double[rotationAngleXY_size];
	check(hipMemcpy(err_XY, err_XY_gpu, sizeof(double)*rotationAngleXY_size, hipMemcpyDeviceToHost), "err_XY hipMemcpy Error");
	double err_XY_min = DBL_MAX;
	for (int i = 0; i < rotationAngleXY_size; i++)
	{
		if (err_XY[i] < err_XY_min)
			err_XY_min = err_XY[i];
	}
	//�ҵ���Сֵ��Ӧ������
	int idx;
	for (int i = 0; i < rotationAngleXY_size; i++)
	{
		if (err_XY[i] == err_XY_min)
		{
			idx = i;
			break;
		}
	}
	//��һ����ת
	float *imageRotated3D_gpu;
	check1(hipMalloc((void**)&imageRotated3D_gpu, sizeof(float) * 200 * 200 * 50), "imageRotated3D_gpu hipMalloc Error", __FILE__, __LINE__);
	ObjRecon_imrotate3_gpu(ObjRecon_gpu, -rotationAngleXY[idx], imageRotated3D_gpu);

	//�� y-z���ͶӰ,����imageRotated3Dһ���������з�������ֵ
	float *image2D_YZ_gpu;
	check1(hipMalloc((void**)&image2D_YZ_gpu, sizeof(float) * 200 * 50), "image2D_YZ_gpu hipMalloc Error", __FILE__, __LINE__);//200��*50�а���imageRotated3D�����������У���matlab�а�������������
	dim3 block_4(32, 32, 1);
	dim3 grid_4((50 + block_4.x - 1) / block_4.x, (200 + block_4.y - 1) / block_4.y, 1);
	kernel_4 << <grid_4, block_4 >> > (imageRotated3D_gpu, image2D_YZ_gpu);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_4 Error");
	//image2D_YZ_gpu��͡���ֵ
	thrust::device_ptr<float> dev_ptr1(image2D_YZ_gpu);
	double image2D_YZ_mean = thrust::reduce(dev_ptr1, dev_ptr1 + size_t(200 * 50), (float)0, thrust::plus<float>()) / (200 * 50) + 14;
	//��ֵ�� y-z�棬����mean��ȡ1�� С�ڵ���mean��ȡ0
	float *img2DBW_YZ_gpu;
	check1(hipMalloc((void**)&img2DBW_YZ_gpu, sizeof(float) * 200 * 50), "img2DBW_YZ_gpu hipMalloc Error", __FILE__, __LINE__);
	int threadNum_5 = 256;
	int blockNum_5 = (200 * 50 - 1) / threadNum_5 + 1;
	kernel_5 << <blockNum_5, threadNum_5 >> > (image2D_YZ_gpu, image2D_YZ_mean, img2DBW_YZ_gpu);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_5 Error");

	//��ÿ���Ƕȵ���� ��ʼ��
	float *template_roYZ_gpu;
	check1(hipMalloc((void**)&template_roYZ_gpu, sizeof(float) * template_roYZ_size), "template_roYZ_gpu hipMalloc Error", __FILE__, __LINE__);
	check(hipMemcpy(template_roYZ_gpu, template_roYZ, sizeof(float)*template_roYZ_size, hipMemcpyHostToDevice), "template_roYZ_gpu hipMemcpy Error");
	double *err_YZ_gpu;
	check1(hipMalloc((void**)&err_YZ_gpu, sizeof(double) * rotationAngleYZ_size), "err_YZ_gpu hipMalloc Error", __FILE__, __LINE__);
	int threadNum_6 = 256;
	int blockNum_6 = (rotationAngleYZ_size - 1) / threadNum_6 + 1;
	kernel_6 << <blockNum_5, threadNum_5 >> > (template_roYZ_gpu, img2DBW_YZ_gpu, rotationAngleYZ_size, err_YZ_gpu);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_6 Error");
	//��err_YZ_gpu����Сֵ����Сֵ������
	double *err_YZ = new double[rotationAngleYZ_size];
	check(hipMemcpy(err_YZ, err_YZ_gpu, sizeof(double)*rotationAngleYZ_size, hipMemcpyDeviceToHost), "err_YZ hipMemcpy Error");
	double err_YZ_min = DBL_MAX;
	for (int i = 0; i < rotationAngleYZ_size; i++)
	{
		if (err_YZ[i] < err_YZ_min)
			err_YZ_min = err_YZ[i];
	}
	int idx2;
	for (int i = 0; i < rotationAngleYZ_size; i++)
	{
		if (err_YZ[i] == err_YZ_min)
		{
			idx2 = i;
			break;
		}
	}
	//imageRotated3D��ת����X������תrotationAngleYZ(idx2)��
	//�Ȱ�imageRotated3D_gpu��ά�ȱ任һ�£��б�ɲ��Σ����α���У��б�ɷ��ţ�(200 * 200 * 50)���(200�� * 50�� * 200)����
	float *imageRotated3D_gpu_1;
	check1(hipMalloc((void**)&imageRotated3D_gpu_1, sizeof(float)*ObjRecon_size), "imageRotated3D_gpu_1 hipMalloc Error", __FILE__, __LINE__);
	dim3 block_7(8, 8, 8);
	dim3 grid_7((200 + block_7.x - 1) / block_7.x, (200 + block_7.y - 1) / block_7.y, (50 + block_7.z - 1) / block_7.z);
	kernel_7 << <grid_7, block_7 >> > (imageRotated3D_gpu, imageRotated3D_gpu_1);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_7 Error");
	//�ڶ�����ת
	float *imageRotated3D_gpu_2;
	check1(hipMalloc((void**)&imageRotated3D_gpu_2, sizeof(float)*ObjRecon_size), "imageRotated3D_gpu_2 hipMalloc Error", __FILE__, __LINE__);
	ObjRecon_imrotate3_X_gpu(imageRotated3D_gpu_1, rotationAngleYZ[idx2], imageRotated3D_gpu_2);
	//�ٰ�ά�ȱ任��ԭ����
	dim3 block_8(8, 8, 8);
	dim3 grid_8((200 + block_7.x - 1) / block_7.x, (200 + block_7.y - 1) / block_7.y, (50 + block_7.z - 1) / block_7.z);
	kernel_8 << <grid_8, block_8 >> > (imageRotated3D_gpu_2, imageRotated3D_gpu);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_8 Error");
	//����imageRotated3D_gpu�ľ�ֵ
	thrust::device_ptr<float> dev_ptr2(imageRotated3D_gpu);
	double imageRotated3D_x_mean = thrust::reduce(dev_ptr2, dev_ptr2 + size_t(200 * 200 * 50), (float)0, thrust::plus<float>()) / (200 * 200 * 50) + 4;

	check(hipMemcpy(ObjRecon, imageRotated3D_gpu, sizeof(float)*ObjRecon_size, hipMemcpyDeviceToHost), "ObjRecon hipMemcpy Error");

	





	//// ��ÿ���Ƕȵ���� ��ʼ��
	//double *err_XY = new double[rotationAngleXY_size];
	//double err_XY_min = DBL_MAX;
	////���ֵ���������ÿһ���Ƕȵ����,GPU�п���ֱ�Ӳ���һ��ִ�����ѭ��
	//for (int i = 0; i < rotationAngleXY_size; i++)
	//{
	//	//������������ľ������
	//	double sum_temp = 0;
	//	for (int j = 0; j < 200; j++)//��ѭ��
	//	{
	//		for (int k = 0; k < 200; k++)//��ѭ��
	//		{
	//			sum_temp += (template_roXY[i * 200 * 200 + j * 200 + k] - img2DBW_XY[j * 200 + k])*(template_roXY[i * 200 * 200 + j * 200 + k] - img2DBW_XY[j * 200 + k]);
	//		}
	//	}
	//	err_XY[i] = sum_temp / (200 * 200);
	//	if (err_XY[i] < err_XY_min)
	//		err_XY_min = err_XY[i];
	//}
	////�ҵ���Сֵ��Ӧ������
	//int idx;
	//for (int i = 0; i < rotationAngleXY_size; i++)
	//{
	//	if (err_XY[i] == err_XY_min)
	//	{
	//		idx = i;
	//		break;
	//	}
	//}
	//
	//ObjRecon��200*200*50��float�����������У�����Z��˳ʱ����תrotationAngleXY[idx]��
	//��ת�Ƕ�Ϊ������ʱ�룬����˳ʱ��
	//float *imageRotated3D = ObjRecon_imrotate3(ObjRecon, -rotationAngleXY[idx]);
	////* Y - Z rotation */
	////�� y-z���ͶӰ,����imageRotated3Dһ���������з�������ֵ
	//float *image2D_YZ = new float[200 * 50];//200��*50�а���imageRotated3D�����������У���matlab�а�������������
	//double image2D_YZ_sum = 0;
	//for (int i = 0; i < 50; i++)//����ѭ��
	//{
	//	for (int j = 0; j < 200; j++)//��ѭ��
	//	{
	//		image2D_YZ[i * 200 + j] = -FLT_MAX;
	//		for (int k = 0; k < 200; k++)//��ѭ������һ�е����ֵ
	//		{
	//			if (image2D_YZ[i * 200 + j] < imageRotated3D[i * 200 * 200 + j * 200 + k])
	//			{
	//				image2D_YZ[i * 200 + j] = imageRotated3D[i * 200 * 200 + j * 200 + k];
	//			}
	//		}
	//		image2D_YZ_sum += image2D_YZ[i * 200 + j];
	//	}
	//}
	//double image2D_YZ_mean = image2D_YZ_sum / (200 * 50) + 14;
	//
	////��ֵ�� y-z�棬����mean��ȡ1�� С�ڵ���mean��ȡ0
	//float *img2DBW_YZ = new float[200 * 50];
	//for (int i = 0; i < 200 * 50; i++)
	//{
	//	if (image2D_YZ[i] > image2D_YZ_mean)
	//		img2DBW_YZ[i] = 1.0;
	//	else
	//		img2DBW_YZ[i] = 0.0;
	//}
	//
	////��ÿ���Ƕȵ���� ��ʼ��
	//double *err_YZ = new double[rotationAngleYZ_size];
	//double err_YZ_min = DBL_MAX;
	//���ֵ���������ÿһ���Ƕȵ���GPU�п���ֱ�Ӳ���һ��ִ�����ѭ��
	//for (int i = 0; i < rotationAngleYZ_size; i++)
	//{
	//	//������������ľ������
	//	double sum_temp = 0;
	//	for (int j = 0; j < 200; j++)//��ѭ��
	//	{
	//		for (int k = 0; k < 50; k++)//��ѭ��
	//		{
	//			//template_roYZ��200��*50��*31���Σ����������У�img2DBW_YZ�����������е�
	//			sum_temp += (template_roYZ[i * 200 * 50 + j * 50 + k] - img2DBW_YZ[k * 200 + j])*(template_roYZ[i * 200 * 50 + j * 50 + k] - img2DBW_YZ[k * 200 + j]);
	//		}
	//	}
	//	err_YZ[i] = sum_temp / (200 * 50);
	//	if (err_YZ[i] < err_YZ_min)
	//		err_YZ_min = err_YZ[i];
	//}
	////�ҵ���Сֵ��Ӧ������
	//int idx2;
	//for (int i = 0; i < rotationAngleYZ_size; i++)
	//{
	//	if (err_YZ[i] == err_YZ_min)
	//	{
	//		idx2 = i;
	//		break;
	//	}
	//}
	//
	////imageRotated3D��ת����X������תrotationAngleYZ(idx2)��
	////float *imageRotated3D_x = ObjRecon_imrotate3_X(imageRotated3D, rotationAngleYZ[idx2]);
	//
	//// Crop Out������ת���3Dͼ���е����г���
	//// ��ֵ����ת���ͼ��
	////����imageRotated3D_x�ľ�ֵ
	//double imageRotated3D_x_sum = 0;
	//for (int i = 0; i < 200 * 200 * 50; i++)
	//{
	//	imageRotated3D_x_sum += imageRotated3D_x[i];
	//}
	//double imageRotated3D_x_mean = imageRotated3D_x_sum / (200 * 200 * 50) + 4;

	int *idx_2 = new int[200 * 200 * 50]();//imageRotated3D_x���ھ�ֵ������
	int idx_2_size = 0;
	for (int i = 0; i < 200 * 200 * 50; i++)
	{
		if (ObjRecon[i] > imageRotated3D_x_mean)
		{
			idx_2_size++;
			idx_2[idx_2_size] = i;
		}
	}
	//idx_2�����ÿһ������ת����imageRotated3D_x��200��*200��*50���Σ����кţ��кţ����κ�
	float *x = new float[idx_2_size]; float x_sum = 0;
	float *y = new float[idx_2_size]; float y_sum = 0;
	float *z = new float[idx_2_size]; float z_sum = 0;
	for (int i = 0; i < idx_2_size; i++)
	{
		z[i] = idx_2[i] / (200 * 200);
		int yushu = idx_2[i] % (200 * 200);
		x[i] = yushu / 200;
		y[i] = yushu % 200;

		x_sum += x[i];
		y_sum += y[i];
		z_sum += z[i];
	}
	int CentroID[3];
	CentroID[0] = int(x_sum / idx_2_size + 0.5);
	CentroID[1] = int(y_sum / idx_2_size + 0.5);
	CentroID[2] = int(z_sum / idx_2_size + 0.5);
	//CentroID������matlab����[89,91,24]���Ҽ������[86,91,24],x���3����npp��ת��matlab�Ľ���������ɵģ����Ҳ��������������

	// ��������������Χ����������������matlab������Ҫ��ȥ1
	// �з�Χ����CentroID(0)-61��CentroID(0)+33�� ���з�Χ����CentroID(2)-38��CentroID(2)+37�������еĲ���
	int XObj = CentroID[0] + 33 - (CentroID[0] - 61) + 1;//��
	int	YObj = CentroID[2] + 37 - (CentroID[2] - 38) + 1;//��
	int	ZObj = 50;//����
	float *ObjReconRed = new float[XObj*YObj*ZObj];
	for (int i = 0; i < ZObj; i++)//����ѭ��
	{
		for (int j = 0; j < XObj; j++)//��ѭ��
		{
			for (int k = 0; k < YObj; k++)//��ѭ��
			{
				ObjReconRed[i*XObj*YObj + j*YObj + k] =
					ObjRecon[i * 200 * 200 + (CentroID[0] - 61 + j) * 200 + CentroID[2] - 38 + k];
			}
		}
	}
	// size of reference atlas
	//int XRef = 95; int YRef = 76; int ZRef = 50;

	//������matlab�е�interp3������������ObjReconRed�������Ľ������matlab��RescaledRed����



	auto time_end = system_clock::now();
	auto duration = duration_cast<microseconds>(time_end - time_start);
	float usetime_total = double(duration.count()) * microseconds::period::num / microseconds::period::den;
	cout << "finish����ʱ��" << usetime_total << endl;
	system("pause");
	return 0;
}


