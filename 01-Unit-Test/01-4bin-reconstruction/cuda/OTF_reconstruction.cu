#include "hip/hip_runtime.h"
#include "header.cuh"
#include <chrono>
using namespace chrono;

void printDeviceProp(const hipDeviceProp_t &prop)
{
	printf("GPU Parament:\n");
	printf(" Device Name : %s.\n", prop.name);
	printf(" totalGlobalMem : %I64d.\n", prop.totalGlobalMem);
	printf(" sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf(" regsPerBlock : %d.\n", prop.regsPerBlock);
	printf(" warpSize : %d.\n", prop.warpSize);
	printf(" memPitch : %d.\n", prop.memPitch);
	printf(" maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf(" maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf(" maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf(" totalConstMem : %d.\n", prop.totalConstMem);
	printf(" major.minor : %d.%d.\n", prop.major, prop.minor);
	printf(" clockRate : %d.\n", prop.clockRate);
	printf(" textureAlignment : %d.\n", prop.textureAlignment);
	printf(" deviceOverlap : %d.\n", prop.deviceOverlap);
	printf(" multiProcessorCount : %d.\n", prop.multiProcessorCount);
	std::printf(" CUDA core: %d\r\n", 2 * prop.multiProcessorCount* prop.maxThreadsPerMultiProcessor / prop.warpSize);
	printf("\n\n");
}
//CUDA Initialization
bool InitCUDA()
{
	int count;
	//Get the number of Cuda-enabled devices
	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		//Print device information
		printDeviceProp(prop);
		//Get the clock frequency of the GPU
		clockRate = prop.clockRate;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}
	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}
void check(hipError_t res, string warningstring)
{
	if (res != hipSuccess)
	{
		printf((warningstring + " !\n").c_str());
		system("pause");
		exit(0);
	}
}
//View GPU operation status
void checkGPUStatus(hipError_t hipGetLastError, string warningstring)
{
	if (hipGetLastError != hipSuccess)
	{
		printf("\n\n");
		printf((warningstring + " !\n").c_str());
		fprintf(stderr, "%s\n", hipGetErrorString(hipGetLastError));
		system("pause");
		exit(0);
	}
}

__global__ void Zhuan_Complex_kernel(float *PSF_1_gpu, hipfftComplex *PSF_1_gpu_Complex, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		PSF_1_gpu_Complex[i].x = PSF_1_gpu[i];
		PSF_1_gpu_Complex[i].y = 0;
	}
}
__global__ void PSF_unshort(float *PSF_1_gpu, unsigned short *PSF, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		if (PSF_1_gpu[i] < 0)
		{
			PSF[i] = 0;
		}
		else if (PSF_1_gpu[i] > 65535)
		{
			PSF[i] = 65535;
		}
		else
		{
			PSF[i] = (int)(PSF_1_gpu[i] + 0.5);
		}
	}
}
__global__ void initial_kernel_1(float *ImgEst, float *Ratio, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		ImgEst[i] = 0;
		Ratio[i] = 1;
	}
}
__global__ void gpuObjRecon_fuzhi(float *gpuObjRecon, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		gpuObjRecon[i] = 1;
	}
}
__global__ void initial_kernel_3(float *gpuObjRecROI, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		gpuObjRecROI[i] = 1;
	}
}
__global__ void ImgExp_ge(unsigned short *Img_gpu, int BkgMean, float *ImgExp, int total)
{
	//Turn the result of the difference less than 0 into 0, greater than 0 rounded, greater than 65535 into 65535
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		if ((Img_gpu[i] - BkgMean) < 0)
		{
			ImgExp[i] = 0;
		}
		else if ((Img_gpu[i] - BkgMean) > 65535)
		{
			ImgExp[i] = 65535;
		}
		else
		{
			ImgExp[i] = (int)((Img_gpu[i] - BkgMean) + 0.5);
		}
	}
}
__global__ void Ratio_fuzhi(float *Ratio, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		Ratio[i] = 1;
	}
}
__global__ void OTF_mul_gpuObjRecon_Complex(hipfftComplex *OTF, hipfftComplex *gpuObjRecon_Complex, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		float aaa = OTF[i].x*gpuObjRecon_Complex[i].x - OTF[i].y*gpuObjRecon_Complex[i].y;//Real part Results
		float bbb = OTF[i].x*gpuObjRecon_Complex[i].y + OTF[i].y*gpuObjRecon_Complex[i].x;//Virtual Part Results
		gpuObjRecon_Complex[i].x = aaa;
		gpuObjRecon_Complex[i].y = bbb;
	}
}
__global__ void ifftshift_real_max(hipfftComplex *OTF, float *float_temp, int PSF_size_1, int PSF_size_2, int PSF_size_3)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	const int k = blockDim.z * blockIdx.z + threadIdx.z;
	int lie_half = PSF_size_2 / 2;
	if (i < PSF_size_1 / 2 && j < PSF_size_2 && k < PSF_size_3)
	{
		//Implement the image ifftshift+real+max, i.e.: divide the image into 4 quadrants, the first and third translation swap, the second and fourth translation swap
		float_temp[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * 512] = OTF[k*PSF_size_1*PSF_size_2 + i*PSF_size_2 + j].x >= 0 ? OTF[k*PSF_size_1*PSF_size_2 + i*PSF_size_2 + j].x : 0;
		float_temp[k*PSF_size_1*PSF_size_2 + i*PSF_size_2 + j] = OTF[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * PSF_size_2].x >= 0 ? OTF[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * PSF_size_2].x : 0;
	}
}
__global__ void ifftshift(hipfftComplex *OTF, float *float_temp, int PSF_size_1, int PSF_size_2, int PSF_size_3, hipfftComplex *OTF_ifftshift)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	const int k = blockDim.z * blockIdx.z + threadIdx.z;
	int lie_half = PSF_size_2 / 2;
	if (i < PSF_size_1 / 2 && j < PSF_size_2 && k < PSF_size_3)
	{
		//Implement the image ifftshift, i.e.: divide the image into 4 quadrants, first and third translation swap, second and fourth translation swap
		OTF_ifftshift[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * 512] = OTF[k*PSF_size_1*PSF_size_2 + i*PSF_size_2 + j];
		OTF_ifftshift[k*PSF_size_1*PSF_size_2 + i*PSF_size_2 + j] = OTF[k*PSF_size_1*PSF_size_2 + (i + PSF_size_1 / 2)*PSF_size_2 + j + lie_half - j / lie_half * PSF_size_2];
	}
}
__global__ void float_temp_sum(float *float_temp, float *ImgEst, int PSF_size_1, int PSF_size_2, int PSF_size_3)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < PSF_size_1 && j < PSF_size_2)
	{
		ImgEst[i*PSF_size_2 + j] = 0;
		for (int k = 0; k < PSF_size_3; k++)
		{
			ImgEst[i*PSF_size_2 + j] += float_temp[k*PSF_size_1*PSF_size_2 + (i*PSF_size_2 + j)];
		}
	}
}
__global__ void Ratio_fuzhi_2(float *ImgExp, float *ImgEst, float Tmp, int SNR, float *Ratio, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		Ratio[i] = ImgExp[i]/(ImgEst[i] + Tmp / SNR);
	}
}
__global__ void Ratio_Complex_ge(float *ImgExp, float *ImgEst, float Tmp, int SNR, hipfftComplex *Ratio_Complex, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		Ratio_Complex[i].x = ImgExp[i] / (ImgEst[i] + Tmp / SNR);
		Ratio_Complex[i].y = 0;
	}
}
__global__ void fftRatio_ge(hipfftComplex *Ratio_Complex, hipfftComplex *fftRatio, int PSF_size_1, int PSF_size_2, int PSF_size_3)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	const int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i < PSF_size_1 && j < PSF_size_2 && k < PSF_size_3)
	{
		fftRatio[k*PSF_size_1*PSF_size_2 + i*PSF_size_2 + j] = Ratio_Complex[i*PSF_size_2 + j];
	}
}
__global__ void fftceshi_gpu_fuzhi(hipfftComplex *PSF_1_gpu_Complex, hipfftComplex *fftceshi_gpu, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		fftceshi_gpu[i] = PSF_1_gpu_Complex[i];
	}
}
__global__ void ifft2_divide(hipfftComplex *OTF, int total, int scale)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		OTF[i].x = OTF[i].x / scale;
		OTF[i].y = OTF[i].y / scale;
	}
}
__global__ void real_multiply(float *gpuObjRecon, float *float_temp, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		gpuObjRecon[i] = gpuObjRecon[i] * float_temp[i];
	}
}
__global__ void fftRatio_mul_conjOTF(hipfftComplex *fftRatio, hipfftComplex *OTF, int total)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		float aaa = fftRatio[i].x*OTF[i].x + fftRatio[i].y*OTF[i].y;//Result of real part of conjugate multiplication
		float bbb = -fftRatio[i].x*OTF[i].y + fftRatio[i].y*OTF[i].x;//Result of the virtual part of the conjugate multiplication
		fftRatio[i].x = aaa;
		fftRatio[i].y = bbb;
	}
}


int main()
{

	//Start Timer
	auto time_start = system_clock::now();

	const char *PSF_1_file = "F:/matlab-cuda-20220125/PSF_1_zhuanzhi_float.dat";//matlab�б��������float����
	const char *X31_file = "F:/matlab-cuda-20220125/r20210924_2_X31_resize.tif";
	const char *OutFile = "F:/matlab-cuda-20220125/result_C.dat";
	FILE *PSF_1_fid = fopen(PSF_1_file, "rb");
	if (PSF_1_fid == NULL)
	{
		cout << "PSF_1_file open failed!" << endl;
		system("pause");
		return 0;
	}
	float *PSF_1 = new float[PSF_size_1*PSF_size_2*PSF_size_3]();
	fread(PSF_1, sizeof(float), PSF_size_1*PSF_size_2*PSF_size_3, PSF_1_fid);

	//Read tif using GDAL, using matlab resampled data
	GDALAllRegister(); OGRRegisterAll();
	CPLSetConfigOption("GDAL_FILENAME_IS_UTF8", "NO");
	CPLSetConfigOption("SHAPE_ENCODING", "");
	GDALDataset* poSrcDS = (GDALDataset*)GDALOpen(X31_file, GA_ReadOnly);  
	if (poSrcDS == NULL)
	{
		cout << "X31_file open failed!" << endl;
		return;
	}
	int wheight = poSrcDS->GetRasterYSize();
	int wwidth = poSrcDS->GetRasterXSize();
	int bandNum = poSrcDS->GetRasterCount();
	GDALDataType dataType = poSrcDS->GetRasterBand(1)->GetRasterDataType();
	unsigned short *Img = new unsigned short[PSF_size_1*PSF_size_2]();
	for (int i = 0; i < bandNum; i++)
	{
		poSrcDS->GetRasterBand(i + 1)->RasterIO(GF_Read, 0, 0, wwidth, wheight, Img, PSF_size_1, PSF_size_2, dataType, 0, 0);
	}
	GDALClose(poSrcDS);

	/*-----------------Preparation----------------*/
	const int rank = 2;
	int n[rank] = { PSF_size_1, PSF_size_2 };//n*m
	int *inembed = n;//The size of the input array
	int istride = 1;//The data in the array is continuous is 1
	int idist = n[0] * n[1];//Memory size of one array
	int *onembed = n;//The output is the size of an array
	int ostride = 1;//1 if the data is continuous after each point of DFT
	int odist = n[0] * n[1];//Output the distance between the first array and the second array, i.e. the distance between the first elements of the two arrays
	int batch = PSF_size_3;//Number of batches
	hipfftHandle fftplanfwd;//Create handle
	hipfftPlanMany(&fftplanfwd, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);

	float *PSF_1_gpu;
	check(hipMalloc((void**)&PSF_1_gpu, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float)), "PSF_1_gpu hipMalloc Error");
	hipfftComplex *PSF_1_gpu_Complex;
	check(hipMalloc((void**)&PSF_1_gpu_Complex, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "PSF_1_gpu_Complex hipMalloc Error");
	hipfftComplex *OTF;
	check(hipMalloc((void**)&OTF, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "OTF hipMalloc Error");
	float *ImgEst;
	check(hipMalloc((void**)&ImgEst, PSF_size_1*PSF_size_2 * sizeof(float)), "ImgEst hipMalloc Error");
	float *Ratio;
	check(hipMalloc((void**)&Ratio, PSF_size_1*PSF_size_2 * sizeof(float)), "Ratio hipMalloc Error");
	float *gpuObjRecon;
	check(hipMalloc((void**)&gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float)), "Ratio hipMalloc Error");
	float *gpuObjRecROI;
	check(hipMalloc((void**)&gpuObjRecROI, ROISize * 2 * ROISize * 2 * PSF_size_3 * sizeof(float)), "gpuObjRecROI hipMalloc Error");
	unsigned short *Img_gpu;
	check(hipMalloc((void**)&Img_gpu, PSF_size_1*PSF_size_2 * sizeof(unsigned short)), "Img_gpu hipMalloc Error");
	float *ImgExp;
	check(hipMalloc((void**)&ImgExp, PSF_size_1*PSF_size_2 * sizeof(float)), "ImgExp hipMalloc Error");
	hipfftComplex *gpuObjRecon_Complex;
	check(hipMalloc((void**)&gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "gpuObjRecon_Complex hipMalloc Error");
	float *float_temp;
	check(hipMalloc((void**)&float_temp, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float)), "float_temp hipMalloc Error");
	hipfftComplex *Ratio_Complex;
	check(hipMalloc((void**)&Ratio_Complex, PSF_size_1*PSF_size_2 * sizeof(hipfftComplex)), "Ratio_Complex hipMalloc Error");
	hipfftComplex *fftRatio;
	check(hipMalloc((void**)&fftRatio, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "fftRatio hipMalloc Error");




	/*-------------------------------start--------------------------------------------*/
	auto time_1 = system_clock::now();
	/*-----fft2 on PSF_1-------*/
	check(hipMemcpy(PSF_1_gpu, PSF_1, PSF_size_1*PSF_size_2*PSF_size_3*sizeof(float), hipMemcpyHostToDevice), "PSF_1_gpu hipMemcpy Error");
	//Convert to complex numbers with virtual part 0
	Zhuan_Complex_kernel << <blockNum_123, threadNum_123 >> > (PSF_1_gpu, PSF_1_gpu_Complex, PSF_size_1*PSF_size_2*PSF_size_3);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "PSF_1_gpu Zhuan_Complex_kernel Error");
	

	////*------------Bulk 2D fft using hipfftPlanMany's method-------------*/
	hipfftExecC2C(fftplanfwd, PSF_1_gpu_Complex, OTF, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "PSF_1_gpu_Complex hipfftExecC2C Error");
	
	////ImgEst is assigned a value of 0 and Ratio is assigned a value of 1
	initial_kernel_1 << <blockNum_12, threadNum_12 >> > (ImgEst, Ratio, PSF_size_1*PSF_size_2);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "initial_kernel_1 Error");
	gpuObjRecon_fuzhi << <blockNum_123, threadNum_123 >> > (gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "gpuObjRecon_fuzhi Error");
	////gpuObjRecROI is assigned a value of 1
	initial_kernel_3 << <blockNum_ROI, threadNum_ROI >> > (gpuObjRecROI, ROISize * 2 * ROISize * 2 * Nz);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "initial_kernel_3 Error");

	//Copy tif data to video memory
	check(hipMemcpy(Img_gpu, Img, PSF_size_1*PSF_size_2 * sizeof(unsigned short), hipMemcpyHostToDevice), "Img_gpu hipMemcpy Error");
	//Subtract the background mean value and put the result in the float type array ImgExp
	ImgExp_ge << <blockNum_12, threadNum_12 >> > (Img_gpu, BkgMean, ImgExp, PSF_size_1*PSF_size_2);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "ImgExp_ge Error");

	//The elements of Ratio and gpuObjRecon are assigned the value 1
	Ratio_fuzhi << <blockNum_12, threadNum_12 >> > (Ratio, PSF_size_1*PSF_size_2);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "Ratio_fuzhi Error");
	gpuObjRecon_fuzhi << <blockNum_123, threadNum_123 >> > (gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "gpuObjRecon_fuzhi Error");


	//iteration
	for (int i = 0; i < ItN; i++)
	{
		////1 fft2(gpuObjRecon)
		Zhuan_Complex_kernel << <blockNum_123, threadNum_123 >> > (gpuObjRecon, gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "Zhuan_Complex_kernel Error");
		hipfftExecC2C(fftplanfwd, gpuObjRecon_Complex, gpuObjRecon_Complex, HIPFFT_FORWARD);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon_Complex hipfftExecC2C Error");

		////2 OTF.*fft2(gpuObjRecon_Complex),The results are placed in gpuObjRecon_Complex
		OTF_mul_gpuObjRecon_Complex << <blockNum_123, threadNum_123 >> > (OTF, gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "2��OTF.*fft2(gpuObjRecon_Complex) Error");

		////3 ifft2(OTF.*fft2(gpuObjRecon)) The inverse conversion requires dividing by the total number of pixels
		hipfftExecC2C(fftplanfwd, gpuObjRecon_Complex, gpuObjRecon_Complex, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon_Complex hipfftExecC2C cufft_inverse Error");
		////4 Divide by the total number of pixels to be correct
		ifft2_divide << <blockNum_123, threadNum_123 >> > (gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3, PSF_size_1*PSF_size_2);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon_Complex ifft2_divide Error");

		////5 ifftshift + real + max(,0), Get the real part of the matrix float_temp, less than 0 assign 0
		ifftshift_real_max << <grid, block >> > (gpuObjRecon_Complex, float_temp, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "5��gpuObjRecon_Complex ifftshift_real_max Error");

		////6 sum( ,3), Calculate the sum in the third dimension and return the matrix ImgEst of PSF_size_1 rows and PSF_size_2 columns
		float_temp_sum << <grid_sum, block_sum >> > (float_temp, ImgEst, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "float_temp_sum Error");
		
		////7��Tmp=mean(   ImgEst(:)   );
		thrust::device_ptr<float> dev_ptr(ImgEst);
		float Tmp = thrust::reduce(dev_ptr, dev_ptr + size_t(PSF_size_1*PSF_size_2), (float)0, thrust::plus<float>()) / (PSF_size_1*PSF_size_2);


		////8��Ratio(1:end,1:end)=ImgExp(1:end,1:end)./(ImgEst(1:end,1:end)+Tmp/SNR),and transformed into a complex matrix with zero virtual part;
		Ratio_Complex_ge << <blockNum_12, threadNum_12 >> > (ImgExp, ImgEst, Tmp, SNR, Ratio_Complex, PSF_size_1*PSF_size_2);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "Ratio_Complex_ge Error");

		////9��fft2(Ratio)
		res = hipfftExecC2C(plan, Ratio_Complex, Ratio_Complex, HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS)
		{
			cout << "Ratio_Complex hipfftExecC2C error:" << res << endl;
			system("pause");
			return;
		}

		////10��repmat,Assign Nz times, Ratio_Complex becomes a three-dimensional fftRatio
		fftRatio_ge << <grid, block >> > (Ratio_Complex, fftRatio, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio_ge Error");

		////11��fftRatio.*conj(OTF), Save to fftRatio
		fftRatio_mul_conjOTF << <blockNum_123, threadNum_123 >> > (fftRatio, OTF, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio_mul_conjOTF Error");

		////12��ifft2(       fftRatio.*conj(OTF)       ), Divided by the total number of pixels
		hipfftExecC2C(fftplanfwd, fftRatio, fftRatio, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio hipfftExecC2C Error");
		ifft2_divide << <blockNum_123, threadNum_123 >> > (fftRatio, PSF_size_1*PSF_size_2*PSF_size_3, PSF_size_1*PSF_size_2);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio ifft2_divide Error");

		////13��max(   real(   ifftshift(   ifftshift(     1),   2)   ),   0);
		ifftshift_real_max << <grid, block >> > (fftRatio, float_temp, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "13��fftRatio ifftshift_real_max Error");

		////14��gpuObjRecon = gpuObjRecon.*max(  )
		real_multiply << <blockNum_123, threadNum_123 >> > (gpuObjRecon, float_temp, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon real_multiply Error");

		//cout <<  i <<  endl << endl << endl;
	}
	//The calculation is completed and the value is placed in PSF_1
	check(hipMemcpy(PSF_1, gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3*sizeof(float), hipMemcpyDeviceToHost), "gpuObjRecon to PSF_1 hipMemcpy Error");

	////matlab is from 157-356 rows, total 356-127+1=200 rows. 157-356 columns, total 356-127+1=200 columns
	int line_start = Nxy / 2 - ROISize; int line_end = Nxy / 2 + ROISize - 1; int line_total = line_end - line_start + 1;
	int col_start = Nxy / 2 - ROISize; 	int col_end = Nxy / 2 + ROISize - 1; int col_total = col_end - col_start + 1;


	//output image
	GDALDriver * pDriver = GetGDALDriverManager()->GetDriverByName("ENVI");
	GDALDataset *ds = pDriver->Create(OutFile, col_total, line_total, PSF_size_3, GDT_Float32, NULL);
	if (ds == NULL)
	{
		cout << "Failed to create output file!" << endl;
		system("pause");
		return 0;
	}
	float *ObjRecon_buffer = new float[col_total];
	for (int band = 0; band < PSF_size_3; band++)
	{
		for (int i = 0; i < line_total; i++)//row
		{
			for (int j = 0; j < col_total; j++)//col
			{
				ObjRecon_buffer[j] = PSF_1[band*PSF_size_1*PSF_size_2 + (i + line_start)*PSF_size_2 + j + col_start];
			}
			ds->GetRasterBand(band + 1)->RasterIO(GF_Write, 0, i, col_total, 1, ObjRecon_buffer, col_total, 1, GDT_Float32, 0, 0);
		}
	}
	
	auto time_end = system_clock::now();
	auto duration1 = duration_cast<microseconds>(time_end - time_1);
	float usetime1 = float(duration1.count()) * microseconds::period::num / microseconds::period::den;
	auto duration2 = duration_cast<microseconds>(time_end - time_start);
	float usetime2 = float(duration2.count()) * microseconds::period::num / microseconds::period::den;
	cout << "Excluding data reads and memory memory requests, the data computation part takes time:" << usetime1 << "second" << endl;
	cout << "total time:" << usetime2 << "second" << endl;
	system("pause");
    return 0;
}



