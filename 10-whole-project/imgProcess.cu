#include "hip/hip_runtime.h"
#include"imgProcess.h"
#include"kexinLibs.h"
#include"initANDcheck.h"

#include "reconstructionCUDA.cuh"
#include "templateMatchingCUDA.cuh"

#include "gdal_alg.h";
#include "gdal_priv.h"
#include <gdal.h>

#include <iomanip>
#include <locale>
#include <sstream>
#include <string>


//#include <chrono>//��׼ģ�������ʱ���йص�ͷ�ļ�
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>

//#include "hip/hip_runtime.h"
//#include ""

//#include <assert.h>
//#include <stdio.h>
//#include <stdlib.h>

#include<iostream>
//#include<string>

using namespace std;
//�ع�����   ��Ҫ����  ����ֱ�����ඨ�������ʼ�����Ժ󵥶�дһ����ʼ������
int clockRate = 1.0;
float scale = 1.0 / 4;
int ItN = 10; // ��������
int ROISize = 100;// ROI ��С ���ù�
int BkgMean = 140;// ������ֵ
int SNR = 200;// SNR���ù�
int NxyExt = 0;//�������ķ�Χ����������0�����������
int PSF_size_1 = 512;
int PSF_size_2 = 512;
int PSF_size_3 = 50;
int Nxy = PSF_size_1 + NxyExt * 2; // �Ǹ����� ����ֱ�Ӵ��Դ���
int Nz = PSF_size_3; // ����ֱ���ó��� ���Դ���  Nz = 50

int threadNum_123 = 256;
int blockNum_123 = (PSF_size_1*PSF_size_2*PSF_size_3 - 1) / threadNum_123 + 1;
int threadNum_12 = 256;
int blockNum_12 = (PSF_size_1*PSF_size_2 - 1) / threadNum_12 + 1;
int threadNum_ROI = 256;
int blockNum_ROI = (ROISize * 2 * ROISize * 2 * Nz - 1) / threadNum_ROI + 1;
dim3 block(8, 8, 8);
dim3 grid((PSF_size_1 + block.x - 1) / block.x, (PSF_size_2 + block.y - 1) / block.y, (PSF_size_3 + block.z - 1) / block.z);
dim3 block_sum(32, 32, 1);
dim3 grid_sum((PSF_size_1 + block.x - 1) / block.x, (PSF_size_2 + block.y - 1) / block.y, 1);

int ObjRecon_size = 200 * 200 * 50;


bool Contour_Area(std::vector<cv::Point> contour1, std::vector<cv::Point> contour2)
{
	return cv::contourArea(contour1) > cv::contourArea(contour2);
}

void FishImageProcess::initialize()
{



	//��ȡPSF��δ�ع����ļ�
	string PSF_1_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/PSF_1_zhuanzhi_float.dat";//matlab�б��������float����
	string X31_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/r20210924_2_X31_resize.tif";
	//��ȡ�Ƕȡ���άģ����Ϣ
	string rotationAngleXY_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/rotationAngleXY.dat";//360��double
	string rotationAngleYZ_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/rotationAngleYZ.dat";//31��double
	string template_roXY_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/templateXY.tif";//200*200*360��float������matlab�������ȴ洢������һ�������ٴ�ڶ�������
	string template_roYZ_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/template_roYZ.dat";//200*50*31��float������matlab�������ȴ洢������һ�������ٴ�ڶ�������
	//��ȡ���ڷ�������fixImage
	string fixImage_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/toAffineWithZBB.tif";


	readPSFfromFile(PSF_1_file);
	readRotationAngleFromFile(rotationAngleXY_file, rotationAngleYZ_file);
	readTemplateFromFile(template_roXY_file, template_roYZ_file);
	readFixImageFromFile(fixImage_file);

	initializeFishReg("anatomyList_4bin.txt");

	prepareGPUMemory();
	processPSF();

	return;
}

void FishImageProcess::readPSFfromFile(std::string filename)
{
	cout << "start read PSF1 from file..." << endl;


	FILE *PSF_1_fid = fopen(filename.data(), "rb");
	if (PSF_1_fid == NULL)
	{
		cout << "PSF_1_file open failed!" << endl;
		system("pause");
		return;
	}

	PSF_1 = new float[PSF_size_1*PSF_size_2*PSF_size_3]();
	fread(PSF_1, sizeof(float), PSF_size_1*PSF_size_2*PSF_size_3, PSF_1_fid);

	cout << "read PSF1 done" << endl;

	return;
}


void FishImageProcess::loadImage(unsigned short* imgbuffer)
{
	Img = imgbuffer;

	return;
}

void FishImageProcess::readImageFromFile(std::string filename)
{
	cout << "read: " << filename << endl;
	//Img = readImgFromFile(filename);
	//ʹ��GDAL��ȡtif��ʹ�õ���matlab�ز����õ�����
	GDALAllRegister(); OGRRegisterAll();
	//����֧������·��
	CPLSetConfigOption("GDAL_FILENAME_IS_UTF8", "NO");
	CPLSetConfigOption("SHAPE_ENCODING", "");
	GDALDataset* poSrcDS = (GDALDataset*)GDALOpen(filename.data(), GA_ReadOnly);
	if (poSrcDS == NULL)
	{
		cout << "image file open failed!" << endl;
		return;
	}
	int wheight = poSrcDS->GetRasterYSize();//��
	int wwidth = poSrcDS->GetRasterXSize();//��
	int bandNum = poSrcDS->GetRasterCount();//������
	GDALDataType dataType = poSrcDS->GetRasterBand(1)->GetRasterDataType();//����

	Img = new unsigned short[PSF_size_1*PSF_size_2]();   //ͼ�������������
	for (int i = 0; i < bandNum; i++)
	{
		////////////////////////////////////��ȡ��ʼ�У�ʼ�У�������������ָ�룬��������������������
		poSrcDS->GetRasterBand(i + 1)->RasterIO(GF_Read, 0, 0, wwidth, wheight, Img, PSF_size_1, PSF_size_2, dataType, 0, 0);
	}
	GDALClose(poSrcDS);

	//check
	//cout << Img[512 * 256 + 256] << endl;

	//cout << "read image file done" << endl;

	return;
}

void FishImageProcess::readImageFromCamera(std::string filename)
{
	//������
	return;
}

void FishImageProcess::readTemplateFromFile(std::string filenameXY, std::string filenameYZ)
{
	cout << "start read templates...." << endl;
	//FILE * template_roXY_fid = fopen(filenameXY.data(), "rb");
	//if (template_roXY_fid == NULL)
	//{
	//	cout << filenameXY << " open failed!" << endl;
	//	system("pause");
	//	return;
	//}
	int template_roXY_size = 200 * 200 * 360;
	template_roXY = new float[template_roXY_size];
	//fread(template_roXY, sizeof(float), template_roXY_size, template_roXY_fid);
	//fclose(template_roXY_fid);

	template_roXY = readImgFromFile(filenameXY);

	cout << "XY template read successful" << endl;



	FILE * template_roYZ_fid = fopen(filenameYZ.data(), "rb");
	if (template_roYZ_fid == NULL)
	{
		cout << filenameYZ << " open failed!" << endl;
		system("pause");
		return;
	}
	int template_roYZ_size = 200 * 50 * 31;
	template_roYZ = new float[template_roYZ_size];
	fread(template_roYZ, sizeof(float), template_roYZ_size, template_roYZ_fid);
	fclose(template_roYZ_fid);
	cout << "YZ template read successful" << endl;

	cout << "read template done" << endl;

	return;
}

void FishImageProcess::readRotationAngleFromFile(std::string filenameAngleXY, std::string filenameAngleYZ)
{
	cout << "start read rotation angle from file.." << endl;
	FILE * rotationAngleXY_fid = fopen(filenameAngleXY.data(), "rb");
	if (rotationAngleXY_fid == NULL)
	{
		cout << filenameAngleXY << " open failed!" << endl;
		system("pause");
		return;
	}
	int rotationAngleXY_size = 360;
	rotationAngleXY = new double[rotationAngleXY_size];
	fread(rotationAngleXY, sizeof(double), rotationAngleXY_size, rotationAngleXY_fid);
	fclose(rotationAngleXY_fid);
	cout << "read XY rotation angle successfule" << endl;

	FILE * rotationAngleYZ_fid = fopen(filenameAngleYZ.data(), "rb");
	if (rotationAngleYZ_fid == NULL)
	{
		cout << filenameAngleYZ << " open failed!" << endl;
		system("pause");
		return;
	}
	int rotationAngleYZ_size = 31;
	rotationAngleYZ = new double[rotationAngleYZ_size];
	fread(rotationAngleYZ, sizeof(double), rotationAngleYZ_size, rotationAngleYZ_fid);
	fclose(rotationAngleYZ_fid);

	cout << "read YZ rotation angle successfule" << endl;
	cout << "read rotation angle done" << endl;

	return;
}

void FishImageProcess::readFixImageFromFile(std::string filename)
{
	float* fixImage = readImgFromFile(filename);
	//cout << nImgSizeX << "   " << nImgSizeY << "   " << bandcount << endl;
	fixtensor = torch::from_blob(fixImage,
		{int(imgSizeAfterCrop_Z), int(imgSizeAfterCrop_Y), int(imgSizeAfterCrop_X) }).toType(torch::kFloat32);
	fixtensor = normalizeTensor(fixtensor);
	cout << "read fix image and convert to normalize tensor" << endl;
	//torch::Device device(torch::kCUDA);
	fixtensor.to(device);
	cout << "copy fix tensor to CUDA" << endl;

	//���紦��ǰ����ͼ���������ǰ��һ��
	cout << "warm up..." << endl;
	for (int i = 0; i < 10; i++)
	{
		model.forward({ fixtensor.to(device),fixtensor.to(device) }).toTensor();
		cout << i << "  " ;
	}
	cout << "model process done" << endl;

	return;
}

void FishImageProcess::initializeFishReg(std::string filename)
{
	FishReg.initialize(filename);

	vector<float> Fix2ZBBAM{ 0.985154,	0.0184487, -0.00942914,
	-0.0166061,	1.13246, -0.102937,
	0.0196408, -0.0078765,	1.25844,
	0.522241, -6.91866, -11.7296 };
	FishReg.getZBB2FixAffineMatrix(Fix2ZBBAM);

	return;
}


void FishImageProcess::prepareGPUMemory()
{
	/*-------׼�������������ڴ桢�Դ棬fft���--------------------------*/
	cout << "start malloc memory..." << endl;

	const int rank = 2;//ά��
	int n[rank] = { PSF_size_1, PSF_size_2 };//n*m
	int *inembed = n;//���������size
	int istride = 1;//����������������Ϊ1
	int idist = n[0] * n[1];//1��������ڴ��С
	int *onembed = n;//�����һ�������size
	int ostride = 1;//ÿ��DFT������������Ϊ1
	int odist = n[0] * n[1];//�����һ��������ڶ�������ľ��룬�������������Ԫ�صľ���
	int batch = PSF_size_3;//�������������

	//��ʼ�����
	hipfftPlanMany(&fftplanfwd, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);

	//�ع�
	check(hipMalloc((void**)&PSF_1_gpu, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float)), "PSF_1_gpu hipMalloc Error");
	check(hipMalloc((void**)&PSF_1_gpu_Complex, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "PSF_1_gpu_Complex hipMalloc Error");
	check(hipMalloc((void**)&OTF, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "OTF hipMalloc Error");
	check(hipMalloc((void**)&ImgEst, PSF_size_1*PSF_size_2 * sizeof(float)), "ImgEst hipMalloc Error");
	check(hipMalloc((void**)&Ratio, PSF_size_1*PSF_size_2 * sizeof(float)), "Ratio hipMalloc Error");
	check(hipMalloc((void**)&gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float)), "Ratio hipMalloc Error");
	check(hipMalloc((void**)&gpuObjRecROI, ROISize * 2 * ROISize * 2 * PSF_size_3 * sizeof(float)), "gpuObjRecROI hipMalloc Error");
	check(hipMalloc((void**)&Img_gpu, PSF_size_1*PSF_size_2 * sizeof(unsigned short)), "Img_gpu hipMalloc Error");
	check(hipMalloc((void**)&ImgExp, PSF_size_1*PSF_size_2 * sizeof(float)), "ImgExp hipMalloc Error");
	check(hipMalloc((void**)&gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "gpuObjRecon_Complex hipMalloc Error");
	check(hipMalloc((void**)&float_temp, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float)), "float_temp hipMalloc Error");
	check(hipMalloc((void**)&Ratio_Complex, PSF_size_1*PSF_size_2 * sizeof(hipfftComplex)), "Ratio_Complex hipMalloc Error");
	check(hipMalloc((void**)&fftRatio, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "fftRatio hipMalloc Error");


	//crop 512*512 to 200*200
	cpuObjRecon = new float[PSF_size_1*PSF_size_2*PSF_size_3]();
	cpuObjRecon_crop = new float[200 * 200 * PSF_size_3];
	check1(hipMalloc((void**)&gpuObjRecon_crop, sizeof(float)*ObjRecon_size), "gpuObjRecon_crop hipMalloc Error", __FILE__, __LINE__);

	//XY��ת
	check1(hipMalloc((void**)&image2D_XY_gpu, sizeof(float) * 200 * 200), "image2D_XY_gpu hipMalloc Error", __FILE__, __LINE__);
	check1(hipMalloc((void**)&img2DBW_XY_gpu, sizeof(float) * 200 * 200), "img2DBW_XY_gpu hipMalloc Error", __FILE__, __LINE__);
	check1(hipMalloc((void**)&template_roXY_gpu, sizeof(float) * template_roXY_size), "template_roXY_gpu hipMalloc Error", __FILE__, __LINE__);
	check(hipMemcpy(template_roXY_gpu, template_roXY, sizeof(float)*template_roXY_size, hipMemcpyHostToDevice), "template_roXY_gpu hipMemcpy Error");
	check1(hipMalloc((void**)&err_XY_gpu, sizeof(double) * rotationAngleXY_size), "err_XY_gpu hipMalloc Error", __FILE__, __LINE__);
	check1(hipMalloc((void**)&imageRotated3D_gpu, sizeof(float) * ObjRecon_size), "imageRotated3D_gpu hipMalloc Error", __FILE__, __LINE__);
	NppiSize Input_Size;//����ͼ���������
	Input_Size.width = 200;
	Input_Size.height = 200;
	check1(hipMalloc((void**)&input_image_gpu, sizeof(float) * 200 * 200), "input_image_gpu hipMalloc Error", __FILE__, __LINE__);
	check1(hipMalloc((void**)&output_image_gpu, sizeof(float)* 200 * 200), "output_image_gpu hipMalloc Error", __FILE__, __LINE__);


	//crop
	cpuObjRotation_crop = new float[200 * 200 * 50];
	check1(hipMalloc((void**)&ObjCropRed_gpu, sizeof(float)*imgSizeAfterCrop_X*imgSizeAfterCrop_Y*imgSizeAfterCrop_Z),
		"ObjReconRed_gpu hipMalloc Error", __FILE__, __LINE__);
	check1(hipMalloc((void**)&imageRotated2D_XY_GPU, sizeof(float) * 200 * 200 * 1), "imageRotated2D_XY hipMalloc Error", __FILE__, __LINE__);
	check1(hipMalloc((void**)&imageRotated2D_XY_BW_GPU, sizeof(float) * 200 * 200 * 1), "imageRotated2D_XY_BW_GPU hipMalloc Error", __FILE__, __LINE__);
	imageRotated2D_XY_BW_CPU = new float[200 * 200]();



	cout << "prepare memory done" << endl;

	return;
}

void FishImageProcess::processPSF()
{
	cout << "��ʼ���ʹ���PSF...." << endl;
	check(hipMemcpy(PSF_1_gpu, PSF_1, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float), hipMemcpyHostToDevice), "PSF_1_gpu hipMemcpy Error");
	//ת���ɸ������鲿��0
	Zhuan_Complex_kernel << <blockNum_123, threadNum_123 >> > (PSF_1_gpu, PSF_1_gpu_Complex, PSF_size_1*PSF_size_2*PSF_size_3);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "PSF_1_gpu Zhuan_Complex_kernel Error");
	//*----ʹ��hipfftPlanMany�ķ�������������άfft---------------------*/
	hipfftExecC2C(fftplanfwd, PSF_1_gpu_Complex, OTF, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "PSF_1_gpu_Complex hipfftExecC2C Error");

	//ImgEst��ֵΪ0��Ratio��ֵΪ1
	initial_kernel_1 << <blockNum_12, threadNum_12 >> > (ImgEst, Ratio, PSF_size_1*PSF_size_2);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "initial_kernel_1 Error");
	gpuObjRecon_fuzhi << <blockNum_123, threadNum_123 >> > (gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "gpuObjRecon_fuzhi Error");
	//gpuObjRecROI��ֵΪ1
	initial_kernel_3 << <blockNum_ROI, threadNum_ROI >> > (gpuObjRecROI, ROISize * 2 * ROISize * 2 * Nz);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "initial_kernel_3 Error");

	cout << "��ʼ���ʹ���PSF....down" << endl;

	return;
}

void FishImageProcess::reconImage()
{
	////*----1��ʹ��hipfftPlan2d�ķ������ж�άfft----------*/
	hipfftHandle plan;
	//hipfftResult res;
	hipfftResult res= hipfftPlan2d(&plan, PSF_size_1, PSF_size_2, HIPFFT_C2C);  
	


	check(hipMemcpy(Img_gpu, Img, PSF_size_1*PSF_size_2 * sizeof(unsigned short), hipMemcpyHostToDevice), "Img_gpu hipMemcpy Error");
	//��ȥ������ֵ���������float���͵�����ImgExp��
	ImgExp_ge << <blockNum_12, threadNum_12 >> > (Img_gpu, BkgMean, ImgExp, PSF_size_1*PSF_size_2);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "ImgExp_ge Error");


	//Ratio��gpuObjRecon��Ԫ�ض���ֵ1
	Ratio_fuzhi << <blockNum_12, threadNum_12 >> > (Ratio, PSF_size_1*PSF_size_2);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "Ratio_fuzhi Error");
	gpuObjRecon_fuzhi << <blockNum_123, threadNum_123 >> > (gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "gpuObjRecon_fuzhi Error");




	//��ʼѭ������
	for (int i = 0; i < ItN; i++)
	{
		////1��fft2(gpuObjRecon)
		Zhuan_Complex_kernel << <blockNum_123, threadNum_123 >> > (gpuObjRecon, gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "Zhuan_Complex_kernel Error");
		hipfftExecC2C(fftplanfwd, gpuObjRecon_Complex, gpuObjRecon_Complex, HIPFFT_FORWARD);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon_Complex hipfftExecC2C Error");

		////2��OTF.*fft2(gpuObjRecon_Complex)���������gpuObjRecon_Complex��
		OTF_mul_gpuObjRecon_Complex << <blockNum_123, threadNum_123 >> > (OTF, gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "2��OTF.*fft2(gpuObjRecon_Complex) Error");

		////3��ifft2(OTF.*fft2(gpuObjRecon))����任��Ҫ���������ظ���
		hipfftExecC2C(fftplanfwd, gpuObjRecon_Complex, gpuObjRecon_Complex, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon_Complex hipfftExecC2C cufft_inverse Error");
		////4������������������ȷ
		ifft2_divide << <blockNum_123, threadNum_123 >> > (gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3, PSF_size_1*PSF_size_2);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon_Complex ifft2_divide Error");

		/*----�ڶ���gpuObjRecon_Complex��ʵ����ȷ���鲿����ȷ������Ĵ���ֻ����gpuObjRecon_Complex��ʵ����û�õ��鲿----------*/

		////5��ifftshift + real + max(,0)�����ʵ������float_temp��С��0�ĸ�ֵ0
		ifftshift_real_max << <grid, block >> > (gpuObjRecon_Complex, float_temp, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "5��gpuObjRecon_Complex ifftshift_real_max Error");

		////6��sum( ,3)���ڵ���ά�ϼ���ͣ�����PSF_size_1��PSF_size_2�еľ���ImgEst
		float_temp_sum << <grid_sum, block_sum >> > (float_temp, ImgEst, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "float_temp_sum Error");

		////7��Tmp=mean(   ImgEst(:)   );
		thrust::device_ptr<float> dev_ptr(ImgEst);
		float Tmp = thrust::reduce(dev_ptr, dev_ptr + size_t(PSF_size_1*PSF_size_2), (float)0, thrust::plus<float>()) / (PSF_size_1*PSF_size_2);

		////8��Ratio(1:end,1:end)=ImgExp(1:end,1:end)./(ImgEst(1:end,1:end)+Tmp/SNR)����ת�ɸ��������鲿Ϊ��;
		Ratio_Complex_ge << <blockNum_12, threadNum_12 >> > (ImgExp, ImgEst, Tmp, SNR, Ratio_Complex, PSF_size_1*PSF_size_2);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "Ratio_Complex_ge Error");

		////9��fft2(Ratio)
		res = hipfftExecC2C(plan, Ratio_Complex, Ratio_Complex, HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS)
		{
			cout << "Ratio_Complex hipfftExecC2C error:" << res << endl;
			system("pause");
			return;
		}

		////10��repmat����ֵNz�飬Ratio_Complex�����ά��fftRatio
		fftRatio_ge << <grid, block >> > (Ratio_Complex, fftRatio, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio_ge Error");


		////11��fftRatio.*conj(OTF)���浽fftRatio��
		fftRatio_mul_conjOTF << <blockNum_123, threadNum_123 >> > (fftRatio, OTF, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio_mul_conjOTF Error");


		////12��ifft2(       fftRatio.*conj(OTF)       )�������������ظ���
		hipfftExecC2C(fftplanfwd, fftRatio, fftRatio, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio hipfftExecC2C Error");
		ifft2_divide << <blockNum_123, threadNum_123 >> > (fftRatio, PSF_size_1*PSF_size_2*PSF_size_3, PSF_size_1*PSF_size_2);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio ifft2_divide Error");

		////13��max(   real(   ifftshift(   ifftshift(     1),   2)   ),   0);
		ifftshift_real_max << <grid, block >> > (fftRatio, float_temp, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "13��fftRatio ifftshift_real_max Error");

		////14��gpuObjRecon = gpuObjRecon.*max(  )
		real_multiply << <blockNum_123, threadNum_123 >> > (gpuObjRecon, float_temp, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon real_multiply Error");

	}

	hipfftDestroy(plan);   //�����һ�䣬hipfftPlan2d���ٲ����Դ�й¶

	if (DEBUG)
	{
		cout << "�ع����" << endl;
	}
	return;
}

void FishImageProcess::cropReconImage()
{
	//������ϣ�ȡֵ����cpuObjRecon
	check(hipMemcpy(cpuObjRecon, gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float), hipMemcpyDeviceToHost), "gpuObjRecon to cpuObjRecon hipMemcpy Error");
	/*  ���crop����CPU����ɵģ��ĳ���GPU�����   */
	//gpuObjRecon crop ��200*200*50
	//����CPU��crop�󴫵�GPU
	////matlab���Ǵ�157��356�У��ܹ�356-157+1=200�С�157-356�У��ܹ�356-157+1=200�С�
	int line_start = Nxy / 2 - ROISize; int line_end = Nxy / 2 + ROISize - 1; int line_total = line_end - line_start + 1;
	int col_start = Nxy / 2 - ROISize; 	int col_end = Nxy / 2 + ROISize - 1; int col_total = col_end - col_start + 1;
	if (DEBUG)
	{
		cout << "line_start: " << line_start << endl;
		cout << "line_end: " << line_end << endl;
		cout << "line_total: " << line_total << endl;
		cout << "col_start: " << col_start << endl;
		cout << "col_end: " << col_end << endl;
		cout << "col_total: " << col_total << endl;
	}

	for (int band = 0; band < PSF_size_3; band++)
	{
		for (int i = 0; i < line_total; i++)//��ѭ��
		{
			for (int j = 0; j < col_total; j++)//��ѭ��
			{
				cpuObjRecon_crop[band * 200 * 200 + i * 200 + j] = cpuObjRecon[band*PSF_size_1*PSF_size_2 + (i + line_start)*PSF_size_2 + j + col_start];
			}
		}
	}
	//float *gpuObjRecon_crop;   //�洢crop���ObjRecon
	check(hipMemcpy(gpuObjRecon_crop, cpuObjRecon_crop, sizeof(float)*ObjRecon_size, hipMemcpyHostToDevice), "gpuObjRecon_crop hipMemcpy Error");

	if (DEBUG)
	{
		//cropReconImage_kernel << <blockNum_123, threadNum_123 >> > (gpuObjRecon, gpuObjRecon_crop);
		cout << "crop�ع��������ݲ�copy��GPU" << endl;
	}
	return;
}


void FishImageProcess::matchingANDrotationXY()
{
	///*   XYƽ���ģ��ƥ�����ת   */
	//if (DEBUG)
	//{
	//	cout << "start XY 2D template matching..." << endl;
	//}
	////GET MIP
	dim3 block_1(32, 32, 1);
	dim3 grid_1((200 + block_1.x - 1) / block_1.x, (200 + block_1.y - 1) / block_1.y, 1);
	kernel_1 << <grid_1, block_1 >> > (gpuObjRecon_crop, 200, 200, image2D_XY_gpu);   
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_1 Error");

	//thrust::device_ptr<float> dev_ptr(image2D_XY_gpu);
	//double image2D_XY_mean = thrust::reduce(dev_ptr, dev_ptr + size_t(200 * 200), (float)0, thrust::plus<float>()) / (200 * 200);
	//if (DEBUG)
	//{
	//	cout << "image2D_XY_mean: " << image2D_XY_mean << endl;
	//}

	//int threadNum_2 = 256;
	//int blockNum_2 = (200 * 200 - 1) / threadNum_2 + 1;
	//kernel_2 << <blockNum_2, threadNum_2 >> > (image2D_XY_gpu, 200 * 200, image2D_XY_mean, img2DBW_XY_gpu);
	//hipDeviceSynchronize();
	//checkGPUStatus(hipGetLastError(), "kernel_2 Error");

	//int threadNum_3 = 256;
	//int blockNum_3 = (rotationAngleXY_size - 1) / threadNum_2 + 1;
	//kernel_3 << <blockNum_3, threadNum_3 >> > (template_roXY_gpu, img2DBW_XY_gpu, rotationAngleXY_size, err_XY_gpu);
	//hipDeviceSynchronize();
	//checkGPUStatus(hipGetLastError(), "kernel_3 Error");


	////��err_XY_gpu����Сֵ
	//double *err_XY = new double[rotationAngleXY_size];
	//check(hipMemcpy(err_XY, err_XY_gpu, sizeof(double)*rotationAngleXY_size, hipMemcpyDeviceToHost), "err_XY hipMemcpy Error");
	//double err_XY_min = DBL_MAX;
	//int idx;  //�ҵ���Сֵ��Ӧ������
	//for (int i = 0; i < rotationAngleXY_size; i++)
	//{
	//	//cout << i << "   " << err_XY[i] << endl;
	//	if (err_XY[i] < err_XY_min)
	//	{
	//		err_XY_min = err_XY[i];
	//		idx = i;
	//	}
	//}
	////cout << "err_XY_min: " << err_XY_min << endl;
	////cout << "rotation XY idx: " << idx << endl;
	////��һ����ת
	//rotationAngleX = -rotationAngleXY[idx];
	//rotationAngleY = 0;
	//rotationAngleX = 166;
	ObjRecon_imrotate3_gpu(gpuObjRecon_crop, -rotationAngleX, imageRotated3D_gpu);

	//std::cout << "imrotate3 rotationAngleX: " << -rotationAngleX << std::endl;

	if (DEBUG)
	{
		cout << "XY 2D templaet matching and rotation done" << endl;
	}

	return;
}

void FishImageProcess::ObjRecon_imrotate3_gpu(float *ObjRecon_gpu, double nAngle, float *imageRotated3D_gpu)
{
	NppiSize Input_Size;//����ͼ���������
	Input_Size.width = 200;
	Input_Size.height = 200;

	/* �����Դ棬��ԭͼ�����Դ� */
	int nSrcPitchCUDA = Input_Size.width * sizeof(float);//ÿ����ռ���ֽ���

	/* ������ת�󳤿� */
	NppiRect Input_ROI;//�ض��������ת���൱�ڲü�ͼ���һ�飬���β���ȫ��ͼ��
	Input_ROI.x = Input_ROI.y = 0;
	Input_ROI.width = Input_Size.width;
	Input_ROI.height = Input_Size.height;
	double aBoundingBox[2][2];
	nppiGetRotateBound(Input_ROI, aBoundingBox, nAngle, 0, 0);
	int bb = ((int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0])) - Input_ROI.width) / 2 + aBoundingBox[0][0];//��ʼ��
	int cc = ((int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1])) - Input_ROI.height) / 2 + aBoundingBox[0][1];//��ʼ��
	aBoundingBox[0][0] = bb;//��ʼ��
	aBoundingBox[0][1] = cc;//��ʼ��
	NppiSize Output_Size;
	Output_Size.width = Input_Size.width;
	Output_Size.height = Input_Size.height;


	/* ת�����ͼ���Դ���� */
	int nDstPitchCUDA = Output_Size.width * sizeof(float);


	//�������Ȥ���Ĵ�С���൱�ڰ����ͼ���ٲü�һ�飬Ӧ������������û���ԣ��������
	NppiRect Output_ROI;
	Output_ROI.x = 0; Output_ROI.y = 0;
	Output_ROI.width = Input_Size.width;
	Output_ROI.height = Input_Size.height;

	for (int i = 0; i < 50; i++)
	{
		check(hipMemcpy(input_image_gpu, ObjRecon_gpu + Input_Size.width*Input_Size.height * i, sizeof(float)*Input_Size.width*Input_Size.height, hipMemcpyDeviceToDevice), "input_image_gpu hipMemcpy Error");
		/* ������ת */
		NppStatus nppRet = nppiRotate_32f_C1R(input_image_gpu, Input_Size, nSrcPitchCUDA, Input_ROI,
			output_image_gpu, nDstPitchCUDA, Output_ROI, nAngle, -aBoundingBox[0][0], -aBoundingBox[0][1], NPPI_INTER_NN);
		assert(nppRet == NPP_NO_ERROR);
		check(hipMemcpy(imageRotated3D_gpu + Input_Size.width*Input_Size.height * i, output_image_gpu, sizeof(float) * Output_Size.width*Output_Size.height, hipMemcpyDeviceToDevice), "output_image hipMemcpy Error");
	}

	return;
}




void FishImageProcess::cropRotatedImage()
{
	if (DEBUG)
	{
		cout << "start crop rotation image..." << endl;
	}
	//	//����imageRotated3D_gpu�ľ�ֵ
	thrust::device_ptr<float> dev_ptr2(imageRotated3D_gpu);
	double imageRotated3D_x_mean = thrust::reduce(dev_ptr2, dev_ptr2 + size_t(ObjRecon_size), (float)0, thrust::plus<float>()) / (ObjRecon_size)+4;

	check(hipMemcpy(cpuObjRotation_crop, imageRotated3D_gpu, sizeof(float)*ObjRecon_size, hipMemcpyDeviceToHost), "ObjRecon hipMemcpy Error");

	////crop
	//int *idx_2 = new int[ObjRecon_size]();//imageRotated3D_x���ھ�ֵ������
	//int idx_2_size = 0;
	//for (int i = 0; i < ObjRecon_size; i++)
	//{
	//	if (cpuObjRotation_crop[i] > imageRotated3D_x_mean)
	//	{
	//		idx_2_size++;
	//		idx_2[idx_2_size] = i;
	//	}
	//}
	////idx_2�����ÿһ������ת����imageRotated3D_x��200��*200��*50���Σ����кţ��кţ����κ�
	//float *x = new float[idx_2_size]; float x_sum = 0;
	//float *y = new float[idx_2_size]; float y_sum = 0;
	//float *z = new float[idx_2_size]; float z_sum = 0;
	//for (int i = 0; i < idx_2_size; i++)
	//{
	//	z[i] = idx_2[i] / (200 * 200);
	//	int yushu = idx_2[i] % (200 * 200);
	//	x[i] = yushu / 200;
	//	y[i] = yushu % 200;

	//	x_sum += x[i];
	//	y_sum += y[i];
	//	z_sum += z[i];
	//}
	//int CentroID[3];
	//CentroID[0] = int(x_sum / idx_2_size + 0.5);
	//CentroID[1] = int(y_sum / idx_2_size + 0.5);
	//CentroID[2] = int(z_sum / idx_2_size + 0.5);
	//if (DEBUG)
	//{
	//	cout << "CentroID: " << CentroID[0] << "   " << CentroID[1] << "  " << CentroID[2] << endl;
	//}
	////CentroID������matlab����[89,91,24]���Ҽ������[86,91,24],x���3����npp��ת��matlab�Ľ���������ɵģ����Ҳ��������������

	//// ��������������Χ����������������matlab������Ҫ��ȥ1
	//// �з�Χ����CentroID(0)-61��CentroID(0)+33�� ���з�Χ����CentroID(2)-38��CentroID(2)+37�������еĲ���
	////int XObj = CentroID[0] + 33 - (CentroID[0] - 61) + 1;//��
	////int	YObj = CentroID[2] + 37 - (CentroID[2] - 38) + 1;//��
	////int	ZObj = 50;//����

	//if (CentroID[0] < 61 || CentroID[1] < 38 || CentroID[0]>167 || CentroID[1]>163)
	//{
	//	cout << "centroID error!!!" << endl;
	//	return;
	//}

	//cropPoint = cv::Point3d(CentroID[0] - 61, CentroID[1] - 38, 0);




	
	dim3 block_1(32, 32, 1);
	dim3 grid_1((200 + block_1.x - 1) / block_1.x, (200 + block_1.y - 1) / block_1.y, 1);
	kernel_1 << <grid_1, block_1 >> > (imageRotated3D_gpu, 200, 200, imageRotated2D_XY_GPU);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_1 Error");

	thrust::device_ptr<float> dev_ptr(imageRotated2D_XY_GPU);
	double imageRotated2D_XY_mean = thrust::reduce(dev_ptr, dev_ptr + size_t(200 * 200), (float)0, thrust::plus<float>()) / (200 * 200);

	cout << "imageRotated2D_XY_mean: " << imageRotated2D_XY_mean << endl;

	
	int threadNum_2 = 256;
	int blockNum_2 = (200 * 200 - 1) / threadNum_2 + 1;
	kernel_2 << <blockNum_2, threadNum_2 >> > (imageRotated2D_XY_GPU, 200 * 200, imageRotated2D_XY_mean, imageRotated2D_XY_BW_GPU);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_2 Error");

	check(hipMemcpy(imageRotated2D_XY_BW_CPU, imageRotated2D_XY_BW_GPU, sizeof(float) * 200 * 200, hipMemcpyDeviceToHost), "ObjRecon hipMemcpy Error");

	cv::Mat temp(200, 200, CV_32FC1, imageRotated2D_XY_BW_CPU);
	cv::Mat temp2 = temp.clone();
	temp.convertTo(temp, CV_8UC1);
	std::vector<std::vector<cv::Point>> contours;
	std::vector<cv::Vec4i> hierarchy;
	cv::findContours(temp, contours, hierarchy, cv::RETR_EXTERNAL, cv::CHAIN_APPROX_NONE, cv::Point(0, 0));

	if (contours.size() == 0)
	{
		cout << "no fish detect...." << endl;
		return;
	}

	cv::Rect rect;
	sort(contours.begin(), contours.end(), Contour_Area);
	rect = cv::boundingRect(contours[0]);
	cv::rectangle(temp2, rect, cv::Scalar(128), 2);


	int CentroID[3];
	CentroID[0] = rect.tl().x;
	CentroID[1] = rect.tl().y;
	CentroID[2] = 0;

	cout << "CentroID[0]:" << CentroID[0] << "  CentroID[1]:" << CentroID[1] << " CentroID[2]:" << CentroID[2] << endl;

	if (CentroID[0] + 95 > 200 || CentroID[1] + 76 > 200 || CentroID[0] - 10 < 0)
	{
		cout << "centroID error!!!" << endl;
		return;
	}

	cropPoint = cv::Point3d(CentroID[0] - 10, CentroID[1], 0);


	dim3 block_10(8, 8, 8);
	dim3 grid_10((imgSizeAfterCrop_X + block_10.x - 1) / block_10.x, (imgSizeAfterCrop_Y + block_10.y - 1) / block_10.y, (imgSizeAfterCrop_Z + block_10.z - 1) / block_10.z);
	//__global__ void kernel_10(float *imageRotated3D_gpu, float *ObjReconRed_gpu, int XObj, int YObj, int ZObj, int CentroID0, int CentroID2)
	//kernel_10 << <grid_10, block_10 >> > (imageRotated3D_gpu, ObjCropRed_gpu, imgSizeAfterCrop_X, imgSizeAfterCrop_Y, imgSizeAfterCrop_Z, CentroID[0], CentroID[1]);
	kernel_11 << <grid_10, block_10 >> > (imageRotated3D_gpu, ObjCropRed_gpu, imgSizeAfterCrop_X, imgSizeAfterCrop_Y, imgSizeAfterCrop_Z, CentroID[0] - 10, CentroID[1]);

	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_10 Error");

	//delete[] idx_2;
	//delete[] x;
	//delete[] y;
	//delete[] z;

	if (DEBUG)
	{
		cout << "crop���" << endl;
	}

	return;
}


void FishImageProcess::libtorchModelProcess()
{
	//libtorch
	//convert image to tensor
	torch::Tensor movingtensor;


	//float* temp1 = new float[50 * 95 * 76]();
	//hipMemcpy(temp1, ObjCropRed_gpu, sizeof(float) * 50 * 95 * 76, hipMemcpyDeviceToHost);
	//movingtensor = torch::from_blob(temp1,
	//{ int(imgSizeAfterCrop_Z), int(imgSizeAfterCrop_Y), int(imgSizeAfterCrop_X) }).toType(torch::kFloat32);

	movingtensor = torch::from_blob(ObjCropRed_gpu,
		{ int(imgSizeAfterCrop_Z), int(imgSizeAfterCrop_Y), int(imgSizeAfterCrop_X) }, torch::kCUDA).toType(torch::kFloat32);
	movingtensor = normalizeTensor(movingtensor);
	if (DEBUG)
	{
		cout << movingtensor.sizes() << endl;
		cout << fixtensor.sizes() << endl;
	}
	//cout << "1111" << endl;
	auto output = model.forward({ movingtensor.to(device),fixtensor.to(device) }).toTensor();
	//auto output = model.forward({ movingtensor,fixtensor }).toTensor();
	//cout << "2222" << endl;
	Moving2FixAM = rescaleAffineMatrix(output);


	if (DEBUG)
	{
		cout << Moving2FixAM.size() << endl;
		for (int aa = 0; aa < Moving2FixAM.size(); aa++)
		{
			cout << Moving2FixAM[aa] << "   ";
		}
	}

	return;
}


std::vector<cv::Point2f> FishImageProcess::ZBB2FishTransform(cv::Rect roi)
{

	std::vector<cv::Point2f> regionInFish;


	FishReg.getRegionFromUser(roi);
	//��rotation/crop/affine����������л�ȡ����
	FishReg.getRotationMatrix(-rotationAngleX, rotationAngleY);
	FishReg.getCropPoint(cropPoint);
	FishReg.getFix2MovingAffineMatrix(Moving2FixAM);

	////cout << endl << "1111" << endl;
	////����ת��
	regionInFish = FishReg.ZBB2FishTransform();


	//std::cout << "getRotationMatrix rotationAngleX: " << -rotationAngleX << std::endl;

	
	FishReg.clear();

	return regionInFish;
}




void FishImageProcess::clear()
{
	
	return;
}

void FishImageProcess::freeMemory()
{
	cout << "free cuda memory..." << endl;

	hipFree(PSF_1_gpu);
	hipFree(PSF_1_gpu_Complex);
	hipFree(OTF);
	hipFree(ImgEst);
	hipFree(Ratio);
	hipFree(gpuObjRecon);
	hipFree(gpuObjRecROI);
	hipFree(Img_gpu);
	hipFree(ImgExp);
	hipFree(gpuObjRecon_Complex);
	hipFree(float_temp);
	hipFree(Ratio_Complex);
	hipFree(fftRatio);
	hipFree(gpuObjRecon_crop);
	hipFree(input_image_gpu);
	hipFree(output_image_gpu);
	hipFree(imageRotated2D_XY_GPU);
	hipFree(imageRotated2D_XY_BW_GPU);


	cout << "done" << endl;
	
	cout << "free cpu memory..." << endl;

	delete[] cpuObjRecon;
	delete[] cpuObjRecon_crop;
	delete[] imageRotated2D_XY_BW_CPU;

	cout << "done" << endl;
	return;
}

FishImageProcess::FishImageProcess(const std::string& model_path) :device(torch::kCUDA)
{
	// is CUDA avaliabel??
	//torch::DeviceType device_type;
	if (torch::cuda::is_available())
	{
		device = torch::kCUDA;
		std::cout << "cuda available" << std::endl;
	}
	else
	{
		device = torch::kCPU;
		std::cout << "cuda not avaliable" << std::endl;
	}


	torch::Tensor tensor1 = torch::eye(3); // (A) tensor-cpu
	torch::Tensor tensor2 = torch::eye(3, device); // (B) tensor-cuda
	std::cout << tensor1 << std::endl;
	std::cout << tensor2 << std::endl;


	try
	{
		model = torch::jit::load(model_path);
	}
	catch (const c10::Error& e)
	{
		std::cerr << "Error loading the model!\n";
		std::exit(EXIT_FAILURE);
	}

	//device = torch::kCUDA;
	model.eval();
	model.to(device);
	std::cout << "load model success" << std::endl;
}