#include "hip/hip_runtime.h"
#include"imgProcess.h"
#include"kexinLibs.h"
//#include"kexinLibs.cpp"
#include"initANDcheck.h"
//#include"initANDcheck.cu"
//#include"header.cuh"
#include "reconstructionCUDA.cuh"
#include "templateMatchingCUDA.cuh"

#include "gdal_alg.h";
#include "gdal_priv.h"
#include <gdal.h>

#include <iomanip>
#include <locale>
#include <sstream>
#include <string>


//#include <chrono>//��׼ģ�������ʱ���йص�ͷ�ļ�
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>

//#include "hip/hip_runtime.h"
//#include ""

//#include <assert.h>
//#include <stdio.h>
//#include <stdlib.h>

#include<iostream>
//#include<string>

using namespace std;
//�ع�����   ��Ҫ����  ����ֱ�����ඨ�������ʼ�����Ժ󵥶�дһ����ʼ������
int clockRate = 1.0;
float scale = 1.0 / 4;
int ItN = 10; // ��������
int ROISize = 100;// ROI ��С ���ù�
int BkgMean = 140;// ������ֵ
int SNR = 200;// SNR���ù�
int NxyExt = 0;//�������ķ�Χ����������0�����������
int PSF_size_1 = 512;
int PSF_size_2 = 512;
int PSF_size_3 = 50;
int Nxy = PSF_size_1 + NxyExt * 2; // �Ǹ����� ����ֱ�Ӵ��Դ���
int Nz = PSF_size_3; // ����ֱ���ó��� ���Դ���  Nz = 50

int threadNum_123 = 256;
int blockNum_123 = (PSF_size_1*PSF_size_2*PSF_size_3 - 1) / threadNum_123 + 1;
int threadNum_12 = 256;
int blockNum_12 = (PSF_size_1*PSF_size_2 - 1) / threadNum_12 + 1;
int threadNum_ROI = 256;
int blockNum_ROI = (ROISize * 2 * ROISize * 2 * Nz - 1) / threadNum_ROI + 1;
dim3 block(8, 8, 8);
dim3 grid((PSF_size_1 + block.x - 1) / block.x, (PSF_size_2 + block.y - 1) / block.y, (PSF_size_3 + block.z - 1) / block.z);
dim3 block_sum(32, 32, 1);
dim3 grid_sum((PSF_size_1 + block.x - 1) / block.x, (PSF_size_2 + block.y - 1) / block.y, 1);

int ObjRecon_size = 200 * 200 * 50;


void FishImageProcess::readPSFfromFile(std::string filename)
{
	cout << "start read PSF1 from file..." << endl;


	FILE *PSF_1_fid = fopen(filename.data(), "rb");
	if (PSF_1_fid == NULL)
	{
		cout << "PSF_1_file open failed!" << endl;
		system("pause");
		return;
	}

	PSF_1 = new float[PSF_size_1*PSF_size_2*PSF_size_3]();
	fread(PSF_1, sizeof(float), PSF_size_1*PSF_size_2*PSF_size_3, PSF_1_fid);

	cout << "read PSF1 done" << endl;

	return;
}

void FishImageProcess::readImageFromFile(std::string filename)
{
	cout << "start read image from file" << endl;
	//Img = readImgFromFile(filename);
	//ʹ��GDAL��ȡtif��ʹ�õ���matlab�ز����õ�����
	GDALAllRegister(); OGRRegisterAll();
	//����֧������·��
	CPLSetConfigOption("GDAL_FILENAME_IS_UTF8", "NO");
	CPLSetConfigOption("SHAPE_ENCODING", "");
	GDALDataset* poSrcDS = (GDALDataset*)GDALOpen(filename.data(), GA_ReadOnly);
	if (poSrcDS == NULL)
	{
		cout << "image file open failed!" << endl;
		return;
	}
	int wheight = poSrcDS->GetRasterYSize();//��
	int wwidth = poSrcDS->GetRasterXSize();//��
	int bandNum = poSrcDS->GetRasterCount();//������
	GDALDataType dataType = poSrcDS->GetRasterBand(1)->GetRasterDataType();//����

	Img = new unsigned short[PSF_size_1*PSF_size_2]();   //ͼ�������������
	for (int i = 0; i < bandNum; i++)
	{
		////////////////////////////////////��ȡ��ʼ�У�ʼ�У�������������ָ�룬��������������������
		poSrcDS->GetRasterBand(i + 1)->RasterIO(GF_Read, 0, 0, wwidth, wheight, Img, PSF_size_1, PSF_size_2, dataType, 0, 0);
	}
	GDALClose(poSrcDS);

	//check
	//cout << Img[512 * 256 + 256] << endl;

	cout << "read image file done" << endl;

	return;
}

void FishImageProcess::readImageFromCamera(std::string filename)
{
	//������
	return;
}

void FishImageProcess::readTemplateFromFile(std::string filenameXY, std::string filenameYZ)
{
	cout << "start read templates...." << endl;
	FILE * template_roXY_fid = fopen(filenameXY.data(), "rb");
	if (template_roXY_fid == NULL)
	{
		cout << filenameXY << " open failed!" << endl;
		system("pause");
		return;
	}
	int template_roXY_size = 200 * 200 * 360;
	template_roXY = new float[template_roXY_size];
	//fread(template_roXY, sizeof(float), template_roXY_size, template_roXY_fid);
	//fclose(template_roXY_fid);

	template_roXY = readImgFromFile(filenameXY);

	cout << "XY template read successful" << endl;



	FILE * template_roYZ_fid = fopen(filenameYZ.data(), "rb");
	if (template_roYZ_fid == NULL)
	{
		cout << filenameYZ << " open failed!" << endl;
		system("pause");
		return;
	}
	int template_roYZ_size = 200 * 50 * 31;
	template_roYZ = new float[template_roYZ_size];
	fread(template_roYZ, sizeof(float), template_roYZ_size, template_roYZ_fid);
	fclose(template_roYZ_fid);
	cout << "YZ template read successful" << endl;

	cout << "read template done" << endl;

	return;
}

void FishImageProcess::readRotationAngleFromFile(std::string filenameAngleXY, std::string filenameAngleYZ)
{
	cout << "start read rotation angle from file.." << endl;
	FILE * rotationAngleXY_fid = fopen(filenameAngleXY.data(), "rb");
	if (rotationAngleXY_fid == NULL)
	{
		cout << filenameAngleXY << " open failed!" << endl;
		system("pause");
		return;
	}
	int rotationAngleXY_size = 360;
	rotationAngleXY = new double[rotationAngleXY_size];
	fread(rotationAngleXY, sizeof(double), rotationAngleXY_size, rotationAngleXY_fid);
	fclose(rotationAngleXY_fid);
	cout << "read XY rotation angle successfule" << endl;

	FILE * rotationAngleYZ_fid = fopen(filenameAngleYZ.data(), "rb");
	if (rotationAngleYZ_fid == NULL)
	{
		cout << filenameAngleYZ << " open failed!" << endl;
		system("pause");
		return;
	}
	int rotationAngleYZ_size = 31;
	rotationAngleYZ = new double[rotationAngleYZ_size];
	fread(rotationAngleYZ, sizeof(double), rotationAngleYZ_size, rotationAngleYZ_fid);
	fclose(rotationAngleYZ_fid);

	cout << "read YZ rotation angle successfule" << endl;
	cout << "read rotation angle done" << endl;

	return;
}

void FishImageProcess::readFixImageFromFile(std::string filename)
{
	float* fixImage = readImgFromFile(filename);
	//cout << nImgSizeX << "   " << nImgSizeY << "   " << bandcount << endl;
	fixtensor = torch::from_blob(fixImage,
		{ int(imgSizeAfterCrop_Z), int(imgSizeAfterCrop_Y), int(imgSizeAfterCrop_X) }).toType(torch::kFloat32);
	fixtensor = normalizeTensor(fixtensor);
	cout << "read fix image and convert to normalize tensor" << endl;
	torch::Device device(torch::kCUDA);
	fixtensor.to(device);
	cout << "copy fix tensor to CUDA" << endl;

	//���紦���һ��ͼ���������ǰ��һ��
	model.forward({ fixtensor.to(device),fixtensor.to(device) }).toTensor();

	return;
}

//void FishImageProcess::readModelFromFile(std::string filename)
//{
//
//}

void FishImageProcess::prepareGPUMemory()
{
	/*-------׼�������������ڴ桢�Դ棬fft���--------------------------*/
	cout << "start malloc memory..." << endl;

	const int rank = 2;//ά��
	int n[rank] = { PSF_size_1, PSF_size_2 };//n*m
	int *inembed = n;//���������size
	int istride = 1;//����������������Ϊ1
	int idist = n[0] * n[1];//1��������ڴ��С
	int *onembed = n;//�����һ�������size
	int ostride = 1;//ÿ��DFT������������Ϊ1
	int odist = n[0] * n[1];//�����һ��������ڶ�������ľ��룬�������������Ԫ�صľ���
	int batch = PSF_size_3;//�������������

	//��ʼ�����
	hipfftPlanMany(&fftplanfwd, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);

	//�ع�
	check(hipMalloc((void**)&PSF_1_gpu, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float)), "PSF_1_gpu hipMalloc Error");
	check(hipMalloc((void**)&PSF_1_gpu_Complex, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "PSF_1_gpu_Complex hipMalloc Error");
	check(hipMalloc((void**)&OTF, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "OTF hipMalloc Error");
	check(hipMalloc((void**)&ImgEst, PSF_size_1*PSF_size_2 * sizeof(float)), "ImgEst hipMalloc Error");
	check(hipMalloc((void**)&Ratio, PSF_size_1*PSF_size_2 * sizeof(float)), "Ratio hipMalloc Error");
	check(hipMalloc((void**)&gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float)), "Ratio hipMalloc Error");
	check(hipMalloc((void**)&gpuObjRecROI, ROISize * 2 * ROISize * 2 * PSF_size_3 * sizeof(float)), "gpuObjRecROI hipMalloc Error");
	check(hipMalloc((void**)&Img_gpu, PSF_size_1*PSF_size_2 * sizeof(unsigned short)), "Img_gpu hipMalloc Error");
	check(hipMalloc((void**)&ImgExp, PSF_size_1*PSF_size_2 * sizeof(float)), "ImgExp hipMalloc Error");
	check(hipMalloc((void**)&gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "gpuObjRecon_Complex hipMalloc Error");
	check(hipMalloc((void**)&float_temp, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float)), "float_temp hipMalloc Error");
	check(hipMalloc((void**)&Ratio_Complex, PSF_size_1*PSF_size_2 * sizeof(hipfftComplex)), "Ratio_Complex hipMalloc Error");
	check(hipMalloc((void**)&fftRatio, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(hipfftComplex)), "fftRatio hipMalloc Error");


	//crop 512*512 to 200*200
	cpuObjRecon = new float[PSF_size_1*PSF_size_2*PSF_size_3]();
	cpuObjRecon_crop = new float[200 * 200 * PSF_size_3];
	check1(hipMalloc((void**)&gpuObjRecon_crop, sizeof(float)*ObjRecon_size), "gpuObjRecon_crop hipMalloc Error", __FILE__, __LINE__);

	//XY��ת
	check1(hipMalloc((void**)&image2D_XY_gpu, sizeof(float) * 200 * 200), "image2D_XY_gpu hipMalloc Error", __FILE__, __LINE__);
	check1(hipMalloc((void**)&img2DBW_XY_gpu, sizeof(float) * 200 * 200), "img2DBW_XY_gpu hipMalloc Error", __FILE__, __LINE__);
	check1(hipMalloc((void**)&template_roXY_gpu, sizeof(float) * template_roXY_size), "template_roXY_gpu hipMalloc Error", __FILE__, __LINE__);
	check(hipMemcpy(template_roXY_gpu, template_roXY, sizeof(float)*template_roXY_size, hipMemcpyHostToDevice), "template_roXY_gpu hipMemcpy Error");
	check1(hipMalloc((void**)&err_XY_gpu, sizeof(double) * rotationAngleXY_size), "err_XY_gpu hipMalloc Error", __FILE__, __LINE__);
	check1(hipMalloc((void**)&imageRotated3D_gpu, sizeof(float) * ObjRecon_size), "imageRotated3D_gpu hipMalloc Error", __FILE__, __LINE__);


	//crop
	cpuObjRotation_crop = new float[200 * 200 * 50];
	check1(hipMalloc((void**)&ObjCropRed_gpu, sizeof(float)*imgSizeAfterCrop_X*imgSizeAfterCrop_Y*imgSizeAfterCrop_Z),
		"ObjReconRed_gpu hipMalloc Error", __FILE__, __LINE__);


	cout << "prepare memory done" << endl;

	return;
}

void FishImageProcess::processPSF()
{
	cout << "��ʼ���ʹ���PSF...." << endl;
	check(hipMemcpy(PSF_1_gpu, PSF_1, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float), hipMemcpyHostToDevice), "PSF_1_gpu hipMemcpy Error");
	//ת���ɸ������鲿��0
	Zhuan_Complex_kernel << <blockNum_123, threadNum_123 >> > (PSF_1_gpu, PSF_1_gpu_Complex, PSF_size_1*PSF_size_2*PSF_size_3);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "PSF_1_gpu Zhuan_Complex_kernel Error");
	//*----ʹ��hipfftPlanMany�ķ�������������άfft---------------------*/
	hipfftExecC2C(fftplanfwd, PSF_1_gpu_Complex, OTF, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "PSF_1_gpu_Complex hipfftExecC2C Error");

	//ImgEst��ֵΪ0��Ratio��ֵΪ1
	initial_kernel_1 << <blockNum_12, threadNum_12 >> > (ImgEst, Ratio, PSF_size_1*PSF_size_2);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "initial_kernel_1 Error");
	gpuObjRecon_fuzhi << <blockNum_123, threadNum_123 >> > (gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "gpuObjRecon_fuzhi Error");
	//gpuObjRecROI��ֵΪ1
	initial_kernel_3 << <blockNum_ROI, threadNum_ROI >> > (gpuObjRecROI, ROISize * 2 * ROISize * 2 * Nz);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "initial_kernel_3 Error");

	cout << "��ʼ���ʹ���PSF....down" << endl;

	return;
}

void FishImageProcess::reconImage()
{
	////*----1��ʹ��hipfftPlan2d�ķ������ж�άfft----------*/
	hipfftHandle plan;
	hipfftResult res = hipfftPlan2d(&plan, PSF_size_1, PSF_size_2, HIPFFT_C2C);



	check(hipMemcpy(Img_gpu, Img, PSF_size_1*PSF_size_2 * sizeof(unsigned short), hipMemcpyHostToDevice), "Img_gpu hipMemcpy Error");
	//��ȥ������ֵ���������float���͵�����ImgExp��
	ImgExp_ge << <blockNum_12, threadNum_12 >> > (Img_gpu, BkgMean, ImgExp, PSF_size_1*PSF_size_2);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "ImgExp_ge Error");

	//Ratio��gpuObjRecon��Ԫ�ض���ֵ1
	Ratio_fuzhi << <blockNum_12, threadNum_12 >> > (Ratio, PSF_size_1*PSF_size_2);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "Ratio_fuzhi Error");
	gpuObjRecon_fuzhi << <blockNum_123, threadNum_123 >> > (gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "gpuObjRecon_fuzhi Error");




	//��ʼѭ������
	for (int i = 0; i < ItN; i++)
	{
		////1��fft2(gpuObjRecon)
		Zhuan_Complex_kernel << <blockNum_123, threadNum_123 >> > (gpuObjRecon, gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "Zhuan_Complex_kernel Error");
		hipfftExecC2C(fftplanfwd, gpuObjRecon_Complex, gpuObjRecon_Complex, HIPFFT_FORWARD);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon_Complex hipfftExecC2C Error");

		////2��OTF.*fft2(gpuObjRecon_Complex)���������gpuObjRecon_Complex��
		OTF_mul_gpuObjRecon_Complex << <blockNum_123, threadNum_123 >> > (OTF, gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "2��OTF.*fft2(gpuObjRecon_Complex) Error");

		////3��ifft2(OTF.*fft2(gpuObjRecon))����任��Ҫ���������ظ���
		hipfftExecC2C(fftplanfwd, gpuObjRecon_Complex, gpuObjRecon_Complex, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon_Complex hipfftExecC2C cufft_inverse Error");
		////4������������������ȷ
		ifft2_divide << <blockNum_123, threadNum_123 >> > (gpuObjRecon_Complex, PSF_size_1*PSF_size_2*PSF_size_3, PSF_size_1*PSF_size_2);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon_Complex ifft2_divide Error");

		/*----�ڶ���gpuObjRecon_Complex��ʵ����ȷ���鲿����ȷ������Ĵ���ֻ����gpuObjRecon_Complex��ʵ����û�õ��鲿----------*/

		////5��ifftshift + real + max(,0)�����ʵ������float_temp��С��0�ĸ�ֵ0
		ifftshift_real_max << <grid, block >> > (gpuObjRecon_Complex, float_temp, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "5��gpuObjRecon_Complex ifftshift_real_max Error");

		////6��sum( ,3)���ڵ���ά�ϼ���ͣ�����PSF_size_1��PSF_size_2�еľ���ImgEst
		float_temp_sum << <grid_sum, block_sum >> > (float_temp, ImgEst, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "float_temp_sum Error");

		////7��Tmp=mean(   ImgEst(:)   );
		thrust::device_ptr<float> dev_ptr(ImgEst);
		float Tmp = thrust::reduce(dev_ptr, dev_ptr + size_t(PSF_size_1*PSF_size_2), (float)0, thrust::plus<float>()) / (PSF_size_1*PSF_size_2);
		/**********************************************************************************************************/
		/*----������ȷ��Tmp����matlab����47424472��C������47424477.675621979�����ǳ��ǳ�С��Ӧ�ÿ��Ժ���
		�ڶ���matlab��51785136��C������51785130.147748277�����Ҳ�ǳ�С�����Ժ���----*/
		/**********************************************************************************************************/

		////8��Ratio(1:end,1:end)=ImgExp(1:end,1:end)./(ImgEst(1:end,1:end)+Tmp/SNR)����ת�ɸ��������鲿Ϊ��;
		Ratio_Complex_ge << <blockNum_12, threadNum_12 >> > (ImgExp, ImgEst, Tmp, SNR, Ratio_Complex, PSF_size_1*PSF_size_2);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "Ratio_Complex_ge Error");

		/*******************************************************************************************/
		/*----������ȷ������ĺͣ�matlab��0.3017935��C������0.301793���ڶ�����ȷ-------------------*/
		/*******************************************************************************************/

		////9��fft2(Ratio)
		res = hipfftExecC2C(plan, Ratio_Complex, Ratio_Complex, HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS)
		{
			cout << "Ratio_Complex hipfftExecC2C error:" << res << endl;
			system("pause");
			return;
		}

		/*******************************************************************************************/
		/*----������ȷ������ĺͺ�matlab��һ����������С�������λ��̫���ˣ�����ֵ�ĺ���һ����-------*/
		/*******************************************************************************************/

		////10��repmat����ֵNz�飬Ratio_Complex�����ά��fftRatio
		fftRatio_ge << <grid, block >> > (Ratio_Complex, fftRatio, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio_ge Error");


		////11��fftRatio.*conj(OTF)���浽fftRatio��
		fftRatio_mul_conjOTF << <blockNum_123, threadNum_123 >> > (fftRatio, OTF, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio_mul_conjOTF Error");


		////12��ifft2(       fftRatio.*conj(OTF)       )�������������ظ���
		hipfftExecC2C(fftplanfwd, fftRatio, fftRatio, HIPFFT_BACKWARD);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio hipfftExecC2C Error");
		ifft2_divide << <blockNum_123, threadNum_123 >> > (fftRatio, PSF_size_1*PSF_size_2*PSF_size_3, PSF_size_1*PSF_size_2);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "fftRatio ifft2_divide Error");

		////13��max(   real(   ifftshift(   ifftshift(     1),   2)   ),   0);
		ifftshift_real_max << <grid, block >> > (fftRatio, float_temp, PSF_size_1, PSF_size_2, PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "13��fftRatio ifftshift_real_max Error");

		////14��gpuObjRecon = gpuObjRecon.*max(  )
		real_multiply << <blockNum_123, threadNum_123 >> > (gpuObjRecon, float_temp, PSF_size_1*PSF_size_2*PSF_size_3);
		hipDeviceSynchronize();
		checkGPUStatus(hipGetLastError(), "gpuObjRecon real_multiply Error");


		//float* test = new float[512 * 512 * 50];
		//check(hipMemcpy(test, gpuObjRecon, sizeof(float) * 512 * 512 * 50, hipMemcpyDeviceToHost), "gpuObjRecon_crop hipMemcpy Error");
		//saveAndCheckImage(test, 512, 512, 50, int2string(2, i) + "gpuObjRecon.tif");

		//cout << "��ɵ�" << i << "��ѭ��" << endl << endl << endl;
	}



	cout << "�ع����" << endl;
	return;
}

void FishImageProcess::cropReconImage()
{
	//������ϣ�ȡֵ����cpuObjRecon
	check(hipMemcpy(cpuObjRecon, gpuObjRecon, PSF_size_1*PSF_size_2*PSF_size_3 * sizeof(float), hipMemcpyDeviceToHost), "gpuObjRecon to cpuObjRecon hipMemcpy Error");
	/*  ���crop����CPU����ɵģ��ĳ���GPU�����   */
	//gpuObjRecon crop ��200*200*50
	//����CPU��crop�󴫵�GPU
	////matlab���Ǵ�157��356�У��ܹ�356-157+1=200�С�157-356�У��ܹ�356-157+1=200�С�
	int line_start = Nxy / 2 - ROISize; int line_end = Nxy / 2 + ROISize - 1; int line_total = line_end - line_start + 1;
	int col_start = Nxy / 2 - ROISize; 	int col_end = Nxy / 2 + ROISize - 1; int col_total = col_end - col_start + 1;
	cout << "line_start: " << line_start <<endl;
	cout << "line_end: " << line_end << endl;
	cout << "line_total: " << line_total << endl;
	cout << "col_start: " << col_start << endl;
	cout << "col_end: " << col_end << endl;
	cout << "col_total: " << col_total << endl;


	for (int band = 0; band < PSF_size_3; band++)
	{
		for (int i = 0; i < line_total; i++)//��ѭ��
		{
			for (int j = 0; j < col_total; j++)//��ѭ��
			{
				cpuObjRecon_crop[band * 200 * 200 + i * 200 + j] = cpuObjRecon[band*PSF_size_1*PSF_size_2 + (i + line_start)*PSF_size_2 + j + col_start];
			}
		}
	}
	//float *gpuObjRecon_crop;   //�洢crop���ObjRecon
	check(hipMemcpy(gpuObjRecon_crop, cpuObjRecon_crop, sizeof(float)*ObjRecon_size, hipMemcpyHostToDevice), "gpuObjRecon_crop hipMemcpy Error");


	//cropReconImage_kernel << <blockNum_123, threadNum_123 >> > (gpuObjRecon, gpuObjRecon_crop);
	cout << "crop�ع��������ݲ�copy��GPU" << endl;
	return;
}


void FishImageProcess::matchingANDrotationXY()
{
	/*   XYƽ���ģ��ƥ�����ת   */
	cout << "start XY 2D template matching..." << endl;
	dim3 block_1(32, 32, 1);
	dim3 grid_1((200 + block_1.x - 1) / block_1.x, (200 + block_1.y - 1) / block_1.y, 1);
	kernel_1 << <grid_1, block_1 >> > (gpuObjRecon_crop, 200, 200, image2D_XY_gpu);   
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_1 Error");

	thrust::device_ptr<float> dev_ptr(image2D_XY_gpu);
	double image2D_XY_mean = thrust::reduce(dev_ptr, dev_ptr + size_t(200 * 200), (float)0, thrust::plus<float>()) / (200 * 200);
	cout << "image2D_XY_mean: " << image2D_XY_mean << endl;

	int threadNum_2 = 256;
	int blockNum_2 = (200 * 200 - 1) / threadNum_2 + 1;
	kernel_2 << <blockNum_2, threadNum_2 >> > (image2D_XY_gpu, 200 * 200, image2D_XY_mean, img2DBW_XY_gpu);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_2 Error");

	int threadNum_3 = 256;
	int blockNum_3 = (rotationAngleXY_size - 1) / threadNum_2 + 1;
	kernel_3 << <blockNum_3, threadNum_3 >> > (template_roXY_gpu, img2DBW_XY_gpu, rotationAngleXY_size, err_XY_gpu);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_3 Error");


	//��err_XY_gpu����Сֵ
	double *err_XY = new double[rotationAngleXY_size];
	check(hipMemcpy(err_XY, err_XY_gpu, sizeof(double)*rotationAngleXY_size, hipMemcpyDeviceToHost), "err_XY hipMemcpy Error");
	double err_XY_min = DBL_MAX;
	int idx;  //�ҵ���Сֵ��Ӧ������
	for (int i = 0; i < rotationAngleXY_size; i++)
	{
		//cout << i << "   " << err_XY[i] << endl;
		if (err_XY[i] < err_XY_min)
		{
			err_XY_min = err_XY[i];
			idx = i;
		}
	}
	//cout << "err_XY_min: " << err_XY_min << endl;
	//cout << "rotation XY idx: " << idx << endl;
	//��һ����ת

	ObjRecon_imrotate3_gpu(gpuObjRecon_crop, -rotationAngleXY[idx], imageRotated3D_gpu);
	cout << "XY 2D templaet matching and rotation done" << endl;

	return;
}

void FishImageProcess::ObjRecon_imrotate3_gpu(float *ObjRecon_gpu, double nAngle, float *imageRotated3D_gpu)
{
	NppiSize Input_Size;//����ͼ���������
	Input_Size.width = 200;
	Input_Size.height = 200;
	/* �����Դ棬��ԭͼ�����Դ� */
	int nSrcPitchCUDA = Input_Size.width * sizeof(float);//ÿ����ռ���ֽ���
	float *input_image_gpu;
	check1(hipMalloc((void**)&input_image_gpu, sizeof(float)*Input_Size.width*Input_Size.height), "input_image_gpu hipMalloc Error", __FILE__, __LINE__);


	/* ������ת�󳤿� */
	NppiRect Input_ROI;//�ض��������ת���൱�ڲü�ͼ���һ�飬���β���ȫ��ͼ��
	Input_ROI.x = Input_ROI.y = 0;
	Input_ROI.width = Input_Size.width;
	Input_ROI.height = Input_Size.height;
	double aBoundingBox[2][2];
	nppiGetRotateBound(Input_ROI, aBoundingBox, nAngle, 0, 0);
	int bb = ((int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0])) - Input_ROI.width) / 2 + aBoundingBox[0][0];//��ʼ��
	int cc = ((int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1])) - Input_ROI.height) / 2 + aBoundingBox[0][1];//��ʼ��
	aBoundingBox[0][0] = bb;//��ʼ��
	aBoundingBox[0][1] = cc;//��ʼ��
	NppiSize Output_Size;
	Output_Size.width = (int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0]));
	Output_Size.height = (int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1]));
	Output_Size.width = Input_Size.width;
	Output_Size.height = Input_Size.height;


	/* ת�����ͼ���Դ���� */
	int nDstPitchCUDA = Output_Size.width * sizeof(float);
	float *output_image_gpu;
	check1(hipMalloc((void**)&output_image_gpu, sizeof(float)*Output_Size.width*Output_Size.height), "output_image_gpu hipMalloc Error", __FILE__, __LINE__);


	//�������Ȥ���Ĵ�С���൱�ڰ����ͼ���ٲü�һ�飬Ӧ������������û���ԣ��������
	NppiRect Output_ROI;
	Output_ROI.x = 0; Output_ROI.y = 0;
	Output_ROI.width = Input_Size.width;
	Output_ROI.height = Input_Size.height;

	for (int i = 0; i < 50; i++)
	{
		check(hipMemcpy(input_image_gpu, ObjRecon_gpu + Input_Size.width*Input_Size.height * i, sizeof(float)*Input_Size.width*Input_Size.height, hipMemcpyDeviceToDevice), "input_image_gpu hipMemcpy Error");
		/* ������ת */
		NppStatus nppRet = nppiRotate_32f_C1R(input_image_gpu, Input_Size, nSrcPitchCUDA, Input_ROI,
			output_image_gpu, nDstPitchCUDA, Output_ROI, nAngle, -aBoundingBox[0][0], -aBoundingBox[0][1], NPPI_INTER_NN);
		assert(nppRet == NPP_NO_ERROR);
		check(hipMemcpy(imageRotated3D_gpu + Input_Size.width*Input_Size.height * i, output_image_gpu, sizeof(float) * Output_Size.width*Output_Size.height, hipMemcpyDeviceToDevice), "output_image hipMemcpy Error");
	}
}


//void FishImageProcess::matchingANDrotationYZ()
//{
//	/*  YZƽ���ģ��ƥ�����ת  */
////�� y-z���ͶӰ,����imageRotated3Dһ���������з�������ֵ
//	float *image2D_YZ_gpu;
//	check1(hipMalloc((void**)&image2D_YZ_gpu, sizeof(float) * 200 * 50), "image2D_YZ_gpu hipMalloc Error", __FILE__, __LINE__);//200��*50�а���imageRotated3D�����������У���matlab�а�������������
//	dim3 block_4(32, 32, 1);
//	dim3 grid_4((50 + block_4.x - 1) / block_4.x, (200 + block_4.y - 1) / block_4.y, 1);
//	kernel_4 << <grid_4, block_4 >> > (imageRotated3D_gpu, image2D_YZ_gpu);
//	hipDeviceSynchronize();
//	checkGPUStatus(hipGetLastError(), "kernel_4 Error");
//	//image2D_YZ_gpu��͡���ֵ
//	thrust::device_ptr<float> dev_ptr1(image2D_YZ_gpu);
//	double image2D_YZ_mean = thrust::reduce(dev_ptr1, dev_ptr1 + size_t(200 * 50), (float)0, thrust::plus<float>()) / (200 * 50) + 14;
//	//��ֵ�� y-z�棬����mean��ȡ1�� С�ڵ���mean��ȡ0
//	float *img2DBW_YZ_gpu;
//	check1(hipMalloc((void**)&img2DBW_YZ_gpu, sizeof(float) * 200 * 50), "img2DBW_YZ_gpu hipMalloc Error", __FILE__, __LINE__);
//	int threadNum_5 = 256;
//	int blockNum_5 = (200 * 50 - 1) / threadNum_5 + 1;
//	kernel_5 << <blockNum_5, threadNum_5 >> > (image2D_YZ_gpu, image2D_YZ_mean, img2DBW_YZ_gpu);
//	hipDeviceSynchronize();
//	checkGPUStatus(hipGetLastError(), "kernel_5 Error");
//
//
//	//��ÿ���Ƕȵ���� ��ʼ��
//	float *template_roYZ_gpu;
//	check1(hipMalloc((void**)&template_roYZ_gpu, sizeof(float) * template_roYZ_size), "template_roYZ_gpu hipMalloc Error", __FILE__, __LINE__);
//	check(hipMemcpy(template_roYZ_gpu, template_roYZ, sizeof(float)*template_roYZ_size, hipMemcpyHostToDevice), "template_roYZ_gpu hipMemcpy Error");
//	double *err_YZ_gpu;
//	check1(hipMalloc((void**)&err_YZ_gpu, sizeof(double) * rotationAngleYZ_size), "err_YZ_gpu hipMalloc Error", __FILE__, __LINE__);
//	int threadNum_6 = 256;
//	int blockNum_6 = (rotationAngleYZ_size - 1) / threadNum_6 + 1;
//	kernel_6 << <blockNum_5, threadNum_5 >> > (template_roYZ_gpu, img2DBW_YZ_gpu, rotationAngleYZ_size, err_YZ_gpu);
//	hipDeviceSynchronize();
//	checkGPUStatus(hipGetLastError(), "kernel_6 Error");
//	//��err_YZ_gpu����Сֵ����Сֵ������
//	double *err_YZ = new double[rotationAngleYZ_size];
//	check(hipMemcpy(err_YZ, err_YZ_gpu, sizeof(double)*rotationAngleYZ_size, hipMemcpyDeviceToHost), "err_YZ hipMemcpy Error");
//	double err_YZ_min = DBL_MAX;
//	for (int i = 0; i < rotationAngleYZ_size; i++)
//	{
//		if (err_YZ[i] < err_YZ_min)
//			err_YZ_min = err_YZ[i];
//	}
//	int idx2;
//	for (int i = 0; i < rotationAngleYZ_size; i++)
//	{
//		if (err_YZ[i] == err_YZ_min)
//		{
//			idx2 = i;
//			break;
//		}
//	}
//	//imageRotated3D��ת����X������תrotationAngleYZ(idx2)��
//	//�Ȱ�imageRotated3D_gpu��ά�ȱ任һ�£��б�ɲ��Σ����α���У��б�ɷ��ţ�(200 * 200 * 50)���(200�� * 50�� * 200)����
//	float *imageRotated3D_gpu_1;
//	check1(hipMalloc((void**)&imageRotated3D_gpu_1, sizeof(float)*ObjRecon_size), "imageRotated3D_gpu_1 hipMalloc Error", __FILE__, __LINE__);
//	dim3 block_7(8, 8, 8);
//	dim3 grid_7((200 + block_7.x - 1) / block_7.x, (200 + block_7.y - 1) / block_7.y, (50 + block_7.z - 1) / block_7.z);
//	kernel_7 << <grid_7, block_7 >> > (imageRotated3D_gpu, imageRotated3D_gpu_1);
//	hipDeviceSynchronize();
//	checkGPUStatus(hipGetLastError(), "kernel_7 Error");
//	//�ڶ�����ת
//	float *imageRotated3D_gpu_2;
//	check1(hipMalloc((void**)&imageRotated3D_gpu_2, sizeof(float)*ObjRecon_size), "imageRotated3D_gpu_2 hipMalloc Error", __FILE__, __LINE__);
//	ObjRecon_imrotate3_X_gpu(imageRotated3D_gpu_1, rotationAngleYZ[idx2], imageRotated3D_gpu_2);
//	//�ٰ�ά�ȱ任��ԭ����
//	dim3 block_8(8, 8, 8);
//	dim3 grid_8((200 + block_7.x - 1) / block_7.x, (200 + block_7.y - 1) / block_7.y, (50 + block_7.z - 1) / block_7.z);
//	kernel_8 << <grid_8, block_8 >> > (imageRotated3D_gpu_2, imageRotated3D_gpu);
//	hipDeviceSynchronize();
//	checkGPUStatus(hipGetLastError(), "kernel_8 Error");
//
//
//	return;
//}




void FishImageProcess::cropRotatedImage()
{
	cout << "start crop rotation image..." << endl;

	//	//����imageRotated3D_gpu�ľ�ֵ
	thrust::device_ptr<float> dev_ptr2(imageRotated3D_gpu);
	double imageRotated3D_x_mean = thrust::reduce(dev_ptr2, dev_ptr2 + size_t(ObjRecon_size), (float)0, thrust::plus<float>()) / (ObjRecon_size)+4;

	check(hipMemcpy(cpuObjRotation_crop, imageRotated3D_gpu, sizeof(float)*ObjRecon_size, hipMemcpyDeviceToHost), "ObjRecon hipMemcpy Error");

	//crop
	int *idx_2 = new int[ObjRecon_size]();//imageRotated3D_x���ھ�ֵ������
	int idx_2_size = 0;
	for (int i = 0; i < ObjRecon_size; i++)
	{
		if (cpuObjRotation_crop[i] > imageRotated3D_x_mean)
		{
			idx_2_size++;
			idx_2[idx_2_size] = i;
		}
	}
	//idx_2�����ÿһ������ת����imageRotated3D_x��200��*200��*50���Σ����кţ��кţ����κ�
	float *x = new float[idx_2_size]; float x_sum = 0;
	float *y = new float[idx_2_size]; float y_sum = 0;
	float *z = new float[idx_2_size]; float z_sum = 0;
	for (int i = 0; i < idx_2_size; i++)
	{
		z[i] = idx_2[i] / (200 * 200);
		int yushu = idx_2[i] % (200 * 200);
		x[i] = yushu / 200;
		y[i] = yushu % 200;

		x_sum += x[i];
		y_sum += y[i];
		z_sum += z[i];
	}
	int CentroID[3];
	CentroID[0] = int(x_sum / idx_2_size + 0.5);
	CentroID[1] = int(y_sum / idx_2_size + 0.5);
	CentroID[2] = int(z_sum / idx_2_size + 0.5);
	cout <<"CentroID: "<< CentroID[0] << "   " << CentroID[1] << "  " << CentroID[2] << endl;
	//CentroID������matlab����[89,91,24]���Ҽ������[86,91,24],x���3����npp��ת��matlab�Ľ���������ɵģ����Ҳ��������������

	// ��������������Χ����������������matlab������Ҫ��ȥ1
	// �з�Χ����CentroID(0)-61��CentroID(0)+33�� ���з�Χ����CentroID(2)-38��CentroID(2)+37�������еĲ���
	//int XObj = CentroID[0] + 33 - (CentroID[0] - 61) + 1;//��
	//int	YObj = CentroID[2] + 37 - (CentroID[2] - 38) + 1;//��
	//int	ZObj = 50;//����

	if (CentroID[0] < 61 || CentroID[1] < 38 || CentroID[0]>167 || CentroID[1]>163)
	{
		cout << "centroID error!!!" << endl;
		return;
	}


	dim3 block_10(8, 8, 8);
	dim3 grid_10((imgSizeAfterCrop_X + block_10.x - 1) / block_10.x, (imgSizeAfterCrop_Y + block_10.y - 1) / block_10.y, (imgSizeAfterCrop_Z + block_10.z - 1) / block_10.z);
	//__global__ void kernel_10(float *imageRotated3D_gpu, float *ObjReconRed_gpu, int XObj, int YObj, int ZObj, int CentroID0, int CentroID2)
	kernel_10 << <grid_10, block_10 >> > (imageRotated3D_gpu, ObjCropRed_gpu, imgSizeAfterCrop_X, imgSizeAfterCrop_Y, imgSizeAfterCrop_Z, CentroID[0], CentroID[1]);
	hipDeviceSynchronize();
	checkGPUStatus(hipGetLastError(), "kernel_10 Error");
	cout << "crop���" << endl;

	return;
}





void FishImageProcess::libtorchModelProcess()
{
	//libtorch
	//convert image to tensor
	torch::Tensor movingtensor;
	movingtensor = torch::from_blob(ObjCropRed_gpu,
		{ int(imgSizeAfterCrop_Z), int(imgSizeAfterCrop_Y), int(imgSizeAfterCrop_X) }, torch::kCUDA).toType(torch::kFloat32);
	movingtensor = normalizeTensor(movingtensor);
	cout << movingtensor.sizes() << endl;
	cout << fixtensor.sizes() << endl;
	cout << "1111" << endl;
	auto output = model.forward({ movingtensor.to(device),fixtensor.to(device) }).toTensor();
	//auto output = model.forward({ movingtensor,fixtensor }).toTensor();
	cout << "2222" << endl;
	std::vector<float> Moving2FixAM = rescaleAffineMatrix(output);


	if (1)
	{
		cout << Moving2FixAM.size() << endl;
		for (int aa = 0; aa < Moving2FixAM.size(); aa++)
		{
			cout << Moving2FixAM[aa] << "   ";
		}
	}
}




void FishImageProcess::clear()
{
	return;
}

void FishImageProcess::freeMemory()
{
	cout << "free cuda memory..." << endl;

	hipFree(PSF_1_gpu);
	hipFree(PSF_1_gpu_Complex);
	hipFree(OTF);
	hipFree(ImgEst);
	hipFree(Ratio);
	hipFree(gpuObjRecon);
	hipFree(gpuObjRecROI);
	hipFree(Img_gpu);
	hipFree(ImgExp);
	hipFree(gpuObjRecon_Complex);
	hipFree(float_temp);
	hipFree(Ratio_Complex);
	hipFree(fftRatio);
	hipFree(gpuObjRecon_crop);

	cout << "done" << endl;
	
	cout << "free cpu memory..." << endl;

	free(cpuObjRecon);
	free(cpuObjRecon_crop);

	cout << "done" << endl;
	return;
}

FishImageProcess::FishImageProcess(const std::string& model_path) :device(torch::kCPU)
{
	// is CUDA avaliabel??
	//torch::DeviceType device_type;
	if (torch::cuda::is_available())
	{
		device = torch::kCUDA;
		std::cout << "cuda available" << std::endl;
	}
	else
	{
		device = torch::kCPU;
		std::cout << "cuda not avaliable" << std::endl;
	}
	try
	{
		model = torch::jit::load(model_path);
	}
	catch (const c10::Error& e)
	{
		std::cerr << "Error loading the model!\n";
		std::exit(EXIT_FAILURE);
	}
	model.eval();
	model.to(device);
	std::cout << "load model success" << std::endl;
}