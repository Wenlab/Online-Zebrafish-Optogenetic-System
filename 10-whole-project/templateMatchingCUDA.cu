#include "hip/hip_runtime.h"
#include"templateMatchingCUDA.cuh"
#include"initANDcheck.h"



using namespace std;
using namespace chrono;
//ͼ�񼸺α任C++ʵ��--����ƽ�ƣ���ת�����У�����
//https://blog.csdn.net/duiwangxiaomi/article/details/109532590


__global__ void kernel_1(float *ObjRecon_gpu, int height, int width, float *image2D_XY_gpu)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//��ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//��ѭ��

	if (i < 200 && j < 200)
	{
		image2D_XY_gpu[i * 200 + j] = ObjRecon_gpu[i * 200 + j];
		for (int b = 0; b < 50; b++)//����ѭ��
		{
			if (image2D_XY_gpu[i * 200 + j] < ObjRecon_gpu[b * 200 * 200 + i * 200 + j])
			{
				image2D_XY_gpu[i * 200 + j] = ObjRecon_gpu[b * 200 * 200 + i * 200 + j];
			}
		}//����ѭ��
	}
}
__global__ void kernel_2(float *image2D_XY_gpu, int total, double image2D_XY_mean, float *img2DBW_XY_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		if (image2D_XY_gpu[i] > image2D_XY_mean)
			img2DBW_XY_gpu[i] = 1.0;
			//img2DBW_XY_gpu[i] = 255;
		else
			img2DBW_XY_gpu[i] = 0.0;
	}

}
__global__ void kernel_3(float *template_roXY_gpu, float *img2DBW_XY_gpu, int rotationAngleXY_size, double *err_XY_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < rotationAngleXY_size)
	{
		//������������ľ������
		double sum_temp = 0;
		for (int j = 0; j < 200; j++)//��ѭ��
		{
			for (int k = 0; k < 200; k++)//��ѭ��
			{
				sum_temp += (template_roXY_gpu[i * 200 * 200 + j * 200 + k] - img2DBW_XY_gpu[j * 200 + k])*
					(template_roXY_gpu[i * 200 * 200 + j * 200 + k] - img2DBW_XY_gpu[j * 200 + k]);
			}
		}
		err_XY_gpu[i] = sum_temp / (200 * 200);
	}
}

__global__ void kernel_4(float *imageRotated3D_gpu, float *image2D_YZ_gpu)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//����ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//��ѭ��

	if (i < 50 && j < 200)
	{
		image2D_YZ_gpu[i * 200 + j] = -FLT_MAX;
		for (int k = 0; k < 200; k++)//��ѭ������һ�е����ֵ
		{
			if (image2D_YZ_gpu[i * 200 + j] < imageRotated3D_gpu[i * 200 * 200 + j * 200 + k])
			{
				image2D_YZ_gpu[i * 200 + j] = imageRotated3D_gpu[i * 200 * 200 + j * 200 + k];
			}
		}
	}
}
__global__ void kernel_5(float *image2D_YZ_gpu, double image2D_YZ_mean, float *img2DBW_YZ_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < 200 * 50)
	{
		if (image2D_YZ_gpu[i] > image2D_YZ_mean)
			img2DBW_YZ_gpu[i] = 1.0;
		else
			img2DBW_YZ_gpu[i] = 0.0;
	}
}
__global__ void kernel_6(float *template_roYZ_gpu, float *img2DBW_YZ_gpu, int rotationAngleYZ_size, double *err_YZ_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < rotationAngleYZ_size)
	{
		//������������ľ������
		double sum_temp = 0;
		for (int j = 0; j < 200; j++)//��ѭ��
		{
			for (int k = 0; k < 50; k++)//��ѭ��
			{
				//template_roYZ��200��*50��*31���Σ����������У�img2DBW_YZ�����������е�
				sum_temp += (template_roYZ_gpu[i * 200 * 50 + j * 50 + k] - img2DBW_YZ_gpu[k * 200 + j])*
					(template_roYZ_gpu[i * 200 * 50 + j * 50 + k] - img2DBW_YZ_gpu[k * 200 + j]);
			}
		}
		err_YZ_gpu[i] = sum_temp / (200 * 50);
	}
}
//ά�ȱ任
__global__ void kernel_7(float *imageRotated3D_gpu, float *imageRotated3D_gpu_1)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//�������ѭ�����������ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//�����ѭ�����������ѭ����������
	const int k = blockDim.z * blockIdx.z + threadIdx.z;//�����ѭ��������Ĳ���ѭ��

	if (i < 200 && j < 200 && k < 50)
	{
		//ObjRecon[i * 200 * 50 + j * 50 + k] = imageRotated3D[199-j][i][49-k];
		imageRotated3D_gpu_1[i * 200 * 50 + j * 50 + k] = imageRotated3D_gpu[(49 - k) * 200 * 200 + (199 - j) * 200 + i];
	}
}
__global__ void kernel_8(float *imageRotated3D_gpu_2, float *imageRotated3D_gpu)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//�������ѭ�����������ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//�����ѭ�����������ѭ����������
	const int k = blockDim.z * blockIdx.z + threadIdx.z;//�����ѭ��������Ĳ���ѭ��

	if (i < 200 && j < 200 && k < 50)//�������ѭ�����������ѭ��
	{
		imageRotated3D_gpu[(49 - k) * 200 * 200 + (199 - j) * 200 + i] = imageRotated3D_gpu_2[i * 200 * 50 + j * 50 + k];
	}
}
__global__ void kernel_9(float *imageRotated3D_gpu, double imageRotated3D_x_mean, int *BWObjRecon_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < 200 * 200 * 50)
	{
		if (imageRotated3D_gpu[i] > imageRotated3D_x_mean)
			BWObjRecon_gpu[i] = 1;
		else
			BWObjRecon_gpu[i] = 0;
	}
}
__global__ void kernel_10(float *imageRotated3D_gpu, float *ObjReconRed_gpu, int XObj, int YObj, int ZObj, int CentroID0, int CentroID2)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;//XObj
	const int y = blockDim.y * blockIdx.y + threadIdx.y;//YObj
	const int z = blockDim.z * blockIdx.z + threadIdx.z;//ZObj

	if (z < ZObj && x < XObj && y < YObj)
	{
		ObjReconRed_gpu[z*XObj*YObj + y * XObj + x] = imageRotated3D_gpu[z * 200 * 200 + (CentroID0 - 61 + y) * 200 + CentroID2 - 38 + x];
	}

}