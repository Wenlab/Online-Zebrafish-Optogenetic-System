#include "hip/hip_runtime.h"
#include"templateMatchingCUDA.cuh"
#include"initANDcheck.h"



using namespace std;
using namespace chrono;
//ͼ�񼸺α任C++ʵ��--����ƽ�ƣ���ת�����У�����
//https://blog.csdn.net/duiwangxiaomi/article/details/109532590


__global__ void kernel_1(float *ObjRecon_gpu, int height, int width, float *image2D_XY_gpu)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//��ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//��ѭ��

	if (i < 200 && j < 200)
	{
		image2D_XY_gpu[i * 200 + j] = ObjRecon_gpu[i * 200 + j];
		for (int b = 0; b < 50; b++)//����ѭ��
		{
			if (image2D_XY_gpu[i * 200 + j] < ObjRecon_gpu[b * 200 * 200 + i * 200 + j])
			{
				image2D_XY_gpu[i * 200 + j] = ObjRecon_gpu[b * 200 * 200 + i * 200 + j];
			}
		}//����ѭ��
	}
}
__global__ void kernel_2(float *image2D_XY_gpu, int total, double image2D_XY_mean, float *img2DBW_XY_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		if (image2D_XY_gpu[i] > image2D_XY_mean)
			img2DBW_XY_gpu[i] = 1.0;
			//img2DBW_XY_gpu[i] = 255;
		else
			img2DBW_XY_gpu[i] = 0.0;
	}

}
__global__ void kernel_3(float *template_roXY_gpu, float *img2DBW_XY_gpu, int rotationAngleXY_size, double *err_XY_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < rotationAngleXY_size)
	{
		//������������ľ������
		double sum_temp = 0;
		for (int j = 0; j < 200; j++)//��ѭ��
		{
			for (int k = 0; k < 200; k++)//��ѭ��
			{
				sum_temp += (template_roXY_gpu[i * 200 * 200 + j * 200 + k] - img2DBW_XY_gpu[j * 200 + k])*
					(template_roXY_gpu[i * 200 * 200 + j * 200 + k] - img2DBW_XY_gpu[j * 200 + k]);
			}
		}
		err_XY_gpu[i] = sum_temp / (200 * 200);
	}
}

__global__ void kernel_4(float *imageRotated3D_gpu, float *image2D_YZ_gpu)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//����ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//��ѭ��

	if (i < 50 && j < 200)
	{
		image2D_YZ_gpu[i * 200 + j] = -FLT_MAX;
		for (int k = 0; k < 200; k++)//��ѭ������һ�е����ֵ
		{
			if (image2D_YZ_gpu[i * 200 + j] < imageRotated3D_gpu[i * 200 * 200 + j * 200 + k])
			{
				image2D_YZ_gpu[i * 200 + j] = imageRotated3D_gpu[i * 200 * 200 + j * 200 + k];
			}
		}
	}
}
__global__ void kernel_5(float *image2D_YZ_gpu, double image2D_YZ_mean, float *img2DBW_YZ_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < 200 * 50)
	{
		if (image2D_YZ_gpu[i] > image2D_YZ_mean)
			img2DBW_YZ_gpu[i] = 1.0;
		else
			img2DBW_YZ_gpu[i] = 0.0;
	}
}
__global__ void kernel_6(float *template_roYZ_gpu, float *img2DBW_YZ_gpu, int rotationAngleYZ_size, double *err_YZ_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < rotationAngleYZ_size)
	{
		//������������ľ������
		double sum_temp = 0;
		for (int j = 0; j < 200; j++)//��ѭ��
		{
			for (int k = 0; k < 50; k++)//��ѭ��
			{
				//template_roYZ��200��*50��*31���Σ����������У�img2DBW_YZ�����������е�
				sum_temp += (template_roYZ_gpu[i * 200 * 50 + j * 50 + k] - img2DBW_YZ_gpu[k * 200 + j])*
					(template_roYZ_gpu[i * 200 * 50 + j * 50 + k] - img2DBW_YZ_gpu[k * 200 + j]);
			}
		}
		err_YZ_gpu[i] = sum_temp / (200 * 50);
	}
}
//ά�ȱ任
__global__ void kernel_7(float *imageRotated3D_gpu, float *imageRotated3D_gpu_1)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//�������ѭ�����������ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//�����ѭ�����������ѭ����������
	const int k = blockDim.z * blockIdx.z + threadIdx.z;//�����ѭ��������Ĳ���ѭ��

	if (i < 200 && j < 200 && k < 50)
	{
		//ObjRecon[i * 200 * 50 + j * 50 + k] = imageRotated3D[199-j][i][49-k];
		imageRotated3D_gpu_1[i * 200 * 50 + j * 50 + k] = imageRotated3D_gpu[(49 - k) * 200 * 200 + (199 - j) * 200 + i];
	}
}
//����X����ת
//void ObjRecon_imrotate3_X_gpu(float *imageRotated3D_gpu_1, double nAngle, float *imageRotated3D_gpu_2)
//{
//	NppiSize Input_Size;//����ͼ���������
//	Input_Size.width = 200;
//	Input_Size.height = 50;
//	/* �����Դ棬��ԭͼ�����Դ� */
//	int nSrcPitchCUDA = Input_Size.width * sizeof(float);//ÿ����ռ���ֽ���
//	float *input_image_gpu;
//	check1(hipMalloc((void**)&input_image_gpu, sizeof(float)*Input_Size.width*Input_Size.height), "input_image_gpu hipMalloc Error", __FILE__, __LINE__);
//
//
//	/* ������ת�󳤿� */
//	NppiRect Input_ROI;//�ض��������ת���൱�ڲü�ͼ���һ�飬���β���ȫ��ͼ��
//	Input_ROI.x = Input_ROI.y = 0;
//	Input_ROI.width = Input_Size.width;
//	Input_ROI.height = Input_Size.height;
//	double aBoundingBox[2][2];
//	nppiGetRotateBound(Input_ROI, aBoundingBox, nAngle, 0, 0);
//	int bb = ((int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0])) - Input_ROI.width) / 2 + aBoundingBox[0][0];//��ʼ��
//	int cc = ((int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1])) - Input_ROI.height) / 2 + aBoundingBox[0][1];//��ʼ��
//	aBoundingBox[0][0] = bb;//��ʼ��
//	aBoundingBox[0][1] = cc;//��ʼ��
//	NppiSize Output_Size;
//	Output_Size.width = (int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0]));
//	Output_Size.height = (int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1]));
//	Output_Size.width = Input_Size.width;
//	Output_Size.height = Input_Size.height;
//
//
//	/* ת�����ͼ���Դ���� */
//	int nDstPitchCUDA = Output_Size.width * sizeof(float);
//	float *output_image_gpu;
//	check1(hipMalloc((void**)&output_image_gpu, sizeof(float)*Output_Size.width*Output_Size.height), "output_image_gpu hipMalloc Error", __FILE__, __LINE__);
//
//
//	//�������Ȥ���Ĵ�С���൱�ڰ����ͼ���ٲü�һ�飬Ӧ������������û���ԣ��������
//	NppiRect Output_ROI;
//	Output_ROI.x = 0; Output_ROI.y = 0;
//	Output_ROI.width = Input_Size.width;
//	Output_ROI.height = Input_Size.height;
//
//	for (int i = 0; i < 200; i++)
//	{
//		check(hipMemcpy(input_image_gpu, imageRotated3D_gpu_1 + Input_Size.width*Input_Size.height * i, sizeof(float)*Input_Size.width*Input_Size.height, hipMemcpyDeviceToDevice), "input_image_gpu hipMemcpy Error");
//		/* ������ת */
//		NppStatus nppRet = nppiRotate_32f_C1R(input_image_gpu, Input_Size, nSrcPitchCUDA, Input_ROI,
//			output_image_gpu, nDstPitchCUDA, Output_ROI, nAngle, -aBoundingBox[0][0], -aBoundingBox[0][1], NPPI_INTER_NN);
//		assert(nppRet == NPP_NO_ERROR);
//		check(hipMemcpy(imageRotated3D_gpu_2 + Input_Size.width*Input_Size.height * i, output_image_gpu, sizeof(float) * Output_Size.width*Output_Size.height, hipMemcpyDeviceToDevice), "output_image hipMemcpy Error");
//	}
//}
//�ٱ任��ԭ����ά��
__global__ void kernel_8(float *imageRotated3D_gpu_2, float *imageRotated3D_gpu)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//�������ѭ�����������ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//�����ѭ�����������ѭ����������
	const int k = blockDim.z * blockIdx.z + threadIdx.z;//�����ѭ��������Ĳ���ѭ��

	if (i < 200 && j < 200 && k < 50)//�������ѭ�����������ѭ��
	{
		imageRotated3D_gpu[(49 - k) * 200 * 200 + (199 - j) * 200 + i] = imageRotated3D_gpu_2[i * 200 * 50 + j * 50 + k];
	}
}
__global__ void kernel_9(float *imageRotated3D_gpu, double imageRotated3D_x_mean, int *BWObjRecon_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < 200 * 200 * 50)
	{
		if (imageRotated3D_gpu[i] > imageRotated3D_x_mean)
			BWObjRecon_gpu[i] = 1;
		else
			BWObjRecon_gpu[i] = 0;
	}
}
__global__ void kernel_10(float *imageRotated3D_gpu, float *ObjReconRed_gpu, int XObj, int YObj, int ZObj, int CentroID0, int CentroID2)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;//XObj
	const int y = blockDim.y * blockIdx.y + threadIdx.y;//YObj
	const int z = blockDim.z * blockIdx.z + threadIdx.z;//ZObj

	if (z < ZObj && x < XObj && y < YObj)
	{
		ObjReconRed_gpu[z*XObj*YObj + y * XObj + x] = imageRotated3D_gpu[z * 200 * 200 + (CentroID0 - 61 + y) * 200 + CentroID2 - 38 + x];
	}

}