#include "hip/hip_runtime.h"
#include"templateMatchingCUDA.cuh"
#include"initANDcheck.h"



using namespace std;
using namespace chrono;
//ͼ�񼸺α任C++ʵ��--����ƽ�ƣ���ת�����У�����
//https://blog.csdn.net/duiwangxiaomi/article/details/109532590


//npp��ͼ����ת����nppiRotate_32f_C1Rʹ�÷�����
//����ת�Ƕ�Ϊ����ͼ������½������ȿ�ʼ���ģ�˳ʱ����ת�����Ͻǿ�ʼ���ģ���ʱ����ת
//����ת�Ƕ�Ϊ����ͼ������½������ȿ�ʼ���ģ���ʱ����ת�����Ͻǿ�ʼ���ģ�˳ʱ����ת
float *ObjRecon_imrotate3(float *ObjRecon, double nAngle)
{
	//float *input_image = new float[200*200];
	float *imageRotated3D = new float[200 * 200 * 50];
	//for (int i = 0; i < 200 * 200; i++)
	//{
	//	input_image[i] = ObjRecon[i];
	//}

	NppiSize Input_Size;//����ͼ���������
	Input_Size.width = 200;
	Input_Size.height = 200;
	/* �����Դ棬��ԭͼ�����Դ� */
	int nSrcPitchCUDA = Input_Size.width * sizeof(float);//ÿ����ռ���ֽ���
	float *input_image_gpu;
	check1(hipMalloc((void**)&input_image_gpu, sizeof(float)*Input_Size.width*Input_Size.height), "input_image_gpu hipMalloc Error", __FILE__, __LINE__);
	//check(hipMemcpy(input_image_gpu, input_image, sizeof(float)*Input_Size.width*Input_Size.height, hipMemcpyHostToDevice), "input_image_gpu hipMemcpy Error");


	/* ������ת�󳤿� */
	NppiRect Input_ROI;//�ض��������ת���൱�ڲü�ͼ���һ�飬���β���ȫ��ͼ��
	Input_ROI.x = Input_ROI.y = 0;
	Input_ROI.width = Input_Size.width;
	Input_ROI.height = Input_Size.height;
	double aBoundingBox[2][2];
	nppiGetRotateBound(Input_ROI, aBoundingBox, nAngle, 0, 0);
	int bb = ((int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0])) - Input_ROI.width) / 2 + aBoundingBox[0][0];//��ʼ��
	int cc = ((int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1])) - Input_ROI.height) / 2 + aBoundingBox[0][1];//��ʼ��
	aBoundingBox[0][0] = bb;//��ʼ��
	aBoundingBox[0][1] = cc;//��ʼ��
	NppiSize Output_Size;
	Output_Size.width = (int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0]));
	Output_Size.height = (int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1]));
	Output_Size.width = Input_Size.width;
	Output_Size.height = Input_Size.height;


	/* ת�����ͼ���Դ���� */
	int nDstPitchCUDA = Output_Size.width * sizeof(float);
	float *output_image_gpu;
	check1(hipMalloc((void**)&output_image_gpu, sizeof(float)*Output_Size.width*Output_Size.height), "output_image_gpu hipMalloc Error", __FILE__, __LINE__);


	//�������Ȥ���Ĵ�С���൱�ڰ����ͼ���ٲü�һ�飬Ӧ������������û���ԣ��������
	NppiRect Output_ROI;
	Output_ROI.x = 0; Output_ROI.y = 0;
	Output_ROI.width = Input_Size.width;
	Output_ROI.height = Input_Size.height;

	for (int i = 0; i < 50; i++)
	{
		check(hipMemcpy(input_image_gpu, ObjRecon + Input_Size.width*Input_Size.height * i, sizeof(float)*Input_Size.width*Input_Size.height, hipMemcpyHostToDevice), "input_image_gpu hipMemcpy Error");
		/* ������ת */
		NppStatus nppRet = nppiRotate_32f_C1R(input_image_gpu, Input_Size, nSrcPitchCUDA, Input_ROI,
			output_image_gpu, nDstPitchCUDA, Output_ROI, nAngle, -aBoundingBox[0][0], -aBoundingBox[0][1], NPPI_INTER_NN);
		assert(nppRet == NPP_NO_ERROR);
		check(hipMemcpy(imageRotated3D + Input_Size.width*Input_Size.height * i, output_image_gpu, sizeof(float) * Output_Size.width*Output_Size.height, hipMemcpyDeviceToHost), "output_image hipMemcpy Error");
	}

	////��תǰ��ĵ�һ�����ηֱ�д��������
	//float *ObjRecon_1 = new float[200 * 200];
	//float *imageRotated3D_1 = new float[200 * 200];
	//for (int i = 0; i < 200*200; i++)
	//{
	//	ObjRecon_1[i] = ObjRecon[i];
	//	imageRotated3D_1[i] = imageRotated3D[i];
	//}
	//GDALDriver * pDriver = GetGDALDriverManager()->GetDriverByName("GTiff");
	//GDALDataset *ds1 = pDriver->Create("ObjRecon_1_c", 200, 200, 1, GDT_Float32, NULL);
	//GDALDataset *ds2 = pDriver->Create("imageRotated3D_1_c", 200, 200, 1, GDT_Float32, NULL);
	//if ((ds1 == NULL) || (ds2 == NULL))
	//{
	//	cout << "create ObjRecon_1 imageRotated3D_1 output_file error!" << endl;
	//	system("pause");
	//	return 0;
	//}
	////��ͼ������Ͻ�һ��һ�е�д
	//ds1->GetRasterBand(1)->RasterIO(GF_Write, 0, 0, 200, 200, ObjRecon_1, 200, 200, GDT_Float32, 0, 0);
	//ds2->GetRasterBand(1)->RasterIO(GF_Write, 0, 0, 200, 200, imageRotated3D_1, 200, 200, GDT_Float32, 0, 0);
	//GDALClose(ds1);
	//GDALClose(ds2);



	return imageRotated3D;
}

//����X����ת
float *ObjRecon_imrotate3_X(float *imageRotated3D, double nAngle)
{
	//imageRotated3D(200*200*50)ת���ɣ��б�ɲ��Σ����α���У��б�ɷ��ţ����200��*50��*200����
	float *ObjRecon = new float[200 * 50 * 200];
	for (int i = 0; i < 200; i++)//�������ѭ�����������ѭ��
	{
		for (int j = 0; j < 200; j++)//�����ѭ�����������ѭ����������
		{
			for (int k = 0; k < 50; k++)//�����ѭ��������Ĳ���ѭ��
			{
				//ObjRecon[i * 200 * 50 + j * 50 + k] = imageRotated3D[199-j][i][49-k];
				ObjRecon[i * 200 * 50 + j * 50 + k] = imageRotated3D[(49 - k) * 200 * 200 + (199 - j) * 200 + i];
			}
		}
	}

	float *imageRotated3D_rotate = new float[200 * 50 * 200];

	NppiSize Input_Size;//����ͼ���������
	Input_Size.width = 200;
	Input_Size.height = 50;
	/* �����Դ棬��ԭͼ�����Դ� */
	int nSrcPitchCUDA = Input_Size.width * sizeof(float);//ÿ����ռ���ֽ���
	float *input_image_gpu;
	check1(hipMalloc((void**)&input_image_gpu, sizeof(float)*Input_Size.width*Input_Size.height), "input_image_gpu hipMalloc Error", __FILE__, __LINE__);


	/* ������ת�󳤿� */
	NppiRect Input_ROI;//�ض��������ת���൱�ڲü�ͼ���һ�飬���β���ȫ��ͼ��
	Input_ROI.x = Input_ROI.y = 0;
	Input_ROI.width = Input_Size.width;
	Input_ROI.height = Input_Size.height;
	double aBoundingBox[2][2];
	nppiGetRotateBound(Input_ROI, aBoundingBox, nAngle, 0, 0);
	int bb = ((int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0])) - Input_ROI.width) / 2 + aBoundingBox[0][0];//��ʼ��
	int cc = ((int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1])) - Input_ROI.height) / 2 + aBoundingBox[0][1];//��ʼ��
	aBoundingBox[0][0] = bb;//��ʼ��
	aBoundingBox[0][1] = cc;//��ʼ��
	NppiSize Output_Size;
	Output_Size.width = (int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0]));
	Output_Size.height = (int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1]));
	Output_Size.width = Input_Size.width;
	Output_Size.height = Input_Size.height;


	/* ת�����ͼ���Դ���� */
	int nDstPitchCUDA = Output_Size.width * sizeof(float);
	float *output_image_gpu;
	check1(hipMalloc((void**)&output_image_gpu, sizeof(float)*Output_Size.width*Output_Size.height), "output_image_gpu hipMalloc Error", __FILE__, __LINE__);


	//�������Ȥ���Ĵ�С���൱�ڰ����ͼ���ٲü�һ�飬Ӧ������������û���ԣ��������
	NppiRect Output_ROI;
	Output_ROI.x = 0; Output_ROI.y = 0;
	Output_ROI.width = Input_Size.width;
	Output_ROI.height = Input_Size.height;

	for (int i = 0; i < 200; i++)
	{
		check(hipMemcpy(input_image_gpu, ObjRecon + Input_Size.width*Input_Size.height * i, sizeof(float)*Input_Size.width*Input_Size.height, hipMemcpyHostToDevice), "input_image_gpu hipMemcpy Error");
		/* ������ת */
		NppStatus nppRet = nppiRotate_32f_C1R(input_image_gpu, Input_Size, nSrcPitchCUDA, Input_ROI,
			output_image_gpu, nDstPitchCUDA, Output_ROI, nAngle, -aBoundingBox[0][0], -aBoundingBox[0][1], NPPI_INTER_NN);
		assert(nppRet == NPP_NO_ERROR);
		check(hipMemcpy(imageRotated3D_rotate + Input_Size.width*Input_Size.height * i, output_image_gpu, sizeof(float) * Output_Size.width*Output_Size.height, hipMemcpyDeviceToHost), "output_image hipMemcpy Error");
	}

	//�ٱ任��ԭ����ά�ȷֲ�
	//200��*50��*200���Ρ�>>200��*200��*50��
	float *imageRotated3D_rotate_return = new float[200 * 50 * 200];
	for (int i = 0; i < 200; i++)//�������ѭ�����������ѭ��
	{
		for (int j = 0; j < 200; j++)//�����ѭ�����������ѭ����������
		{
			for (int k = 0; k < 50; k++)//�����ѭ��������Ĳ���ѭ��
			{
				imageRotated3D_rotate_return[(49 - k) * 200 * 200 + (199 - j) * 200 + i] = imageRotated3D_rotate[i * 200 * 50 + j * 50 + k];
			}
		}
	}

	return imageRotated3D_rotate_return;
}

//�ز�����ָ������
int reSampleGDAL(const char* pszSrcFile, const char* pszOutFile, int newWidth, int newHeight, GDALResampleAlg eResample)
{
	GDALAllRegister();
	CPLSetConfigOption("GDAL_FILENAME_IS_UTF8", "NO");
	GDALDataset* pDSrc = (GDALDataset*)GDALOpen(pszSrcFile, GA_Update);
	if (pDSrc == NULL)
	{
		cout << "���ļ�ʧ�ܣ�" << endl;
		return -1;
	}
	int width = pDSrc->GetRasterXSize();
	int height = pDSrc->GetRasterYSize();
	int nBandCount = pDSrc->GetRasterCount();
	GDALDataType dataType = pDSrc->GetRasterBand(1)->GetRasterDataType();
	char* pszSrcWKT = const_cast<char*>(pDSrc->GetProjectionRef());
	double dGeoTrans[6] = { 0 };
	pDSrc->GetGeoTransform(dGeoTrans);
	double dOldGeoTrans0 = dGeoTrans[0];
	//���û��ͶӰ����Ϊ����һ��    
	if (strlen(pszSrcWKT) <= 0)
	{
		OGRSpatialReference oSRS;
		oSRS.SetWellKnownGeogCS("WGS84");
		oSRS.exportToWkt(&pszSrcWKT);
		pDSrc->SetProjection(pszSrcWKT);
		dGeoTrans[0] = 30.0;
		dGeoTrans[3] = 30.0;
		dGeoTrans[1] = 0.00001;
		dGeoTrans[5] = -0.00001;
		pDSrc->SetGeoTransform(dGeoTrans);
	}


	/*************** �����ز�����ĳߴ���ȡͶӰ��Ϣ **************/
	float fResX = (1.0 * newWidth) / width;
	float fResY = (1.0 * newHeight) / height;
	//��ȡ�ز�����Ϣ
	dGeoTrans[1] = dGeoTrans[1] / fResX;
	dGeoTrans[5] = dGeoTrans[5] / fResY;
	int nNewWidth = static_cast<int>(newWidth + 0.5);
	int nNewHeight = static_cast<int>(newHeight + 0.5);
	//��������ļ�
	GDALDriver* pDriver = GetGDALDriverManager()->GetDriverByName("GTiff");
	if (pDriver == NULL)
	{
		GDALClose((GDALDatasetH)pDSrc);
		cout << "�����ļ�ʧ�ܣ�" << endl;
		return -2;
	}
	//����������ݼ�  
	GDALDataset* pDDst = pDriver->Create(pszOutFile, nNewWidth, nNewHeight, nBandCount, dataType, NULL);
	if (pDDst == NULL)
	{
		GDALClose((GDALDatasetH)pDSrc);
		cout << "�����ļ�ʧ�ܣ�" << endl;
		return -2;
	}
	pDDst->SetProjection(pszSrcWKT);
	pDDst->SetGeoTransform(dGeoTrans);



	//�����ز���ǰ��Ķ�Ӧ��ϵ��׼������...
	void* hTransformArg = GDALCreateGenImgProjTransformer2((GDALDatasetH)pDSrc, (GDALDatasetH)pDDst, NULL); //GDALCreateGenImgProjTransformer((GDALDatasetH) pDSrc,pszSrcWKT,(GDALDatasetH) pDDst,pszSrcWKT,FALSE,0.0,1);  
	if (hTransformArg == NULL)
	{
		GDALClose((GDALDatasetH)pDSrc);
		GDALClose((GDALDatasetH)pDDst);
		cout << "��������г���" << endl;
		return -3;
	}

	GDALWarpOptions* psWo = GDALCreateWarpOptions();
	psWo->papszWarpOptions = CSLDuplicate(NULL);
	psWo->eWorkingDataType = dataType;
	psWo->eResampleAlg = eResample;
	psWo->hSrcDS = (GDALDatasetH)pDSrc;
	psWo->hDstDS = (GDALDatasetH)pDDst;
	psWo->pfnTransformer = GDALGenImgProjTransform;
	psWo->pTransformerArg = hTransformArg;
	psWo->nBandCount = nBandCount;
	psWo->panSrcBands = (int*)CPLMalloc(nBandCount * sizeof(int));
	psWo->panDstBands = (int*)CPLMalloc(nBandCount * sizeof(int));
	for (int i = 0; i < nBandCount; i++)
	{
		psWo->panSrcBands[i] = i + 1;
		psWo->panDstBands[i] = i + 1;
	}


	GDALWarpOperation oWo;
	if (oWo.Initialize(psWo) != CE_None)
	{
		GDALClose((GDALDatasetH)pDSrc);
		GDALClose((GDALDatasetH)pDDst);

		cout << "��������г���" << endl;
		return -3;
	}

	oWo.ChunkAndWarpImage(0, 0, nNewWidth, nNewHeight);

	//�ƺ�
	GDALDestroyGenImgProjTransformer(hTransformArg);
	GDALDestroyWarpOptions(psWo);
	GDALFlushCache(pDDst);
	GDALClose((GDALDatasetH)pDSrc);
	GDALClose((GDALDatasetH)pDDst);
	return 0;
}

//�ز�����ָ������
int reSampleGDAL_1(float *ArrayBand, int width, int height, int nBandCount, GDALDataType dataType,
	float *ArrayBand_out, int newWidth, int newHeight, GDALResampleAlg eResample)
{
	//��������ļ�
	GDALDriver *pDriver = GetGDALDriverManager()->GetDriverByName("MEM");
	if (pDriver == NULL)
	{
		cout << "����GDALDriverʧ�ܣ�" << endl;
		return -2;
	}
	GDALDataset *pDSrc = pDriver->Create("", width, height, nBandCount, dataType, NULL);
	pDSrc->RasterIO(GF_Write, 0, 0, width, height, ArrayBand, width, height, dataType, nBandCount, NULL, 0, 0, 0);
	//����ͶӰ
	char* pszSrcWKT;
	double dGeoTrans[6] = { 0 };
	OGRSpatialReference oSRS;
	oSRS.SetWellKnownGeogCS("WGS84");
	oSRS.exportToWkt(&pszSrcWKT);
	pDSrc->SetProjection(pszSrcWKT);
	dGeoTrans[0] = 30.0;
	dGeoTrans[3] = 30.0;
	dGeoTrans[1] = 0.00001;
	dGeoTrans[5] = -0.00001;
	pDSrc->SetGeoTransform(dGeoTrans);

	/*************** �����ز�����ĳߴ���ȡͶӰ��Ϣ **************/
	float fResX = (1.0 * newWidth) / width;
	float fResY = (1.0 * newHeight) / height;
	//��ȡ�ز�����Ϣ
	dGeoTrans[1] = dGeoTrans[1] / fResX;
	dGeoTrans[5] = dGeoTrans[5] / fResY;
	int nNewWidth = static_cast<int>(newWidth + 0.5);
	int nNewHeight = static_cast<int>(newHeight + 0.5);

	//����������ݼ�  
	//GDALDriver* pDriver1 = GetGDALDriverManager()->GetDriverByName("GTiff");
	GDALDataset* pDDst = pDriver->Create("",
		nNewWidth, nNewHeight, nBandCount, dataType, NULL);
	if (pDDst == NULL)
	{
		GDALClose((GDALDatasetH)pDSrc);
		cout << "�����ļ�ʧ�ܣ�" << endl;
		return -2;
	}
	pDDst->SetProjection(pszSrcWKT);
	pDDst->SetGeoTransform(dGeoTrans);



	//�����ز���ǰ��Ķ�Ӧ��ϵ��׼������...
	void* hTransformArg = GDALCreateGenImgProjTransformer2((GDALDatasetH)pDSrc, (GDALDatasetH)pDDst, NULL); //GDALCreateGenImgProjTransformer((GDALDatasetH) pDSrc,pszSrcWKT,(GDALDatasetH) pDDst,pszSrcWKT,FALSE,0.0,1);  
	if (hTransformArg == NULL)
	{
		GDALClose((GDALDatasetH)pDSrc);
		GDALClose((GDALDatasetH)pDDst);
		cout << "��������г���" << endl;
		return -3;
	}

	GDALWarpOptions* psWo = GDALCreateWarpOptions();
	psWo->papszWarpOptions = CSLDuplicate(NULL);
	psWo->eWorkingDataType = dataType;
	psWo->eResampleAlg = eResample;
	psWo->hSrcDS = (GDALDatasetH)pDSrc;
	psWo->hDstDS = (GDALDatasetH)pDDst;
	psWo->pfnTransformer = GDALGenImgProjTransform;
	psWo->pTransformerArg = hTransformArg;
	psWo->nBandCount = nBandCount;
	psWo->panSrcBands = (int*)CPLMalloc(nBandCount * sizeof(int));
	psWo->panDstBands = (int*)CPLMalloc(nBandCount * sizeof(int));
	for (int i = 0; i < nBandCount; i++)
	{
		psWo->panSrcBands[i] = i + 1;
		psWo->panDstBands[i] = i + 1;
	}


	GDALWarpOperation oWo;
	if (oWo.Initialize(psWo) != CE_None)
	{
		GDALClose((GDALDatasetH)pDSrc);
		GDALClose((GDALDatasetH)pDDst);

		cout << "��������г���" << endl;
		return -3;
	}

	oWo.ChunkAndWarpImage(0, 0, nNewWidth, nNewHeight);

	pDDst->RasterIO(GF_Read, 0, 0, nNewWidth, nNewHeight, ArrayBand_out, nNewWidth, nNewHeight, dataType, nBandCount, NULL, 0, 0, 0);


	//�ƺ�
	GDALDestroyGenImgProjTransformer(hTransformArg);
	GDALDestroyWarpOptions(psWo);
	GDALFlushCache(pDDst);
	GDALClose((GDALDatasetH)pDSrc);
	GDALClose((GDALDatasetH)pDDst);
	return 0;
}


__global__ void kernel_1(float *ObjRecon_gpu, int height, int width, float *image2D_XY_gpu)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//��ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//��ѭ��

	if (i < 200 && j < 200)
	{
		image2D_XY_gpu[i * 200 + j] = ObjRecon_gpu[i * 200 + j];
		for (int b = 0; b < 50; b++)//����ѭ��
		{
			if (image2D_XY_gpu[i * 200 + j] < ObjRecon_gpu[b * 200 * 200 + i * 200 + j])
			{
				image2D_XY_gpu[i * 200 + j] = ObjRecon_gpu[b * 200 * 200 + i * 200 + j];
			}
		}//����ѭ��
	}
}
__global__ void kernel_2(float *image2D_XY_gpu, int total, double image2D_XY_mean, float *img2DBW_XY_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < total)
	{
		if (image2D_XY_gpu[i] > image2D_XY_mean)
			img2DBW_XY_gpu[i] = 1.0;
			//img2DBW_XY_gpu[i] = 255;
		else
			img2DBW_XY_gpu[i] = 0.0;
	}

}
__global__ void kernel_3(float *template_roXY_gpu, float *img2DBW_XY_gpu, int rotationAngleXY_size, double *err_XY_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < rotationAngleXY_size)
	{
		//������������ľ������
		double sum_temp = 0;
		for (int j = 0; j < 200; j++)//��ѭ��
		{
			for (int k = 0; k < 200; k++)//��ѭ��
			{
				sum_temp += (template_roXY_gpu[i * 200 * 200 + j * 200 + k] - img2DBW_XY_gpu[j * 200 + k])*
					(template_roXY_gpu[i * 200 * 200 + j * 200 + k] - img2DBW_XY_gpu[j * 200 + k]);
			}
		}
		err_XY_gpu[i] = sum_temp / (200 * 200);
	}
}

__global__ void kernel_4(float *imageRotated3D_gpu, float *image2D_YZ_gpu)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//����ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//��ѭ��

	if (i < 50 && j < 200)
	{
		image2D_YZ_gpu[i * 200 + j] = -FLT_MAX;
		for (int k = 0; k < 200; k++)//��ѭ������һ�е����ֵ
		{
			if (image2D_YZ_gpu[i * 200 + j] < imageRotated3D_gpu[i * 200 * 200 + j * 200 + k])
			{
				image2D_YZ_gpu[i * 200 + j] = imageRotated3D_gpu[i * 200 * 200 + j * 200 + k];
			}
		}
	}
}
__global__ void kernel_5(float *image2D_YZ_gpu, double image2D_YZ_mean, float *img2DBW_YZ_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < 200 * 50)
	{
		if (image2D_YZ_gpu[i] > image2D_YZ_mean)
			img2DBW_YZ_gpu[i] = 1.0;
		else
			img2DBW_YZ_gpu[i] = 0.0;
	}
}
__global__ void kernel_6(float *template_roYZ_gpu, float *img2DBW_YZ_gpu, int rotationAngleYZ_size, double *err_YZ_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < rotationAngleYZ_size)
	{
		//������������ľ������
		double sum_temp = 0;
		for (int j = 0; j < 200; j++)//��ѭ��
		{
			for (int k = 0; k < 50; k++)//��ѭ��
			{
				//template_roYZ��200��*50��*31���Σ����������У�img2DBW_YZ�����������е�
				sum_temp += (template_roYZ_gpu[i * 200 * 50 + j * 50 + k] - img2DBW_YZ_gpu[k * 200 + j])*
					(template_roYZ_gpu[i * 200 * 50 + j * 50 + k] - img2DBW_YZ_gpu[k * 200 + j]);
			}
		}
		err_YZ_gpu[i] = sum_temp / (200 * 50);
	}
}
//ά�ȱ任
__global__ void kernel_7(float *imageRotated3D_gpu, float *imageRotated3D_gpu_1)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//�������ѭ�����������ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//�����ѭ�����������ѭ����������
	const int k = blockDim.z * blockIdx.z + threadIdx.z;//�����ѭ��������Ĳ���ѭ��

	if (i < 200 && j < 200 && k < 50)
	{
		//ObjRecon[i * 200 * 50 + j * 50 + k] = imageRotated3D[199-j][i][49-k];
		imageRotated3D_gpu_1[i * 200 * 50 + j * 50 + k] = imageRotated3D_gpu[(49 - k) * 200 * 200 + (199 - j) * 200 + i];
	}
}
//����X����ת
void ObjRecon_imrotate3_X_gpu(float *imageRotated3D_gpu_1, double nAngle, float *imageRotated3D_gpu_2)
{
	NppiSize Input_Size;//����ͼ���������
	Input_Size.width = 200;
	Input_Size.height = 50;
	/* �����Դ棬��ԭͼ�����Դ� */
	int nSrcPitchCUDA = Input_Size.width * sizeof(float);//ÿ����ռ���ֽ���
	float *input_image_gpu;
	check1(hipMalloc((void**)&input_image_gpu, sizeof(float)*Input_Size.width*Input_Size.height), "input_image_gpu hipMalloc Error", __FILE__, __LINE__);


	/* ������ת�󳤿� */
	NppiRect Input_ROI;//�ض��������ת���൱�ڲü�ͼ���һ�飬���β���ȫ��ͼ��
	Input_ROI.x = Input_ROI.y = 0;
	Input_ROI.width = Input_Size.width;
	Input_ROI.height = Input_Size.height;
	double aBoundingBox[2][2];
	nppiGetRotateBound(Input_ROI, aBoundingBox, nAngle, 0, 0);
	int bb = ((int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0])) - Input_ROI.width) / 2 + aBoundingBox[0][0];//��ʼ��
	int cc = ((int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1])) - Input_ROI.height) / 2 + aBoundingBox[0][1];//��ʼ��
	aBoundingBox[0][0] = bb;//��ʼ��
	aBoundingBox[0][1] = cc;//��ʼ��
	NppiSize Output_Size;
	Output_Size.width = (int)ceil(fabs(aBoundingBox[1][0] - aBoundingBox[0][0]));
	Output_Size.height = (int)ceil(fabs(aBoundingBox[1][1] - aBoundingBox[0][1]));
	Output_Size.width = Input_Size.width;
	Output_Size.height = Input_Size.height;


	/* ת�����ͼ���Դ���� */
	int nDstPitchCUDA = Output_Size.width * sizeof(float);
	float *output_image_gpu;
	check1(hipMalloc((void**)&output_image_gpu, sizeof(float)*Output_Size.width*Output_Size.height), "output_image_gpu hipMalloc Error", __FILE__, __LINE__);


	//�������Ȥ���Ĵ�С���൱�ڰ����ͼ���ٲü�һ�飬Ӧ������������û���ԣ��������
	NppiRect Output_ROI;
	Output_ROI.x = 0; Output_ROI.y = 0;
	Output_ROI.width = Input_Size.width;
	Output_ROI.height = Input_Size.height;

	for (int i = 0; i < 200; i++)
	{
		check(hipMemcpy(input_image_gpu, imageRotated3D_gpu_1 + Input_Size.width*Input_Size.height * i, sizeof(float)*Input_Size.width*Input_Size.height, hipMemcpyDeviceToDevice), "input_image_gpu hipMemcpy Error");
		/* ������ת */
		NppStatus nppRet = nppiRotate_32f_C1R(input_image_gpu, Input_Size, nSrcPitchCUDA, Input_ROI,
			output_image_gpu, nDstPitchCUDA, Output_ROI, nAngle, -aBoundingBox[0][0], -aBoundingBox[0][1], NPPI_INTER_NN);
		assert(nppRet == NPP_NO_ERROR);
		check(hipMemcpy(imageRotated3D_gpu_2 + Input_Size.width*Input_Size.height * i, output_image_gpu, sizeof(float) * Output_Size.width*Output_Size.height, hipMemcpyDeviceToDevice), "output_image hipMemcpy Error");
	}
}
//�ٱ任��ԭ����ά��
__global__ void kernel_8(float *imageRotated3D_gpu_2, float *imageRotated3D_gpu)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;//�������ѭ�����������ѭ��
	const int j = blockDim.y * blockIdx.y + threadIdx.y;//�����ѭ�����������ѭ����������
	const int k = blockDim.z * blockIdx.z + threadIdx.z;//�����ѭ��������Ĳ���ѭ��

	if (i < 200 && j < 200 && k < 50)//�������ѭ�����������ѭ��
	{
		imageRotated3D_gpu[(49 - k) * 200 * 200 + (199 - j) * 200 + i] = imageRotated3D_gpu_2[i * 200 * 50 + j * 50 + k];
	}
}
__global__ void kernel_9(float *imageRotated3D_gpu, double imageRotated3D_x_mean, int *BWObjRecon_gpu)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < 200 * 200 * 50)
	{
		if (imageRotated3D_gpu[i] > imageRotated3D_x_mean)
			BWObjRecon_gpu[i] = 1;
		else
			BWObjRecon_gpu[i] = 0;
	}
}
__global__ void kernel_10(float *imageRotated3D_gpu, float *ObjReconRed_gpu, int XObj, int YObj, int ZObj, int CentroID0, int CentroID2)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;//XObj
	const int y = blockDim.y * blockIdx.y + threadIdx.y;//YObj
	const int z = blockDim.z * blockIdx.z + threadIdx.z;//ZObj

	if (z < ZObj && x < XObj && y < YObj)
	{
		ObjReconRed_gpu[z*XObj*YObj + y * XObj + x] = imageRotated3D_gpu[z * 200 * 200 + (CentroID0 - 61 + y) * 200 + CentroID2 - 38 + x];
	}

}