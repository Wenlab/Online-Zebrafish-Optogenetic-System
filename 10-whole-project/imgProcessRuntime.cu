#define _CRT_SECURE_NO_WARNINGS

#include"imgProcess.h"
//#include "imgProcess.cu"
#include"kexinLibs.h"

#include<vector>
#include<iostream>

using std::string;
using std::vector;
using std::cout;
using std::endl;



int main()
{
	//��ȡPSF��δ�ع����ļ�
	string PSF_1_file = "F:/ITK/cuda-test/data/PSF_1_zhuanzhi_float.dat";//matlab�б��������float����
	string X31_file = "F:/ITK/cuda-test/data/r20210924_2_X31_resize.tif";
	//��ȡ�Ƕȡ���άģ����Ϣ
	string rotationAngleXY_file = "F:/ITK/cuda-test/data/rotationAngleXY.dat";//360��double
	string rotationAngleYZ_file = "F:/ITK/cuda-test/data/rotationAngleYZ.dat";//31��double
	string template_roXY_file = "F:/ITK/cuda-test-lib/data/templateXY.tif";//200*200*360��float������matlab�������ȴ洢������һ�������ٴ�ڶ�������
	string template_roYZ_file = "F:/ITK/cuda-test/data/template_roYZ.dat";//200*50*31��float������matlab�������ȴ洢������һ�������ٴ�ڶ�������
	//��ȡ���ڷ�������fixImage
	string fixImage_file = "F:/ITK/cuda-test/data/toAffineWithZBB.tif";


	string imgBeforeRecon_path = "F:\\ITK\\cuda-test-lib\\data\\r20210824_X10_4bin";
	vector<string> imgBeforeRecon_Names;
	getFileNames(imgBeforeRecon_path, imgBeforeRecon_Names);

	string modelPath = "affineNetScript_TM_0621.pt";

	FishImageProcess fishImgProc(modelPath);
	
	fishImgProc.readPSFfromFile(PSF_1_file);
	fishImgProc.readRotationAngleFromFile(rotationAngleXY_file, rotationAngleYZ_file);
	fishImgProc.readTemplateFromFile(template_roXY_file, template_roYZ_file);
	fishImgProc.readFixImageFromFile(fixImage_file);

	fishImgProc.prepareGPUMemory();
	fishImgProc.processPSF();

	for (int i = 0; i < 1; i++)
	{
		fishImgProc.readImageFromFile(imgBeforeRecon_Names[i]);
		fishImgProc.reconImage();//�ع���������ͼ��
		fishImgProc.cropReconImage();   

		//rotation
		fishImgProc.matchingANDrotationXY();

		//crop
		fishImgProc.cropRotatedImage();

		//crop�Ľ������movingTensor����fixTensorһ���������紦��
		fishImgProc.libtorchModelProcess();

		//save and check
		string saveName1 = "F:\\ITK\\cuda-test-lib\\data\\testRecon\\" + int2string(4, i) + ".tif";
		string saveName2 = "F:\\ITK\\cuda-test-lib\\data\\testMatchingXY\\" + int2string(4, i) + ".tif";
		string saveName3 = "F:\\ITK\\cuda-test-lib\\data\\testCrop\\" + int2string(4, i) + ".tif";
		
		//76 * 95 * 50
		//200*200*50
		//test reconstruction
		float* temp = new float[200 * 200 * 50]();
		hipMemcpy(temp, fishImgProc.gpuObjRecon_crop, sizeof(float) * 200 * 200 * 50, hipMemcpyDeviceToHost);
		saveAndCheckImage(temp, 200, 200, 50, saveName1);
		//test rotation
		float* temp1 = new float[200 * 200 * 50]();
		hipMemcpy(temp1, fishImgProc.imageRotated3D_gpu, sizeof(float) * 200 * 200 * 50, hipMemcpyDeviceToHost);
		saveAndCheckImage(temp1, 200 , 200 , 50, saveName2);
		//test crop
		float* temp2 = new float[76 * 95 * 50]();
		hipMemcpy(temp2, fishImgProc.ObjCropRed_gpu, sizeof(float) * 76 * 95 * 50, hipMemcpyDeviceToHost);
		saveAndCheckImage(temp2, 76 , 95 , 49, saveName3);

		free(temp);
		free(temp1);
		free(temp2);
	}

	fishImgProc.freeMemory();

	return 0;
}