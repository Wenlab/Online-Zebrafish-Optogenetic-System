
#include <hip/hip_runtime.h>
//#define _CRT_SECURE_NO_WARNINGS
//
//#include"imgProcess.h"
////#include "imgProcess.cu"
//#include"kexinLibs.h"
//
//#include<vector>
//#include<iostream>
//
//#include"Timer.h"
//
//using std::string;
//using std::vector;
//using std::cout;
//using std::endl;
//
//
//
//int main()
//{
//	//��ʱ
//	Timer timer1;
//	
//
//
//	//��ȡPSF��δ�ع����ļ�
//	string PSF_1_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/PSF_1_zhuanzhi_float.dat";//matlab�б��������float����
//	string X31_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/r20210924_2_X31_resize.tif";
//	//��ȡ�Ƕȡ���άģ����Ϣ
//	string rotationAngleXY_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/rotationAngleXY.dat";//360��double
//	string rotationAngleYZ_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/rotationAngleYZ.dat";//31��double
//	string template_roXY_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/templateXY.tif";//200*200*360��float������matlab�������ȴ洢������һ�������ٴ�ڶ�������
//	string template_roYZ_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/template_roYZ.dat";//200*50*31��float������matlab�������ȴ洢������һ�������ٴ�ڶ�������
//	//��ȡ���ڷ�������fixImage
//	string fixImage_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/toAffineWithZBB.tif";
//
//
//	string imgBeforeRecon_path = "D:/kexin/Online-Zebrafish-Optogenetic/data/r20210824_X10_4bin";
//	vector<string> imgBeforeRecon_Names;
//	getFileNames(imgBeforeRecon_path, imgBeforeRecon_Names);
//
//	string modelPath = "affineNetScript_TM_0621_3080.pt";
//
//	FishImageProcess fishImgProc(modelPath);
//	
//	fishImgProc.readPSFfromFile(PSF_1_file);
//	fishImgProc.readRotationAngleFromFile(rotationAngleXY_file, rotationAngleYZ_file);
//	fishImgProc.readTemplateFromFile(template_roXY_file, template_roYZ_file);
//	fishImgProc.readFixImageFromFile(fixImage_file);
//	                                                                                                                                   
//	fishImgProc.initializeFishReg("anatomyList_4bin.txt");
//
//	fishImgProc.prepareGPUMemory();
//	fishImgProc.processPSF();
//
//	for (int i = 0; i < imgBeforeRecon_Names.size(); i++)
//	{
//		fishImgProc.readImageFromFile(imgBeforeRecon_Names[i]);
//
//		timer1.start();
//		fishImgProc.reconImage();//�ع���������ͼ��
//		fishImgProc.cropReconImage();   
//
//		//rotation
//		fishImgProc.matchingANDrotationXY();
//
//		//crop
//		fishImgProc.cropRotatedImage();
//
//		////crop�Ľ������movingTensor����fixTensorһ���������紦��
//		fishImgProc.libtorchModelProcess();
//
//		////���rotation/crop/affine������������ת��
//		std::vector<cv::Point3f> points = fishImgProc.ZBB2FishTransform();
//
//
//		timer1.stop();
//		cout << "time cost: " << timer1.getElapsedTimeInMilliSec() << " ms" << endl;
//
//		//for (int j = 0; j < points.size(); j++)
//		//{
//		//	cout << points[j] << endl;
//		//}
//
//		//save and check
//		string saveName1 = "D:/kexin/Online-Zebrafish-Optogenetic/data/testRecon/" + int2string(4, i) + ".tif";
//		string saveName2 = "D:/kexin/Online-Zebrafish-Optogenetic/data/testMatchingXY/" + int2string(4, i) + ".tif";
//		string saveName3 = "D:/kexin/Online-Zebrafish-Optogenetic/data/testCrop/" + int2string(4, i) + ".tif";
//		
//		//76 * 95 * 50
//		//200*200*50
//		//test reconstruction
//		float* temp = new float[200 * 200 * 50]();
//		cudaMemcpy(temp, fishImgProc.gpuObjRecon_crop, sizeof(float) * 200 * 200 * 50, cudaMemcpyDeviceToHost);
//		saveAndCheckImage(temp, 200, 200, 50, saveName1);
//		//test rotation
//		float* temp1 = new float[200 * 200 * 50]();
//		cudaMemcpy(temp1, fishImgProc.imageRotated3D_gpu, sizeof(float) * 200 * 200 * 50, cudaMemcpyDeviceToHost);
//		saveAndCheckImage(temp1, 200 , 200 , 50, saveName2);
//		//test crop
//		float* temp2 = new float[76 * 95 * 50]();
//		cudaMemcpy(temp2, fishImgProc.ObjCropRed_gpu, sizeof(float) * 76 * 95 * 50, cudaMemcpyDeviceToHost);
//		saveAndCheckImage(temp2, 76 , 95 , 49, saveName3);
//
//		free(temp);
//		free(temp1);
//		free(temp2);
//	}
//
//	fishImgProc.freeMemory();
//
//	return 0;
//}